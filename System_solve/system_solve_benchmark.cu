#include "hip/hip_runtime.h"
#include <string> 
#include <omp.h>

#include "utils.h"

#include "mkl.h"
#include "hipsolver.h"
#include "hipsolver.h"
#include <hipsparse.h>



// cusolver has HIPSOLVER_STATUS_SUCCESS and not hipSuccess, but they are the same
// this seems again kinda hacky
#define cudaErrchk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cusolverErrchk(ans) { cusolverAssert((ans), __FILE__, __LINE__); }
inline void cusolverAssert(hipsolverStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPSOLVER_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUSOLVERassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}


#define cublasErrchk(ans) { cublasAssert((ans), __FILE__, __LINE__); }
inline void cublasAssert(hipblasStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPBLAS_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUBLASassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}

#define cusparseErrchk(ans) { cusparseAssert((ans), __FILE__, __LINE__); }
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPSPARSE_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUSPARSEassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}

hipsolverHandle_t CreateCusolverDnHandle(int device) {
    if (hipSetDevice(device) != hipSuccess) {
        throw std::runtime_error("Failed to set CUDA device.");
    }
    hipsolverHandle_t handle;
    cusolverErrchk(hipsolverDnCreate(&handle));
    return handle;
}


double solve_mkl_dgesv(
    double *matrix_dense,
    double *rhs,
    double *reference_solution,
    int matrix_size,
    double abstol,
    double reltol,
    bool flag_verbose)
{

    double time = -1.0;


    int ipiv[matrix_size];
    int nrhs = 1;
    int info;
    time = -omp_get_wtime();
    info = LAPACKE_dgesv(LAPACK_COL_MAJOR, matrix_size, nrhs,
                        matrix_dense, matrix_size, ipiv, rhs, matrix_size);
    time += omp_get_wtime();

    if(info != 0){
        std::printf("Error in MKL dgesv\n");
        std::printf("info: %d\n", info);
        if(info > 0){
            std::printf("Singular");
        }
    }

    if(flag_verbose){
        std::printf("MKL dgesv done\n");
    }
    double relative_error[1];
    if(!assert_array_magnitude<double>(rhs,
            reference_solution, 
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        
        
        std::printf("Error: MKL dgesv solution is not the same as the reference solution\n");
    }
    else{
        std::printf("MKL dgesv solution is the same as the reference solution\n");
    }
    return time;
}

double solve_mkl_dposv(
    double *matrix_dense,
    double *rhs,
    double *reference_solution,
    int matrix_size,
    double abstol,
    double reltol,
    bool flag_verbose)
{

    double time = -1.0;


    int nrhs = 1;
    int info;
    time = -omp_get_wtime();
    char uplo = 'U';
    info = LAPACKE_dposv(LAPACK_COL_MAJOR,
                        uplo,
                        matrix_size,
                        nrhs,
                        matrix_dense,
                        matrix_size,
                        rhs,
                        matrix_size);
    time += omp_get_wtime();

    if(info != 0){
        std::printf("Error in MKL dposv\n");
        std::printf("info: %d\n", info);
        if(info > 0){
            std::printf("Singular");
        }
    }

    if(flag_verbose){
        std::printf("MKL dposv done\n");
    }
    double relative_error[1];
    if(!assert_array_magnitude<double>(rhs,
            reference_solution, 
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: MKL dposv solution is not the same as the reference solution\n");
    }
    else{
        std::printf("MKL dposv solution is the same as the reference solution\n");
    }
    return time;
}

double solve_mkl_dgbsv(
    double *matrix_band,
    double *rhs,
    double *reference_solution,
    int matrix_size,
    int kl,
    int ku,
    double abstol,
    double reltol,
    bool flag_verbose)
{

    double time = -1.0;


    int ipiv[matrix_size];
    int nrhs = 1;
    int info;
    int ldab = 2*kl + ku + 1;
    time = -omp_get_wtime();
    info = LAPACKE_dgbsv(LAPACK_COL_MAJOR, matrix_size, kl, ku, nrhs,
            matrix_band, ldab,
            ipiv, rhs, matrix_size);
    time += omp_get_wtime();

    if(info != 0){
        std::printf("Error in MKL dgbsv\n");
        std::printf("info: %d\n", info);
        if(info > 0){
            std::printf("Singular");
        }
    }

    if(flag_verbose){
        std::printf("MKL dgesv done\n");
    }
    double relative_error[1];
    if(!assert_array_magnitude<double>(rhs,
            reference_solution,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: MKL dgbsv solution is not the same as the reference solution\n");
    }
    else{
        std::printf("MKL dgbsv solution is the same as the reference solution\n");
    }
    return time;
}


double solve_mkl_dpbsv(
    double *matrix_band,
    double *rhs,
    double *reference_solution,
    int matrix_size,
    int kd,
    double abstol,
    double reltol,
    bool flag_verbose)
{

    double time = -1.0;


    int nrhs = 1;
    int info;
    int ldab = kd + 1;
    char order = 'U';
    time = -omp_get_wtime();
    info = LAPACKE_dpbsv(LAPACK_COL_MAJOR,
                order,
                matrix_size,
                kd,
                nrhs,
                matrix_band,
                ldab,
                rhs,
                matrix_size);

    time += omp_get_wtime();

    if(info != 0){
        std::printf("Error in MKL LAPACKE_dpbsv\n");
        std::printf("info: %d\n", info);
        if(info > 0){
            std::printf("Singular");
        }
    }

    if(flag_verbose){
        std::printf("MKL pbsv done\n");
    }
    double relative_error[1];
    if(!assert_array_magnitude<double>(rhs,
            reference_solution,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: MKL dpbsv solution is not the same as the reference solution\n");
    }
    else{
        std::printf("MKL dpbsv solution is the same as the reference solution\n");
    }
    return time;
}


double solve_cusolver_dense_LU(
    double *matrix_dense_h,
    double *rhs_h,
    double *reference_solution_h,
    int matrix_size,
    double abstol,
    double reltol,
    bool flag_verbose)
{

    double time = -1.0;
    hipStream_t stream = NULL;
    hipsolverHandle_t handle = CreateCusolverDnHandle(0);
    cudaErrchk(hipStreamCreate(&stream));
    cusolverErrchk(hipsolverSetStream(handle, stream));



    int info_h = 0;
    int bufferSize = 0;

    double *matrix_dense_d = NULL;
    double *rhs_d = NULL;
    int *ipiv_d = NULL;
    int *info_d = NULL;
    double *buffer = NULL;

    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&info_d, sizeof(int)))
    cudaErrchk(hipMalloc((void**)&matrix_dense_d, matrix_size*matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&ipiv_d, matrix_size*sizeof(int)));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(matrix_dense_d, matrix_dense_h, matrix_size*matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemset(info_d, 0, sizeof(int)));
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));


    //figure out extra amount of memory needed
    cusolverErrchk(hipsolverDnDgetrf_bufferSize(handle, matrix_size, matrix_size,
                                            (double *)matrix_dense_d,
                                              matrix_size, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, sizeof(double) * bufferSize));

    //LU factorization
    if(flag_verbose){
        std::printf("LU factorization\n");
    }
    time = -omp_get_wtime();
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    cusolverErrchk(hipsolverDnDgetrf(handle, matrix_size, matrix_size,
                                matrix_dense_d, matrix_size, buffer, ipiv_d, info_d));
    
    //copy info to host
    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));

    if (info_h != 0) {
        fprintf(stderr, "Error: LU factorization failed\n");
    }
    else{
        std::printf("LU factorization done\n");
    }

    if(flag_verbose){
        std::printf("Back substitution\n");
    }
    //back substitution
    cusolverErrchk(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, matrix_size,
                                    1, matrix_dense_d, matrix_size, ipiv_d,
                                    rhs_d, matrix_size, info_d));
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    time += omp_get_wtime();


    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));
    if (info_h != 0) {
        fprintf(stderr, "Error: Back substitution failed\n");
    }
    else{
        std::printf("Back substitution done\n");
    }

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, rhs_d, matrix_size*sizeof(double), hipMemcpyDeviceToHost));
    double relative_error[1];
    if(!assert_array_magnitude<double>(rhs_h,
            reference_solution_h,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: CuSolver LU solution is not the same as the reference solution\n");
    }
    else{
        std::printf("CuSolver LU solution is the same as the reference solution\n");
    }


    if (info_d) {
        cudaErrchk(hipFree(info_d));
    }
    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if (matrix_dense_d) {
        cudaErrchk(hipFree(matrix_dense_d));
    }
    if(rhs_d) {
        cudaErrchk(hipFree(rhs_d));
    }
    if (ipiv_d) {
        cudaErrchk(hipFree(ipiv_d));
    }


    if (handle) {
        cusolverErrchk(hipsolverDnDestroy(handle));
    }
    if (stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }

    return time;
}


double solve_cusolver_dense_CHOL(
    double *matrix_dense_h,
    double *rhs_h,
    double *reference_solution_h,
    int matrix_size,
    double abstol,
    double reltol,
    bool flag_verbose)
{

    double time = -1.0;
    hipStream_t stream = NULL;
    hipsolverHandle_t handle = CreateCusolverDnHandle(0);
    cudaErrchk(hipStreamCreate(&stream));
    cusolverErrchk(hipsolverSetStream(handle, stream));



    int info_h = 0;
    int bufferSize = 0;

    double *matrix_dense_d = NULL;
    double *rhs_d = NULL;
    int *info_d = NULL;
    double *buffer = NULL;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&info_d, sizeof(int)))
    cudaErrchk(hipMalloc((void**)&matrix_dense_d, matrix_size*matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(matrix_dense_d, matrix_dense_h, matrix_size*matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemset(info_d, 0, sizeof(int)));
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));


    //figure out extra amount of memory needed
    cusolverErrchk(hipsolverDnDpotrf_bufferSize(handle, uplo, matrix_size,
                                            (double *)matrix_dense_d,
                                              matrix_size, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, sizeof(double) * bufferSize));

    //LU factorization
    if(flag_verbose){
        std::printf("CHOL factorization\n");
    }
    time = -omp_get_wtime();
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    cusolverErrchk(hipsolverDnDpotrf(handle, uplo, matrix_size,
                                matrix_dense_d, matrix_size, buffer, bufferSize, info_d));
    
    //copy info to host
    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));

    if (info_h != 0) {
        fprintf(stderr, "Error: CHOL factorization failed\n");
    }
    else{
        std::printf("CHOL factorization done\n");
    }

    if(flag_verbose){
        std::printf("Back substitution\n");
    }
    //back substitution
    cusolverErrchk(hipsolverDnDpotrs(handle, uplo, matrix_size,
                                    1, matrix_dense_d, matrix_size,
                                    rhs_d, matrix_size, info_d));
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    time += omp_get_wtime();


    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));
    if (info_h != 0) {
        fprintf(stderr, "Error: Back substitution failed\n");
    }
    else{
        std::printf("Back substitution done\n");
    }

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, rhs_d, matrix_size*sizeof(double), hipMemcpyDeviceToHost));
    double relative_error[1];
    if(!assert_array_magnitude<double>(rhs_h,
            reference_solution_h,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: CuSolver CHOL solution is not the same as the reference solution\n");
    }
    else{
        std::printf("CuSolver CHOL solution is the same as the reference solution\n");
    }


    if (info_d) {
        cudaErrchk(hipFree(info_d));
    }
    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if (matrix_dense_d) {
        cudaErrchk(hipFree(matrix_dense_d));
    }
    if(rhs_d) {
        cudaErrchk(hipFree(rhs_d));
    }

    if (handle) {
        cusolverErrchk(hipsolverDnDestroy(handle));
    }
    if (stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }

    return time;
}


double solve_cusparse_CG(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    double *starting_guess_h,
    int nnz,
    int matrix_size,
    double abstol,
    double reltol,
    double restol,
    bool flag_verbose,
    int *steps_taken,
    double *relative_error)
{

    double time = -1.0;
    hipStream_t stream = NULL;
    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    cudaErrchk(hipStreamCreate(&stream));
    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    int *col_indices_d = NULL;
    int *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double dot;

    hipsparseSpMatDescr_t matA = NULL;

    const int max_iter = 100000;
    double a, b, na;
    double alpha, beta, alpham1, r0, r1;
    size_t bufferSize = 0;
    void *buffer = NULL;

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;


    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrix_size+1)*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrix_size * sizeof(double)));

    /* Wrap raw data into cuSPARSE generic API objects */
    cusparseErrchk(hipsparseCreateCsr(&matA, matrix_size, matrix_size,
                                        nnz, row_indptr_d, col_indices_d, data_d,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));



    hipsparseDnVecDescr_t vecx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecx, matrix_size, x_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrix_size, p_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecAx, matrix_size, Ax_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrix_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(x_d, starting_guess_h, matrix_size * sizeof(double), hipMemcpyHostToDevice));    

    //figure out extra amount of memory needed
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, bufferSize));


    //begin CG
    time = -omp_get_wtime();
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    // calc A*x
    cusparseErrchk(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    // r = b - A*x
    cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpham1, Ax_d, 1, rhs_d, 1));
    cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));


    int k = 1;
    while (r1 > restol * restol && k <= max_iter) {
        if(k > 1){
            b = r1 / r0;
            cublasErrchk(hipblasDscal(cublasHandle, matrix_size, &b, p_d, 1));
            cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpha, rhs_d, 1, p_d, 1));            
        }
        else {
            cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, rhs_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, p_d, 1, Ax_d, 1, &dot));
        a = r1 / dot;

        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &a, p_d, 1, x_d, 1));
        na = -a;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &na, Ax_d, 1, rhs_d, 1));

        r0 = r1;
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));
        cudaErrchk(hipStreamSynchronize(stream));

        k++;
    }

    steps_taken[0] = k;
    std::printf("iteration = %3d, residual = %e\n", k, sqrt(r1));


    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    time += omp_get_wtime();

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrix_size * sizeof(double), hipMemcpyDeviceToHost));

    if(!assert_array_magnitude<double>(
            rhs_h,
            reference_solution_h,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: CG solution is not the same as the reference solution\n");
    }
    else{
        std::printf("CG solution is the same as the reference solution\n");
    }



    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(cublasHandle) {
        cublasErrchk(hipblasDestroy(cublasHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }
    if(matA) {
        cusparseErrchk(hipsparseDestroySpMat(matA));
    }
    if(vecx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecx));
    }
    if(vecAx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecAx));
    }
    if(vecp) {
        cusparseErrchk(hipsparseDestroyDnVec(vecp));
    }

    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }
    if(p_d){
        cudaErrchk(hipFree(p_d));
    }
    if(Ax_d){
        cudaErrchk(hipFree(Ax_d));
    }

    return time;
}

double solve_cusparse_ILU_CG(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    int nnz,
    int matrix_size,
    double abstol,
    double reltol,
    double restol,
    bool flag_verbose,
    int *steps_taken,
    double *relative_error)
{

    double time = -1.0;

    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    hipStream_t stream = NULL;
    cudaErrchk(hipStreamCreate(&stream));

    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    int *col_indices_d = NULL;
    int *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double *valsILU0_d = NULL;
    double *zm1_d = NULL;
    double *zm2_d = NULL;
    double *rm2_d = NULL;
    double *omega_d = NULL;
    double *y_d = NULL;

    const int max_iter = 100000;
    double alpha, beta, r1;
    double numerator, denominator, nalpha;
    const double doubleone = 1.0;
    const double doublezero = 0.0;

    alpha = 1.0;
    beta = 0.0;


    hipsparseSpMatDescr_t matA = NULL;
    hipsparseSpMatDescr_t matM_lower = NULL;
    hipsparseSpMatDescr_t matM_upper = NULL;
    hipsparseFillMode_t   fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t   diag_unit     = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t   fill_upper    = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t   diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;


    int                 bufferSizeLU = 0;
    size_t              bufferSizeMV, bufferSizeL, bufferSizeU;
    void*               bufferLU_d, *bufferMV_d,  *bufferL_d, *bufferU_d;
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseMatDescr_t   matLU;
    csrilu02Info_t      infoILU = NULL;


    /* Description of the A matrix */
    hipsparseMatDescr_t descr = 0;
    cusparseErrchk(hipsparseCreateMatDescr(&descr));
    cusparseErrchk(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseErrchk(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrix_size+1)*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&y_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&omega_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&valsILU0_d, nnz * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&zm1_d, (matrix_size) * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&zm2_d, (matrix_size) * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&rm2_d, (matrix_size) * sizeof(double)));


    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseDnVecDescr_t vecp = NULL, vecX=NULL, vecY = NULL, vecR = NULL, vecZM1=NULL;
    hipsparseDnVecDescr_t vecomega = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrix_size, p_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecX, matrix_size, x_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecY, matrix_size, y_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecR, matrix_size, rhs_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecZM1, matrix_size, zm1_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecomega, matrix_size, omega_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrix_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(valsILU0_d, data_d, nnz*sizeof(double), hipMemcpyDeviceToDevice));
    // setting starting guess to zero
    cudaErrchk(hipMemset(x_d, 0.0, matrix_size*sizeof(double)))


    cusparseErrchk(hipsparseCreateCsr(
        &matA, matrix_size, matrix_size, nnz, row_indptr_d, col_indices_d, data_d, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    
    //Lower Part 
     cusparseErrchk(hipsparseCreateCsr(&matM_lower, matrix_size, matrix_size, nnz, row_indptr_d, col_indices_d, valsILU0_d,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    cusparseErrchk(hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_lower, sizeof(fill_lower)));
    cusparseErrchk(hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_unit, sizeof(diag_unit)));
    // M_upper
    cusparseErrchk(hipsparseCreateCsr(&matM_upper, matrix_size, matrix_size, nnz, row_indptr_d, col_indices_d, valsILU0_d,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    cusparseErrchk(hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_upper, sizeof(fill_upper)));
    cusparseErrchk(hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_non_unit,
                                              sizeof(diag_non_unit)));


    /* Create ILU(0) info object */
    cusparseErrchk(hipsparseCreateCsrilu02Info(&infoILU));
    cusparseErrchk(hipsparseCreateMatDescr(&matLU) );
    cusparseErrchk(hipsparseSetMatType(matLU, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    cusparseErrchk(hipsparseSetMatIndexBase(matLU, HIPSPARSE_INDEX_BASE_ZERO) );

    /* Allocate workspace for cuSPARSE */
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matA,
        vecp, &doublezero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
        &bufferSizeMV));
    cudaErrchk( hipMalloc(&bufferMV_d, bufferSizeMV) );

    cusparseErrchk(hipsparseDcsrilu02_bufferSize(
        cusparseHandle, matrix_size, nnz, matLU, data_d, row_indptr_d, col_indices_d, infoILU, &bufferSizeLU));
    cudaErrchk( hipMalloc(&bufferLU_d, bufferSizeLU) );

    cusparseErrchk(hipsparseSpSV_createDescr(&spsvDescrL) );
    cusparseErrchk(hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matM_lower, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL));
    cudaErrchk(hipMalloc(&bufferL_d, bufferSizeL) );

    cusparseErrchk(hipsparseSpSV_createDescr(&spsvDescrU) );
    cusparseErrchk(hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matM_upper, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSizeU));
    cudaErrchk(hipMalloc(&bufferU_d, bufferSizeU) );



    //begin CG
    time = -omp_get_wtime();
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    /* Preconditioned Conjugate Gradient using ILU.
       --------------------------------------------
       Follows the description by Golub & Van Loan,
       "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

    printf("Convergence of CG using ILU(0) preconditioning: \n");



    /* Perform analysis for ILU(0) */
    cusparseErrchk(hipsparseDcsrilu02_analysis(
        cusparseHandle, matrix_size, nnz, descr, valsILU0_d, row_indptr_d, col_indices_d, infoILU,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, bufferLU_d));

    /* generate the ILU(0) factors */
    cusparseErrchk(hipsparseDcsrilu02(
        cusparseHandle, matrix_size, nnz, matLU, valsILU0_d, row_indptr_d, col_indices_d, infoILU,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, bufferLU_d));

    /* perform triangular solve analysis */
    cusparseErrchk(hipsparseSpSV_analysis(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone,
        matM_lower, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, bufferL_d));

    cusparseErrchk(hipsparseSpSV_analysis(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone,
        matM_upper, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, bufferU_d));

    // /* reset the initial guess of the solution to zero */
    // for (int i = 0; i < matrix_size; i++)
    // {
    //     x[i] = 0.0;
    // }
    // cudaErrchk(hipMemcpy(
    //     rhs_d, rhs, matrix_size * sizeof(double), hipMemcpyHostToDevice));
    // cudaErrchk(hipMemcpy(
    //     x_d, x, matrix_size * sizeof(double), hipMemcpyHostToDevice));

    int k = 0;
    cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));

    while (r1 > restol * restol && k <= max_iter)
    {
        // preconditioner application: zm1_d = U^-1 L^-1 rhs_d
        cusparseErrchk(hipsparseSpSV_solve(cusparseHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone,
            matM_lower, vecR, vecY, HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT,
            spsvDescrL) );
            
        cusparseErrchk(hipsparseSpSV_solve(cusparseHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matM_upper,
            vecY, vecZM1,
            HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT,
            spsvDescrU));
        k++;

        if (k == 1)
        {
            cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, zm1_d, 1, p_d, 1));
        }
        else
        {
            cublasErrchk(hipblasDdot(
                cublasHandle, matrix_size, rhs_d, 1, zm1_d, 1, &numerator));
            cublasErrchk(hipblasDdot(
                cublasHandle, matrix_size, rm2_d, 1, zm2_d, 1, &denominator));
            beta = numerator / denominator;
            cublasErrchk(hipblasDscal(cublasHandle, matrix_size, &beta, p_d, 1));
            cublasErrchk(hipblasDaxpy(
                cublasHandle, matrix_size, &doubleone, zm1_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matA,
            vecp, &doublezero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
            bufferMV_d));
    
        cublasErrchk(hipblasDdot(
            cublasHandle, matrix_size, rhs_d, 1, zm1_d, 1, &numerator));
        cublasErrchk(hipblasDdot(
            cublasHandle, matrix_size, p_d, 1, omega_d, 1, &denominator));

        alpha = numerator / denominator;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpha, p_d, 1, x_d, 1));
        cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, rhs_d, 1, rm2_d, 1));
        cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, zm1_d, 1, zm2_d, 1));
        nalpha = -alpha;
        cublasErrchk(hipblasDaxpy(
            cublasHandle, matrix_size, &nalpha, omega_d, 1, rhs_d, 1));
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));
    }

    steps_taken[0] = k;
    std::printf("iteration = %3d, residual = %e \n", k, sqrt(r1));


    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    time += omp_get_wtime();

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrix_size * sizeof(double), hipMemcpyDeviceToHost));

    if(!assert_array_magnitude<double>(rhs_h,
            reference_solution_h,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: ILU CG solution is not the same as the reference solution\n");
    }
    else{
        std::printf("ILU CG solution is the same as the reference solution\n");
    }


    /* Destroy descriptors */
    if(descr) {
        cusparseErrchk(hipsparseDestroyMatDescr(descr));
    }
    if(matA) {
        cusparseErrchk(hipsparseDestroySpMat(matA));
    }
    if(vecp) {
        cusparseErrchk(hipsparseDestroyDnVec(vecp));
    }
    if(vecX) {
        cusparseErrchk(hipsparseDestroyDnVec(vecX));
    }
    if(vecY) {
        cusparseErrchk(hipsparseDestroyDnVec(vecY));
    }
    if(vecR) {
        cusparseErrchk(hipsparseDestroyDnVec(vecR));
    }
    if(vecZM1) {
        cusparseErrchk(hipsparseDestroyDnVec(vecZM1));
    }
    if(vecomega) {
        cusparseErrchk(hipsparseDestroyDnVec(vecomega));
    }
    if(matM_lower) {
        cusparseErrchk(hipsparseDestroySpMat(matM_lower));
    }
    if(matM_upper) {
        cusparseErrchk(hipsparseDestroySpMat(matM_upper));
    }
    if(matLU) {
        cusparseErrchk(hipsparseDestroyMatDescr(matLU));
    }
    if(spsvDescrL) {
        cusparseErrchk(hipsparseSpSV_destroyDescr(spsvDescrL));
    }
    if(spsvDescrU) {
        cusparseErrchk(hipsparseSpSV_destroyDescr(spsvDescrU));
    }
    if(infoILU) {
        cusparseErrchk(hipsparseDestroyCsrilu02Info(infoILU));
    }


    //Destroy handles
    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(cublasHandle) {
        cublasErrchk(hipblasDestroy(cublasHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }


    // Destroy buffer
    //bufferLU_d, *bufferMV_d,  *bufferL_d, *bufferU_d;
    if (bufferLU_d) {
        cudaErrchk(hipFree(bufferLU_d));
    }
    if (bufferMV_d) {
        cudaErrchk(hipFree(bufferMV_d));
    }
    if (bufferL_d) {
        cudaErrchk(hipFree(bufferL_d));
    }
    if (bufferU_d) {
        cudaErrchk(hipFree(bufferU_d));
    }


    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }
    if(y_d){
        cudaErrchk(hipFree(y_d));
    }
    if(p_d){
        cudaErrchk(hipFree(p_d));
    }
    if(omega_d){
        cudaErrchk(hipFree(omega_d));
    }
    if(Ax_d){
        cudaErrchk(hipFree(Ax_d));
    }
    if(valsILU0_d){
        cudaErrchk(hipFree(valsILU0_d));
    }
    if(zm1_d){
        cudaErrchk(hipFree(zm1_d));
    }
    if(zm2_d){
        cudaErrchk(hipFree(zm2_d));
    }
    if(rm2_d){
        cudaErrchk(hipFree(rm2_d));
    }

    return time;
}

void extract_diagonal_values(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_indptr[i]; j < row_indptr[i+1]; j++){
            if(col_indices[j] == i){
                diagonal_values_inv_sqrt[i] = 1/std::sqrt(data[j]);
                break;
            }
        }
    }

}

__global__ void jacobi_precondition_array(
    double *array,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        array[i] = array[i] * diagonal_values_inv_sqrt[i];
    }

}

__global__ void jacobi_unprecondition_array(
    double *array,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        array[i] = array[i] * 1/diagonal_values_inv_sqrt[i];
    }

}




__global__ void jacobi_precondition_matrix(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_indptr[i]; j < row_indptr[i+1]; j++){
            data[j] = data[j] *
            diagonal_values_inv_sqrt[i] * diagonal_values_inv_sqrt[col_indices[j]];
        }
    }

}


double solve_cusparse_CG_jacobi(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    double *starting_guess_h,
    int nnz,
    int matrix_size,
    double abstol,
    double reltol,
    double restol,
    bool flag_verbose,
    int *steps_taken,
    double *relative_error)
{

    double time = -1.0;
    hipStream_t stream = NULL;
    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    cudaErrchk(hipStreamCreate(&stream));
    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    int *col_indices_d = NULL;
    int *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double dot;

    hipsparseSpMatDescr_t matA = NULL;

    const int max_iter = 100000;
    double a, b, na;
    double alpha, beta, alpham1, r0, r1;
    size_t bufferSize = 0;
    void *buffer = NULL;

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;

    double *diagonal_values_inv_sqrt_h = (double *)malloc(matrix_size * sizeof(double));
    extract_diagonal_values(
        data_h,
        col_indices_h,
        row_indptr_h,
        diagonal_values_inv_sqrt_h,
        matrix_size
    );
    double *diagonal_values_inv_sqrt_d = NULL;
    


    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrix_size+1)*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void**)&diagonal_values_inv_sqrt_d, matrix_size*sizeof(double)));

    /* Wrap raw data into cuSPARSE generic API objects */
    cusparseErrchk(hipsparseCreateCsr(&matA, matrix_size, matrix_size,
                                        nnz, row_indptr_d, col_indices_d, data_d,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));



    hipsparseDnVecDescr_t vecx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecx, matrix_size, x_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrix_size, p_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecAx, matrix_size, Ax_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrix_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(x_d, starting_guess_h, matrix_size * sizeof(double), hipMemcpyHostToDevice));    
    cudaErrchk(hipMemcpy(diagonal_values_inv_sqrt_d, diagonal_values_inv_sqrt_h, matrix_size * sizeof(double), hipMemcpyHostToDevice));


    // precondition the matrix and right hand side
    // do it directly and not as solving another system
    int num_threads = 256;
    int num_blocks = (matrix_size + num_threads - 1) / num_threads;


    //figure out extra amount of memory needed
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, bufferSize));


    //begin CG
    time = -omp_get_wtime();

    // scale rhs
    jacobi_precondition_array<<<num_blocks, num_threads>>>(
        rhs_d,
        diagonal_values_inv_sqrt_d,
        matrix_size    
    );
    cudaErrchk( hipDeviceSynchronize() );
    // scale matrix
    jacobi_precondition_matrix<<<num_blocks, num_threads>>>(
        data_d,
        col_indices_d,
        row_indptr_d,
        diagonal_values_inv_sqrt_d,
        matrix_size
    );
    cudaErrchk( hipDeviceSynchronize() );
    // scale starting guess
    jacobi_unprecondition_array<<<num_blocks, num_threads>>>(
        x_d,
        diagonal_values_inv_sqrt_d,
        matrix_size    
    );

    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    // calc A*x
    cusparseErrchk(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    // r = b - A*x
    cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpham1, Ax_d, 1, rhs_d, 1));
    cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));


    int k = 1;
    while (r1 > restol * restol && k <= max_iter) {
        if(k > 1){
            b = r1 / r0;
            cublasErrchk(hipblasDscal(cublasHandle, matrix_size, &b, p_d, 1));
            cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpha, rhs_d, 1, p_d, 1));            
        }
        else {
            cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, rhs_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, p_d, 1, Ax_d, 1, &dot));
        a = r1 / dot;

        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &a, p_d, 1, x_d, 1));
        na = -a;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &na, Ax_d, 1, rhs_d, 1));

        r0 = r1;
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));
        cudaErrchk(hipStreamSynchronize(stream));

        k++;
    }

    steps_taken[0] = k;
    std::printf("iteration = %3d, residual = %e\n", k, sqrt(r1));


    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    

    // unprecondition solution
    jacobi_precondition_array<<<num_blocks, num_threads>>>(
        x_d,
        diagonal_values_inv_sqrt_d,
        matrix_size    
    );
    time += omp_get_wtime();
    cudaErrchk( hipDeviceSynchronize() );
    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrix_size * sizeof(double), hipMemcpyDeviceToHost));


    if(!assert_array_magnitude<double>(
            rhs_h,
            reference_solution_h,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: Jacobi CG solution is not the same as the reference solution\n");
    }
    else{
        std::printf("Jacobi CG solution is the same as the reference solution\n");
    }




    cusparseErrchk(hipsparseDestroy(cusparseHandle));

    cublasErrchk(hipblasDestroy(cublasHandle));

    cudaErrchk(hipStreamDestroy(stream));

    cusparseErrchk(hipsparseDestroySpMat(matA));

    cusparseErrchk(hipsparseDestroyDnVec(vecx));

    cusparseErrchk(hipsparseDestroyDnVec(vecAx));

    cusparseErrchk(hipsparseDestroyDnVec(vecp));


    cudaErrchk(hipFree(buffer));


    cudaErrchk(hipFree(data_d));

    cudaErrchk(hipFree(col_indices_d));


    cudaErrchk(hipFree(row_indptr_d));


    cudaErrchk(hipFree(rhs_d));


    cudaErrchk(hipFree(x_d));


    cudaErrchk(hipFree(p_d));


    cudaErrchk(hipFree(Ax_d));


    cudaErrchk(hipFree(diagonal_values_inv_sqrt_d));

    free(diagonal_values_inv_sqrt_h);

    return time;
}

double solve_cusolver_sparse_CHOL(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    int nnz,
    int matrix_size,
    double abstol,
    double reltol,
    bool flag_verbose)
{


    hipsolverSpHandle_t handle = NULL;
    hipsparseHandle_t cusparseHandle = NULL; /* used in residual evaluation */
    hipStream_t stream = NULL;
    hipsparseMatDescr_t descrA = NULL;

    cudaErrchk(hipStreamCreate(&stream));
    cusolverErrchk(hipsolverSpCreate(&handle));
    cusparseErrchk(hipsparseCreate(&cusparseHandle));

    cusolverErrchk(hipsolverSpSetStream(handle, stream));
    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));


    double *data_d = NULL;
    int *col_indices_d = NULL;
    int *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;

    const int reorder = 0;
    int singularity = 0;
    double singular_tol = 1.e-12;
    double time = -1.0;

    cudaErrchk(hipMalloc((void **)&row_indptr_d, sizeof(int) * (matrix_size + 1)));
    cudaErrchk(hipMalloc((void **)&col_indices_d, sizeof(int) * nnz));
    cudaErrchk(hipMalloc((void **)&data_d, sizeof(double) * nnz));
    cudaErrchk(hipMalloc((void **)&rhs_d, sizeof(double) * matrix_size));
    cudaErrchk(hipMalloc((void **)&x_d, sizeof(double) * matrix_size));

    // load data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, sizeof(int) * (matrix_size + 1), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, sizeof(int) * nnz, hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, sizeof(double) * nnz, hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, sizeof(double) * matrix_size, hipMemcpyHostToDevice));

    cudaErrchk(hipMemset(x_d, 0.0, matrix_size*sizeof(double)))


    cusparseErrchk(hipsparseCreateMatDescr(&descrA));
    cusparseErrchk(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseErrchk(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    time = -omp_get_wtime();
    if(flag_verbose){
        std::printf("Cholesky factorization\n");
    }
    cudaErrchk(hipStreamSynchronize(stream));

    cusolverErrchk(hipsolverSpDcsrlsvchol(
        handle, matrix_size, nnz, descrA, data_d, row_indptr_d, col_indices_d,
        rhs_d, singular_tol, reorder, x_d, &singularity));

    cudaErrchk(hipStreamSynchronize(stream));
    if(flag_verbose){
        std::printf("Cholesky factorization done\n");
    }
    time += omp_get_wtime();

    if (0 <= singularity) {
        printf("WARNING: the matrix is singular at row %d under tolerance (%E)\n",
            singularity, singular_tol);
    }


    cudaErrchk(hipMemcpy(rhs_h, x_d, matrix_size * sizeof(double), hipMemcpyDeviceToHost));

    double relative_error[1];

    if(!assert_array_magnitude<double>(rhs_h,
            reference_solution_h,
            abstol,
            reltol,
            matrix_size,
            relative_error)){
        std::printf("Error: CHOL solution is not the same as the reference solution\n");
    }
    else{
        std::printf("CHOL solution is the same as the reference solution\n");
    }

    //Destroy handles
    if(handle) {
        cusolverErrchk(hipsolverSpDestroy(handle));
    }
    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }
    if(descrA) {
        cusparseErrchk(hipsparseDestroyMatDescr(descrA));
    }

    //Destroy buffers
    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }


    return time;
}
