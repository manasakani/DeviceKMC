#include "hip/hip_runtime.h"
#include "utils_gpu.h"


__global__ void _extract_diagonal_inv_sqrt(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_indptr[i]; j < row_indptr[i+1]; j++){
            if(col_indices[j] == i){
                diagonal_values_inv_sqrt[i] = 1/std::sqrt(data[j]);
                break;
            }
        }
    }

}



void extract_diagonal_inv_sqrt(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int block_size = 1024;
    int num_blocks = (matrix_size + block_size - 1) / block_size;
    _extract_diagonal_inv_sqrt<<<num_blocks, block_size>>>(
        data,
        col_indices,
        row_indptr,
        diagonal_values_inv_sqrt,
        matrix_size
    );
}

__global__ void _extract_diagonal_inv(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_indptr[i]; j < row_indptr[i+1]; j++){
            if(col_indices[j] == i){
                diagonal_values_inv_sqrt[i] = 1/data[j];
                break;
            }
        }
    }

}



void extract_diagonal_inv(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int block_size = 1024;
    int num_blocks = (matrix_size + block_size - 1) / block_size;
    _extract_diagonal_inv<<<num_blocks, block_size>>>(
        data,
        col_indices,
        row_indptr,
        diagonal_values_inv_sqrt,
        matrix_size
    );
}

__global__ void _precondition_vector_gpu(
    double *array,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        array[i] = array[i] * diagonal_values_inv_sqrt[i];
    }

}
void precondition_vector_gpu(
    double *array,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int block_size = 1024;
    int num_blocks = (matrix_size + block_size - 1) / block_size;
    _precondition_vector_gpu<<<num_blocks, block_size>>>(
        array,
        diagonal_values_inv_sqrt,
        matrix_size
    );
}

__global__ void _unpreecondition_vector_gpu(
    double *array,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        array[i] = array[i] * 1/diagonal_values_inv_sqrt[i];
    }

}

void unpreecondition_vector_gpu(
    double *array,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int block_size = 1024;
    int num_blocks = (matrix_size + block_size - 1) / block_size;
    _unpreecondition_vector_gpu<<<num_blocks, block_size>>>(
        array,
        diagonal_values_inv_sqrt,
        matrix_size
    );
}


__global__ void _symmetric_precondition_matrix_gpu(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_indptr[i]; j < row_indptr[i+1]; j++){
            data[j] = data[j] *
            diagonal_values_inv_sqrt[i] * diagonal_values_inv_sqrt[col_indices[j]];
        }
    }
}

void symmetric_precondition_matrix_gpu(
    double *data,
    int *col_indices,
    int *row_indptr,
    double *diagonal_values_inv_sqrt,
    int matrix_size
)
{
    int block_size = 1024;
    int num_blocks = (matrix_size + block_size - 1) / block_size;
    _symmetric_precondition_matrix_gpu<<<num_blocks, block_size>>>(
        data,
        col_indices,
        row_indptr,
        diagonal_values_inv_sqrt,
        matrix_size
    );
}

__global__ void _invert_array(
    double *array_in,
    double *array_out,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        array_out[i] = 1/array_in[i];
    }
}

void invert_array(
    double *array_in,
    double *array_out,
    int matrix_size
)
{
    int block_size = 1024;
    int num_blocks = (matrix_size + block_size - 1) / block_size;
    _invert_array<<<num_blocks, block_size>>>(
        array_in,
        array_out,
        matrix_size
    );
}
