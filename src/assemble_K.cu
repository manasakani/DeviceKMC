#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include "gpu_buffers.h"
#include <iostream>
#include <omp.h>


#define NUM_THREADS 512

// returns true if thing is present in the array of things
template <typename T>
__device__ int is_in_array_gpu_og(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

template <typename T>
int is_in_array_cpu(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

__device__ double site_dist_gpu_og(double pos1x, double pos1y, double pos1z,
                                double pos2x, double pos2y, double pos2z,
                                double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

double site_dist_cpu(double pos1x, double pos1y, double pos1z,
                    double pos2x, double pos2y, double pos2z,
                    double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

__global__ void create_K_og(
    double *X,
    const double *posx, const double *posy, const double *posz,
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const double *lattice, const bool pbc, const double d_high_G, const double d_low_G,
    const double nn_dist, const int N, const int num_metals)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto idx = tid_total; idx < (size_t) N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        bool metal1 = is_in_array_gpu_og(metals, element[i], num_metals);
        bool metal2 = is_in_array_gpu_og(metals, element[j], num_metals);
        bool ischarged1 = site_charge[i] != 0;
        bool ischarged2 = site_charge[j] != 0;
        bool isVacancy1 = element[i] == VACANCY;
        bool isVacancy2 = element[j] == VACANCY;
        bool cvacancy1 = isVacancy1 && !ischarged1;
        bool cvacancy2 = isVacancy2 && !ischarged2;
        double dist = site_dist_gpu_og(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);

        bool neighbor = false;
        if (dist < nn_dist && i != j)
            neighbor = true;

        // direct terms:
        if (i != j && neighbor)
        {
            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                X[N * (i) + (j)] = -d_high_G;
            }
            else
            {
                X[N * (i) + (j)] = -d_low_G;
            }
        }
    }
}


template <int NTHREADS>
__global__ void diagonal_sum_og(double *A, double *diag, int N)
{

    int num_threads = blockDim.x;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int block_id = blockIdx.x;

    int tid = threadIdx.x;

    __shared__ double buf[NTHREADS];

    for (auto idx = block_id; idx < N * blocks_per_row; idx += gridDim.x)
    {

        int ridx = idx / blocks_per_row;
        int scol = (idx % blocks_per_row) * num_threads;
        int lcol = min(N, scol + num_threads);

        buf[tid] = 0.0;
        if (tid + scol < lcol)
        {
            buf[tid] = A[ridx * N + scol + tid];
        }

        int width = num_threads / 2;
        while (width != 0)
        {
            __syncthreads();
            if (tid < width)
            {
                buf[tid] += buf[tid + width];
            }
            width /= 2;
        }

        if (tid == 0)
        {
            atomicAdd(diag + ridx, buf[0]);
        }
    }
}

__global__ void set_diag_og(double *A, double *diag, int N)
{
    int didx = blockIdx.x * blockDim.x + threadIdx.x;
    if (didx < N)
    {
        double tmp = A[didx * N + didx];
        A[didx * N + didx] = 2 * tmp - diag[didx];
    }
}

template<typename T>
void sparse_to_dense(
    T *dense_matrix,
    T *data,
    int *col_indices,
    int *row_ptr,
    int matrix_size)
{

    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            // could not work for complex data type
            dense_matrix[i*matrix_size + j] = T(0);
        }
    }

    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            dense_matrix[i*matrix_size + col_indices[j]] = data[j];
        }
    }
}


template<typename T>
bool assert_array_magnitude(
    T *array_test,
    T *array_ref,
    double abstol,
    double reltol,
    int size)
{
    double sum_difference = 0.0;
    double sum_ref = 0.0;
    for (int i = 0; i < size; i++) {
        sum_difference += std::abs(array_test[i] - array_ref[i]) * std::abs(array_test[i] - array_ref[i]);
        sum_ref += std::abs(array_ref[i])*std::abs(array_ref[i]);

    }
    sum_difference = std::sqrt(sum_difference);
    sum_ref = std::sqrt(sum_ref);
    if (sum_difference > reltol * sum_ref + abstol) {
        std::printf("Arrays are in magnitude not the same\n");
        std::cout << "Difference " << sum_difference << std::endl;
        std::cout << "Relative " << sum_difference/sum_ref << std::endl;
        std::cout << "Mixed tolerance " << reltol * sum_ref + abstol << std::endl;
        return false;
    }

    return true;
}


double assemble_K_og(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double nn_dist,
                              const int num_metals, int kmc_step_count,
                              double *K_h)
{
    int N_interface = N - (N_left_tot + N_right_tot);

    double *VL, *VR;
    gpuErrchk( hipMalloc((void **)&VL, N_left_tot * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&VR, N_right_tot * sizeof(double)) );

    double *gpu_k;
    double *gpu_diag;
    gpuErrchk( hipMalloc((void **)&gpu_k, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&gpu_diag, N * sizeof(double)) );
    gpuErrchk( hipMemset(gpu_k, 0, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    double time = -omp_get_wtime();
    //  BUILDING THE CONDUCTIVITY MATRIX
    int num_threads = 512;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * N;

    // compute the off-diagonal elements of K
    create_K_og<<<num_blocks, num_threads>>>(
        gpu_k, gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        gpubuf.lattice, pbc, d_high_G, d_low_G,
        nn_dist, N, num_metals);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Update the diagonal of K
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );
    diagonal_sum_og<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    num_blocks = (N - 1) / num_threads + 1;
    set_diag_og<<<num_blocks, num_threads>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    time += omp_get_wtime();

    gpuErrchk(hipMemcpy(K_h, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost));


    hipFree(gpu_diag);
    hipFree(VL);
    hipFree(VR);
    hipFree(gpu_k);

    return time;
}

int count_nnz(
    double *array,
    int size
)
{
    int count = 0;
    for (int i = 0; i < size; i++) {
        if (array[i] != 0.0) {
            count++;
        }
    }
    return count;
}


int calc_nnz(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size
){
    int nnz = 0;
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                nnz++;
            }
        }
    }
    return nnz;
}

void calc_nnz_per_row(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row
){
    
    for(int i = 0; i < matrix_size; i++){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row[i] = nnz_row;
    }
}


bool assert_nnz(
    double *matrix,
    int *row_ptr,
    int *col_indices,
    int nnz,
    int matrix_size
)
{
    bool nnz_match = true;

    // match that all the elements in csr indices are no zero
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(matrix[i*matrix_size + col_indices[j]] == 0.0){
                nnz_match = false;
            }
        }
    }

    // match that element not in csr indices are zero
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){

            bool in_csr = false;
            for(int k = row_ptr[i]; k < row_ptr[i+1]; k++){
                if(col_indices[k] == j){
                    in_csr = true;
                }
            }

            if(!in_csr && matrix[i*matrix_size + j] != 0.0){
                nnz_match = false;
            }

        }
    }

    return nnz_match;
}


template<typename T>
T reduce_array(
    T *array,
    int size
)
{
    T reduction = T(0);
    for (int i = 0; i < size; i++) {
        reduction += array[i];
    }
    return reduction;
}

template<typename T>
void modified_exclusive_scan(
    T *array,
    T *excl_scan,
    int size
)
{
    // saves additional the sum of all elements
    T reduction = T(0);
    for (int i = 0; i < size+1; i++) {
        excl_scan[i] = reduction;
        reduction += array[i];
    }
}




void assemble_K_indices(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row,
    int *row_ptr,
    int *col_indices)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    // loop first over rows, then over columns
    for(int i = 0; i < matrix_size; i++){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
        
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                col_indices[row_ptr[i] + nnz_row] = j;
                nnz_row++;
            }
        }
    }
}


void calc_off_diagonal_K_cpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    int num_metals,
    double d_high_G, double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{
    #pragma omp parallel for schedule(dynamic)
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                bool metal1 = is_in_array_cpu(metals, element[i], num_metals);
                bool metal2 = is_in_array_cpu(metals, element[col_indices[j]], num_metals);
                bool ischarged1 = site_charge[i] != 0;
                bool ischarged2 = site_charge[col_indices[j]] != 0;
                bool isVacancy1 = element[i] == VACANCY;
                bool isVacancy2 = element[col_indices[j]] == VACANCY;
                bool cvacancy1 = isVacancy1 && !ischarged1;
                bool cvacancy2 = isVacancy2 && !ischarged2;
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    data[j] = -d_high_G;
                }
                else
                {
                    data[j] = -d_low_G;
                }
            }
        }
    }
}


__global__ void calc_off_diagonal_K_gpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    int num_metals,
    double d_high_G, double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{
    // parallelize over rows
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j = row_ptr[idx]; j < row_ptr[idx+1]; j++){
        if(idx != col_indices[j]){
            bool metal1 = is_in_array_gpu_og(metals, element[idx], num_metals);
            bool metal2 = is_in_array_gpu_og(metals, element[col_indices[j]], num_metals);
            bool ischarged1 = site_charge[idx] != 0;
            bool ischarged2 = site_charge[col_indices[j]] != 0;
            bool isVacancy1 = element[idx] == VACANCY;
            bool isVacancy2 = element[col_indices[j]] == VACANCY;
            bool cvacancy1 = isVacancy1 && !ischarged1;
            bool cvacancy2 = isVacancy2 && !ischarged2;
            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                data[j] = -d_high_G;
            }
            else
            {
                data[j] = -d_low_G;
            }
        }
    }
    
}



void calc_diagonal_K_cpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size
)
{
    #pragma omp parallel for schedule(dynamic)
    for(int i = 0; i < matrix_size; i++){
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += data[j];
            }
        }
        //write the diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] = -tmp;
            }
        }
    }
}

__global__ void calc_diagonal_K_gpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    //reduce the elements in the row
    double tmp = 0.0;
    for(int j = row_ptr[idx]; j < row_ptr[idx+1]; j++){
        if(idx != col_indices[j]){
            tmp += data[j];
        }
    }
    //write the diagonal element
    for(int j = row_ptr[idx]; j < row_ptr[idx+1]; j++){
        if(idx == col_indices[j]){
            data[j] = -tmp;
        }
    }
    
}


void assemble_K_cpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{

    calc_off_diagonal_K_cpu(
        metals, element, site_charge,
        num_metals,
        d_high_G, d_low_G,
        matrix_size,
        col_indices,
        row_ptr,
        data);

    calc_diagonal_K_cpu(col_indices, row_ptr, data, matrix_size);


}


void assemble_K_gpu(
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int matrix_size,
    int *col_indices_d,
    int *row_ptr_d,
    double *data_d
)
{
    int threads = 512;
    int blocks = (matrix_size + threads - 1) / threads;

    calc_off_diagonal_K_gpu<<<blocks, threads>>>(
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        matrix_size,
        col_indices_d,
        row_ptr_d,
        data_d);

    calc_diagonal_K_gpu<<<blocks, threads>>>(col_indices_d, row_ptr_d, data_d, matrix_size);
}


void test_assemble_K(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double nn_dist,
                              const int num_metals, int kmc_step_count)
{


    double *K_og;
    K_og = (double *)malloc(N * N * sizeof(double));
    double *K_sparse_cpu_assemble;
    double *K_sparse_gpu_assemble;
    K_sparse_cpu_assemble = (double *)malloc(N * N * sizeof(double));
    K_sparse_gpu_assemble = (double *)malloc(N * N * sizeof(double));

    double reltol = 1e-12;
    double abstol = 1e-12;

    // original code for dense assemble on the gpu
    assemble_K_og(handle, gpubuf, N, N_left_tot, N_right_tot, Vd, pbc, d_high_G, d_low_G, nn_dist, num_metals, kmc_step_count, K_og);

    // load the data from the gpu for the sparse assemble
    double *site_posx = (double *)malloc(N * sizeof(double));
    double *site_posy = (double *)malloc(N * sizeof(double));
    double *site_posz = (double *)malloc(N * sizeof(double));
    double *lattice = (double *)malloc(3 * sizeof(double));
    //gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
    //const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    ELEMENT *metals = (ELEMENT *)malloc(num_metals * sizeof(ELEMENT));
    ELEMENT *element = (ELEMENT *)malloc(N * sizeof(ELEMENT));
    int *site_charge = (int *)malloc(N * sizeof(int));

    gpuErrchk(hipMemcpy(site_posx, gpubuf.site_x, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(site_posy, gpubuf.site_y, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(site_posz, gpubuf.site_z, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(lattice, gpubuf.lattice, 3 * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(metals, gpubuf.metal_types, num_metals * sizeof(ELEMENT), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(element, gpubuf.site_element, N * sizeof(ELEMENT), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(site_charge, gpubuf.site_charge, N * sizeof(int), hipMemcpyDeviceToHost));

    // count the nonzero elements of the dense K
    int nnz_og = count_nnz(K_og, N * N);

    // assemble the nonzero elements of K
    int nnz = calc_nnz(site_posx, site_posy, site_posz, lattice, pbc, nn_dist, N);

    double *data_h = (double *)malloc(nnz * sizeof(double));
    int *col_indices_h = (int *)malloc(nnz * sizeof(int));
    int *row_ptr_h = (int *)malloc((N + 1) * sizeof(int));
    int nnz_per_row[N];
    // calculate the nnz per row
    calc_nnz_per_row(site_posx, site_posy, site_posz, lattice, pbc, nn_dist, N, nnz_per_row);
    // exclusive sum to get the row ptr
    modified_exclusive_scan<int>(nnz_per_row, row_ptr_h, N);

    // reduce the nnz per row to get the nnz for testing
    int nnz_reduce = reduce_array<int>(nnz_per_row, N);
    // by convention the last element of the row ptr is the nnz
    int nnz_scan = row_ptr_h[N];

    // nnz from counting
    if(nnz_og != nnz){
        std::cout << "nnz mismatch" << std::endl;
        std::cout << "nnz_og " << nnz_og << std::endl;
        std::cout << "nnz " << nnz << std::endl;
    }
    else{
        std::cout << "nnz match" << std::endl;
    }
    // nnz from reducing nnz per row
    if(nnz_og != nnz_reduce){
        std::cout << "nnz_reduce mismatch" << std::endl;
        std::cout << "nnz_og " << nnz_og << std::endl;
        std::cout << "nnz_reduce " << nnz_reduce << std::endl;
    }
    else{
        std::cout << "nnz_reduce match" << std::endl;
    }
    // nnz from exclusive scan of nnz per row
    if(nnz_og != nnz_scan){
        std::cout << "nnz_scan mismatch" << std::endl;
        std::cout << "nnz_og " << nnz_og << std::endl;
        std::cout << "nnz_scan " << nnz_scan << std::endl;
    }
    else{
        std::cout << "nnz_scan match" << std::endl;
    }

    // assemble the indices of K
    assemble_K_indices(
        site_posx, site_posy, site_posz,
        lattice, pbc,
        nn_dist,
        N,
        nnz_per_row,
        row_ptr_h,
        col_indices_h
    );

    // test if the indices are correct
    // i.e. that in the dense matrix onlz elements
    // at positions given by indices and ptr are non zero
    bool right_indices = assert_nnz(
        K_og,
        row_ptr_h,
        col_indices_h,
        nnz,
        N);
    if(!right_indices){
        std::cout << "indices mismatch" << std::endl;
    }
    else{
        std::cout << "indices match" << std::endl;
    }



    assemble_K_cpu(
        metals, element, site_charge,
        num_metals,
        d_high_G, d_low_G,
        N,
        col_indices_h,
        row_ptr_h,
        data_h
    );
    sparse_to_dense<double>(K_sparse_cpu_assemble, data_h, col_indices_h, row_ptr_h, N);

    if(!assert_array_magnitude(K_sparse_cpu_assemble, K_og, abstol, reltol, N * N)){
        std::cout << "K_sparse_cpu_assemble and K_og mismatch" << std::endl;
    }
    else{
        std::cout << "K_sparse_cpu_assemble and K_og match" << std::endl;
    }

    // assemble the data on the gpu


    double *data_d;
    int *col_indices_d;
    int *row_ptr_d;

    gpuErrchk( hipMalloc((void **)&data_d, nnz * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&col_indices_d, nnz * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&row_ptr_d, (N + 1) * sizeof(int)) );

    // copy the nonzero elements of K to the gpu
    gpuErrchk( hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(row_ptr_d, row_ptr_h, (N + 1) * sizeof(int), hipMemcpyHostToDevice) );


    assemble_K_gpu(
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        num_metals,
        d_high_G, d_low_G,
        N,
        col_indices_d,
        row_ptr_d,
        data_d
    );

    // unload sparse matrix
    gpuErrchk( hipMemcpy(data_h, data_d, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    sparse_to_dense<double>(K_sparse_gpu_assemble, data_h, col_indices_h, row_ptr_h, N);



    if(!assert_array_magnitude(K_sparse_gpu_assemble, K_og, abstol, reltol, N * N)){
        std::cout << "K_sparse_gpu_assemble and K_og mismatch" << std::endl;
    }
    else{
        std::cout << "K_sparse_gpu_assemble and K_og match" << std::endl;
    }


    // start of the benchmark
    int num_measurents = 10;

    double times_cpu[num_measurents];
    double times_gpu[num_measurents];
    double times_gpu_og[num_measurents];


    for(int i = 0; i < num_measurents; i++){
        times_gpu_og[i] = assemble_K_og(handle, gpubuf, N, N_left_tot, N_right_tot, Vd, pbc, d_high_G, d_low_G, nn_dist, num_metals, kmc_step_count, K_og);
        std::cout << "times_gpu_og " << times_gpu_og[i] << std::endl;
    }

    for(int i = 0; i < num_measurents; i++){
        times_cpu[i] = omp_get_wtime();
        assemble_K_cpu(
            metals, element, site_charge,
            num_metals,
            d_high_G, d_low_G,
            N,
            col_indices_h,
            row_ptr_h,
            data_h
        );
        times_cpu[i] = omp_get_wtime() - times_cpu[i];
        std::cout << "times_cpu " << times_cpu[i] << std::endl;
    }

    for(int i = 0; i < num_measurents; i++){
        times_gpu[i] = omp_get_wtime();
        assemble_K_gpu(
            gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
            num_metals,
            d_high_G, d_low_G,
            N,
            col_indices_d,
            row_ptr_d,
            data_d
        );
        times_gpu[i] = omp_get_wtime() - times_gpu[i];
        std::cout << "times_gpu " << times_gpu[i] << std::endl;
    }




    gpuErrchk( hipFree(data_d) );
    gpuErrchk( hipFree(col_indices_d) );
    gpuErrchk( hipFree(row_ptr_d) );

    free(K_og);
    free(K_sparse_cpu_assemble);
    free(K_sparse_gpu_assemble);
    free(site_posx);
    free(site_posy);
    free(site_posz);
    free(lattice);
    free(metals);
    free(element);
    free(site_charge);
    free(data_h);
    free(col_indices_h);
    free(row_ptr_h);
    std::cout << "K matrix assembled" << std::endl;




    std::cin.ignore();

    // TODO : calculate the RHS
    //  SOLVING FOR THE NEGATIVE INTERNAL POTENTIALS (KSUB)
    // prepare contact potentials

    // double *gpu_k_sub;
    // gpuErrchk( hipMalloc((void **)&gpu_k_sub, N_interface * sizeof(double)) ); 
    // gpuErrchk( hipMemset(gpu_k_sub, 0, N_interface * sizeof(double)) );
    // blocks_per_row = (N_left_tot - 1) / num_threads + 1;
    // num_blocks = blocks_per_row * N_interface;

    // thrust::device_ptr<double> VL_ptr = thrust::device_pointer_cast(VL);
    // thrust::fill(VL_ptr, VL_ptr + N_left_tot, -Vd/2);
    // thrust::device_ptr<double> VR_ptr = thrust::device_pointer_cast(VR);
    // thrust::fill(VR_ptr, VR_ptr + N_right_tot, Vd/2);


    // diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>
    //     (&gpu_k[N_left_tot * N], gpu_diag, VL, N, N_interface, N_left_tot);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>
    //     (&gpu_k[N_left_tot * N + N - N_right_tot], gpu_diag, VR, N, N_interface, N_right_tot);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // set_diag_K<<<blocks_per_row, num_threads>>>(gpu_k_sub, gpu_diag, N_interface);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    

    // hipFree(gpu_k_sub);


}

