#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include "gpu_buffers.h"
#include <iostream>
#include <omp.h>
#include <hipcub/hipcub.hpp>

#define NUM_THREADS 512

// returns true if thing is present in the array of things
template <typename T>
__device__ int is_in_array_gpu_og(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

template <typename T>
int is_in_array_cpu(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

__device__ double site_dist_gpu_og(double pos1x, double pos1y, double pos1z,
                                double pos2x, double pos2y, double pos2z,
                                double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

double site_dist_cpu(double pos1x, double pos1y, double pos1z,
                    double pos2x, double pos2y, double pos2z,
                    double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

__global__ void create_K_og(
    double *X,
    const double *posx, const double *posy, const double *posz,
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const double *lattice, const bool pbc, const double d_high_G, const double d_low_G,
    const double cutoff_radius, const int N, const int num_metals)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto idx = tid_total; idx < (size_t) N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        bool metal1 = is_in_array_gpu_og(metals, element[i], num_metals);
        bool metal2 = is_in_array_gpu_og(metals, element[j], num_metals);
        bool ischarged1 = site_charge[i] != 0;
        bool ischarged2 = site_charge[j] != 0;
        bool isVacancy1 = element[i] == VACANCY;
        bool isVacancy2 = element[j] == VACANCY;
        bool cvacancy1 = isVacancy1 && !ischarged1;
        bool cvacancy2 = isVacancy2 && !ischarged2;
        double dist = site_dist_gpu_og(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);

        bool neighbor = false;
        if (dist < cutoff_radius && i != j)
            neighbor = true;

        // direct terms:
        if (i != j && neighbor)
        {
            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                X[N * (i) + (j)] = -d_high_G;
            }
            else
            {
                X[N * (i) + (j)] = -d_low_G;
            }
        }
    }
}


template <int NTHREADS>
__global__ void diagonal_sum_og(double *A, double *diag, int N)
{

    int num_threads = blockDim.x;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int block_id = blockIdx.x;

    int tid = threadIdx.x;

    __shared__ double buf[NTHREADS];

    for (auto idx = block_id; idx < N * blocks_per_row; idx += gridDim.x)
    {

        int ridx = idx / blocks_per_row;
        int scol = (idx % blocks_per_row) * num_threads;
        int lcol = min(N, scol + num_threads);

        buf[tid] = 0.0;
        if (tid + scol < lcol)
        {
            buf[tid] = A[ridx * N + scol + tid];
        }

        int width = num_threads / 2;
        while (width != 0)
        {
            __syncthreads();
            if (tid < width)
            {
                buf[tid] += buf[tid + width];
            }
            width /= 2;
        }

        if (tid == 0)
        {
            atomicAdd(diag + ridx, buf[0]);
        }
    }
}

__global__ void set_diag_og(double *A, double *diag, int N)
{
    int didx = blockIdx.x * blockDim.x + threadIdx.x;
    if (didx < N)
    {
        double tmp = A[didx * N + didx];
        A[didx * N + didx] = 2 * tmp - diag[didx];
    }
}

template<typename T>
void sparse_to_dense(
    T *dense_matrix,
    T *data,
    int *col_indices,
    int *row_ptr,
    int matrix_size)
{

    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            // could not work for complex data type
            dense_matrix[i*matrix_size + j] = T(0);
        }
    }

    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            dense_matrix[i*matrix_size + col_indices[j]] = data[j];
        }
    }
}


template<typename T>
bool assert_array_magnitude(
    T *array_test,
    T *array_ref,
    double abstol,
    double reltol,
    int size)
{
    double sum_difference = 0.0;
    double sum_ref = 0.0;
    for (int i = 0; i < size; i++) {
        sum_difference += std::abs(array_test[i] - array_ref[i]) * std::abs(array_test[i] - array_ref[i]);
        sum_ref += std::abs(array_ref[i])*std::abs(array_ref[i]);

    }
    sum_difference = std::sqrt(sum_difference);
    sum_ref = std::sqrt(sum_ref);
    if (sum_difference > reltol * sum_ref + abstol) {
        std::printf("Arrays are in magnitude not the same\n");
        std::cout << "Difference " << sum_difference << std::endl;
        std::cout << "Relative " << sum_difference/sum_ref << std::endl;
        std::cout << "Mixed tolerance " << reltol * sum_ref + abstol << std::endl;
        return false;
    }

    return true;
}


double assemble_K_og(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double cutoff_radius,
                              const int num_metals, int kmc_step_count,
                              double *K_h)
{

    double *VL, *VR;
    gpuErrchk( hipMalloc((void **)&VL, N_left_tot * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&VR, N_right_tot * sizeof(double)) );

    double *gpu_k;
    double *gpu_diag;
    gpuErrchk( hipMalloc((void **)&gpu_k, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&gpu_diag, N * sizeof(double)) );
    gpuErrchk( hipMemset(gpu_k, 0, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    double time = -omp_get_wtime();
    //  BUILDING THE CONDUCTIVITY MATRIX
    int num_threads = 512;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * N;

    // compute the off-diagonal elements of K
    create_K_og<<<num_blocks, num_threads>>>(
        gpu_k, gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        gpubuf.lattice, pbc, d_high_G, d_low_G,
        cutoff_radius, N, num_metals);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Update the diagonal of K
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );
    diagonal_sum_og<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    num_blocks = (N - 1) / num_threads + 1;
    set_diag_og<<<num_blocks, num_threads>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    time += omp_get_wtime();

    gpuErrchk(hipMemcpy(K_h, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost));


    hipFree(gpu_diag);
    hipFree(VL);
    hipFree(VR);
    hipFree(gpu_k);

    return time;
}

int count_nnz(
    double *array,
    int size
)
{
    int count = 0;
    for (int i = 0; i < size; i++) {
        if (array[i] != 0.0) {
            count++;
        }
    }
    return count;
}


int calc_nnz(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size
){
    int nnz = 0;
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                nnz++;
            }
        }
    }
    return nnz;
}

void calc_nnz_per_row(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row
){
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row[i] = nnz_row;
    }
}

__global__ void calc_nnz_per_row_gpu(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row_d
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // TODO optimize this with a 2D grid instead of 1D
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row_d[i] = nnz_row;
    }

}

__global__ void calc_nnz_per_row_gpu_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int *nnz_per_row_d
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // TODO optimize this with a 2D grid instead of 1D
    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int col = 0; col < block_size_j; col++){
            int i = block_start_i + row;
            int j = block_start_j + col;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row_d[row] = nnz_row;
    }

}



bool assert_nnz(
    double *matrix,
    int *row_ptr,
    int *col_indices,
    int nnz,
    int matrix_size
)
{
    bool nnz_match = true;

    // match that all the elements in csr indices are no zero
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(matrix[i*matrix_size + col_indices[j]] == 0.0){
                nnz_match = false;
            }
        }
    }

    // match that element not in csr indices are zero
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){

            bool in_csr = false;
            for(int k = row_ptr[i]; k < row_ptr[i+1]; k++){
                if(col_indices[k] == j){
                    in_csr = true;
                }
            }

            if(!in_csr && matrix[i*matrix_size + j] != 0.0){
                nnz_match = false;
            }

        }
    }

    return nnz_match;
}


template<typename T>
T reduce_array(
    T *array,
    int size
)
{
    T reduction = T(0);
    for (int i = 0; i < size; i++) {
        reduction += array[i];
    }
    return reduction;
}

template<typename T>
void modified_exclusive_scan(
    T *array,
    T *excl_scan,
    int size
)
{
    // saves additional the sum of all elements
    T reduction = T(0);
    for (int i = 0; i < size+1; i++) {
        excl_scan[i] = reduction;
        reduction += array[i];
    }
}



void assemble_K_indices(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row,
    int *row_ptr,
    int *col_indices)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    // loop first over rows, then over columns
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
        
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                col_indices[row_ptr[i] + nnz_row] = j;
                nnz_row++;
            }
        }
    }
}

__global__ void assemble_K_indices_gpu(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row_d,
    int *row_ptr_d,
    int *col_indices_d)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //TODO can be optimized with a 2D grid instead of 1D
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
        
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                col_indices_d[row_ptr_d[i] + nnz_row] = j;
                nnz_row++;
            }
        }
    }
}


__global__ void assemble_K_indices_gpu_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int *nnz_per_row_d,
    int *row_ptr_d,
    int *col_indices_d)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //TODO can be optimized with a 2D grid instead of 1D
    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int col = 0; col < block_size_j; col++){
            int i = block_start_i + row;
            int j = block_start_j + col;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                col_indices_d[row_ptr_d[row] + nnz_row] = col;
                nnz_row++;
            }
        }
    }
}


void calc_off_diagonal_K_cpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    int num_metals,
    double d_high_G, double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                bool metal1 = is_in_array_cpu(metals, element[i], num_metals);
                bool metal2 = is_in_array_cpu(metals, element[col_indices[j]], num_metals);
                bool ischarged1 = site_charge[i] != 0;
                bool ischarged2 = site_charge[col_indices[j]] != 0;
                bool isVacancy1 = element[i] == VACANCY;
                bool isVacancy2 = element[col_indices[j]] == VACANCY;
                bool cvacancy1 = isVacancy1 && !ischarged1;
                bool cvacancy2 = isVacancy2 && !ischarged2;
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    data[j] = -d_high_G;
                }
                else
                {
                    data[j] = -d_low_G;
                }
            }
        }
    }
}


__global__ void calc_off_diagonal_K_gpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    int num_metals,
    double d_high_G, double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{
    // parallelize over rows
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                bool metal1 = is_in_array_gpu_og(metals, element[i], num_metals);
                bool metal2 = is_in_array_gpu_og(metals, element[col_indices[j]], num_metals);
                bool ischarged1 = site_charge[i] != 0;
                bool ischarged2 = site_charge[col_indices[j]] != 0;
                bool isVacancy1 = element[i] == VACANCY;
                bool isVacancy2 = element[col_indices[j]] == VACANCY;
                bool cvacancy1 = isVacancy1 && !ischarged1;
                bool cvacancy2 = isVacancy2 && !ischarged2;
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    data[j] = -d_high_G;
                }
                else
                {
                    data[j] = -d_low_G;
                }
            }
        }
    }
}



void calc_diagonal_K_cpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size
)
{
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += data[j];
            }
        }
        //write the diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] = -tmp;
            }
        }
    }
}

__global__ void calc_diagonal_K_gpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += data[j];
            }
        }
        //write the diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] = -tmp;
            }
        }
    }
}


void assemble_K_cpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{

    calc_off_diagonal_K_cpu(
        metals, element, site_charge,
        num_metals,
        d_high_G, d_low_G,
        matrix_size,
        col_indices,
        row_ptr,
        data);

    calc_diagonal_K_cpu(col_indices, row_ptr, data, matrix_size);


}


void assemble_K_gpu(
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int matrix_size,
    int *col_indices_d,
    int *row_ptr_d,
    double *data_d
)
{
    int threads = 512;
    int blocks = (matrix_size + threads - 1) / threads;

    calc_off_diagonal_K_gpu<<<blocks, threads>>>(
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        matrix_size,
        col_indices_d,
        row_ptr_d,
        data_d);

    calc_diagonal_K_gpu<<<blocks, threads>>>(col_indices_d, row_ptr_d, data_d, matrix_size);
}


void indices_creation_cpu(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    const int matrix_size,
    int **col_indices,
    int **row_ptr,
    int *nnz
)
{
    int nnz_per_row[matrix_size];
    *row_ptr = (int *)malloc((matrix_size + 1) * sizeof(int));

    // calculate the nnz per row
    calc_nnz_per_row(posx, posy, posz, lattice, pbc, cutoff_radius, matrix_size, nnz_per_row);
    
    // exclusive sum to get the row ptr
    modified_exclusive_scan<int>(nnz_per_row, (*row_ptr), matrix_size);

    // by convention the last element of the row ptr is the nnz
    nnz[0] = (*row_ptr)[matrix_size];

    *col_indices = (int *)malloc(nnz[0] * sizeof(int));
    

    // assemble the indices of K
    assemble_K_indices(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        matrix_size,
        nnz_per_row,
        (*row_ptr),
        (*col_indices)
    );
}

void indices_creation_gpu(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const int matrix_size,
    int **col_indices_d,
    int **row_ptr_d,
    int *nnz
)
{
    // parallelize over rows
    int threads = 512;
    int blocks = (matrix_size + threads - 1) / threads;

    int *nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)row_ptr_d, (matrix_size + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&nnz_per_row_d, matrix_size * sizeof(int)) );
    gpuErrchk(hipMemset((*row_ptr_d), 0, (matrix_size + 1) * sizeof(int)) );

    // calculate the nnz per row
    calc_nnz_per_row_gpu<<<blocks, threads>>>(posx_d, posy_d, posz_d, lattice_d, pbc, cutoff_radius, matrix_size, nnz_per_row_d);

    void     *temp_storage_d = NULL;
    size_t   temp_storage_bytes = 0;
    // determines temporary device storage requirements for inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, matrix_size);
    // Allocate temporary storage for inclusive prefix sum
    gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
    // Run inclusive prefix sum
    // inclusive sum starting at second value to get the row ptr
    // which is the same as inclusive sum starting at first value and last value filled with nnz
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, matrix_size);
    
    // nnz is the same as (*row_ptr_d)[matrix_size]
    gpuErrchk( hipMemcpy(nnz, (*row_ptr_d) + matrix_size, sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMalloc((void **)col_indices_d, nnz[0] * sizeof(int)) );

    // assemble the indices of K
    assemble_K_indices_gpu<<<blocks, threads>>>(
        posx_d, posy_d, posz_d,
        lattice_d, pbc,
        cutoff_radius,
        matrix_size,
        nnz_per_row_d,
        (*row_ptr_d),
        (*col_indices_d)
    );

    hipFree(temp_storage_d);
    hipFree(nnz_per_row_d);
}








void indices_creation_gpu_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int **col_indices_d,
    int **row_ptr_d,
    int *nnz
)
{
    // parallelize over rows
    int threads = 512;
    int blocks = (block_size_i + threads - 1) / threads;

    int *nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)row_ptr_d, (block_size_i + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&nnz_per_row_d, block_size_i * sizeof(int)) );
    gpuErrchk(hipMemset((*row_ptr_d), 0, (block_size_i + 1) * sizeof(int)) );

    // calculate the nnz per row
    calc_nnz_per_row_gpu_off_diagonal_block<<<blocks, threads>>>(posx_d, posy_d, posz_d, lattice_d, pbc, cutoff_radius,
        block_size_i, block_size_j, block_start_i, block_start_j, nnz_per_row_d);

    void     *temp_storage_d = NULL;
    size_t   temp_storage_bytes = 0;
    // determines temporary device storage requirements for inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, block_size_i);
    // Allocate temporary storage for inclusive prefix sum
    gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
    // Run inclusive prefix sum
    // inclusive sum starting at second value to get the row ptr
    // which is the same as inclusive sum starting at first value and last value filled with nnz
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, block_size_i);
    
    // nnz is the same as (*row_ptr_d)[block_size_i]
    gpuErrchk( hipMemcpy(nnz, (*row_ptr_d) + block_size_i, sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMalloc((void **)col_indices_d, nnz[0] * sizeof(int)) );

    // assemble the indices of K
    assemble_K_indices_gpu_off_diagonal_block<<<blocks, threads>>>(
        posx_d, posy_d, posz_d,
        lattice_d, pbc,
        cutoff_radius,
        block_size_i,
        block_size_j,
        block_start_i,
        block_start_j,
        nnz_per_row_d,
        (*row_ptr_d),
        (*col_indices_d)
    );

    hipFree(temp_storage_d);
    hipFree(nnz_per_row_d);
}

__global__ void row_reduce_K_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    double *rows_reduced_d
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        double tmp = 0.0;
        for(int col = 0; col < block_size_j; col++){
            int i = block_start_i + row;
            int j = block_start_j + col;

            bool metal1 = is_in_array_gpu_og(metals_d, element_d[i], num_metals);
            bool metal2 = is_in_array_gpu_og(metals_d, element_d[j], num_metals);
            bool ischarged1 = site_charge_d[i] != 0;
            bool ischarged2 = site_charge_d[j] != 0;
            bool isVacancy1 = element_d[i] == VACANCY;
            bool isVacancy2 = element_d[j] == VACANCY;
            bool cvacancy1 = isVacancy1 && !ischarged1;
            bool cvacancy2 = isVacancy2 && !ischarged2;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i], posx_d[j], posy_d[j], posz_d[j], lattice_d[0], lattice_d[1], lattice_d[2], pbc);

            if (dist < cutoff_radius)
            {
                // sign is switched since the diagonal is positive
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    tmp += d_high_G;
                }
                else
                {
                    tmp += d_low_G;
                }
            }            
        }
        rows_reduced_d[row] = tmp;

    }

}


__global__ void row_reduce_K_off_diagonal_block_with_precomputing(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int *col_indices_d,
    int *row_ptr_d,
    double *rows_reduced_d
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        double tmp = 0.0;
        for(int col = row_ptr_d[row]; col < row_ptr_d[row+1]; col++){
            int i = block_start_i + row;
            int j = block_start_j + col_indices_d[col];

            bool metal1 = is_in_array_gpu_og(metals_d, element_d[i], num_metals);
            bool metal2 = is_in_array_gpu_og(metals_d, element_d[j], num_metals);
            bool ischarged1 = site_charge_d[i] != 0;
            bool ischarged2 = site_charge_d[j] != 0;
            bool isVacancy1 = element_d[i] == VACANCY;
            bool isVacancy2 = element_d[j] == VACANCY;
            bool cvacancy1 = isVacancy1 && !ischarged1;
            bool cvacancy2 = isVacancy2 && !ischarged2;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i], posx_d[j], posy_d[j], posz_d[j], lattice_d[0], lattice_d[1], lattice_d[2], pbc);

            if (dist < cutoff_radius)
            {
                // sign is switched since the diagonal is positive
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    tmp += d_high_G;
                }
                else
                {
                    tmp += d_low_G;
                }
            }            
        }
        rows_reduced_d[row] = tmp;

    }

}


__global__ void add_vector_to_diagonal(
    double *data,
    int *row_ptr,
    int *col_indices,
    int matrix_size,
    double *vector
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] += vector[i];
            }
        }
    }
}

void test_assemble_A(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int K_size,
    int contact_left_size,
    int contact_right_size,
    double **A_data,
    int **A_row_ptr,
    int **A_col_indices,
    int *A_nnz,
    double **K_left_reduced,
    double **K_right_reduced
)
{


    int system_size = K_size - contact_left_size - contact_right_size;
    std::cout << "system size " << system_size << std::endl;

    gpuErrchk(hipMalloc((void **)K_left_reduced, system_size * sizeof(double)));
    gpuErrchk(hipMalloc((void **)K_right_reduced, system_size * sizeof(double)));


    // parallelize over rows
    int threads = 512;
    int blocks = (system_size + threads - 1) / threads;

    // shift site position to the device
    // reduce the matrix size to the system size
    // works since the positions are ordered

    indices_creation_gpu(
        posx + contact_left_size,
        posy + contact_left_size,
        posz + contact_left_size,
        lattice, pbc,
        cutoff_radius,
        system_size,
        A_col_indices,
        A_row_ptr,
        A_nnz
    );

    // allocate the data array
    gpuErrchk(hipMalloc((void **)A_data, A_nnz[0] * sizeof(double)));
    gpuErrchk(hipMemset((*A_data), 0, A_nnz[0] * sizeof(double)));

    // assemble only smaller part of K
    assemble_K_gpu(
        metals_d,
        element_d + contact_left_size,
        site_charge_d + contact_left_size,
        num_metals,
        d_high_G, d_low_G,
        system_size,
        *A_col_indices,
        *A_row_ptr,
        *A_data
    );


    // TODO possible faster to calculate once the off diagonal block indices
    // then do the reduction from these given indices


    // reduce the left part of K
    // block starts at i = contact_left_size (first downshifted row)
    // block starts at j = 0 (first column)
    row_reduce_K_off_diagonal_block<<<blocks, threads>>>(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        system_size,
        contact_left_size,
        contact_left_size,
        0,
        *K_left_reduced
    );

    // reduce the right part of K
    // block starts at i = contact_left_size (first downshifted row)
    // block starts at j = contact_left_size + system_size (first column)
    row_reduce_K_off_diagonal_block<<<blocks, threads>>>(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        system_size,
        contact_right_size,
        contact_left_size,
        contact_left_size + system_size,
        *K_right_reduced
    );

    // add left and right part of K to the diagonal of the data array
    add_vector_to_diagonal<<<blocks, threads>>>(
        *A_data,
        *A_row_ptr,
        *A_col_indices,
        system_size,
        *K_left_reduced
    );
    add_vector_to_diagonal<<<blocks, threads>>>(
        *A_data,
        *A_row_ptr,
        *A_col_indices,
        system_size,
        *K_right_reduced
    );
}




void test_assemble_A_with_only_precomputing_indices(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int K_size,
    int contact_left_size,
    int contact_right_size,
    double **A_data,
    int **A_row_ptr,
    int **A_col_indices,
    int *A_nnz,
    double **K_left_reduced,
    double **K_right_reduced
)
{


    int system_size = K_size - contact_left_size - contact_right_size;
    std::cout << "system size " << system_size << std::endl;

    gpuErrchk(hipMalloc((void **)K_left_reduced, system_size * sizeof(double)));
    gpuErrchk(hipMalloc((void **)K_right_reduced, system_size * sizeof(double)));


    // parallelize over rows
    int threads = 512;
    int blocks = (system_size + threads - 1) / threads;

    // shift site position to the device
    // reduce the matrix size to the system size
    // works since the positions are ordered

    indices_creation_gpu(
        posx + contact_left_size,
        posy + contact_left_size,
        posz + contact_left_size,
        lattice, pbc,
        cutoff_radius,
        system_size,
        A_col_indices,
        A_row_ptr,
        A_nnz
    );

    // allocate the data array
    gpuErrchk(hipMalloc((void **)A_data, A_nnz[0] * sizeof(double)));
    gpuErrchk(hipMemset((*A_data), 0, A_nnz[0] * sizeof(double)));

    // assemble only smaller part of K
    assemble_K_gpu(
        metals_d,
        element_d + contact_left_size,
        site_charge_d + contact_left_size,
        num_metals,
        d_high_G, d_low_G,
        system_size,
        *A_col_indices,
        *A_row_ptr,
        *A_data
    );


    // TODO possible faster to calculate once the off diagonal block indices
    // then do the reduction from these given indices
    int *contact_left_row_ptr = NULL;
    int *contact_left_col_indices = NULL;
    int contact_left_nnz;
    int *contact_right_row_ptr = NULL;
    int *contact_right_col_indices = NULL;
    int contact_right_nnz;

    indices_creation_gpu_off_diagonal_block(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        system_size,
        contact_left_size,
        contact_left_size,
        0,
        &contact_left_col_indices,
        &contact_left_row_ptr,
        &contact_left_nnz
    );
    std::cout << "contact_left_nnz " << contact_left_nnz << std::endl;

    indices_creation_gpu_off_diagonal_block(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        system_size,
        contact_right_size,
        contact_left_size,
        contact_left_size + system_size,
        &contact_right_col_indices,
        &contact_right_row_ptr,
        &contact_right_nnz
    );
    std::cout << "contact_right_nnz " << contact_right_nnz << std::endl;

    // reduce the left part of K
    // block starts at i = contact_left_size (first downshifted row)
    // block starts at j = 0 (first column)
    row_reduce_K_off_diagonal_block_with_precomputing<<<blocks, threads>>>(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        system_size,
        contact_left_size,
        contact_left_size,
        0,
        contact_left_col_indices,
        contact_left_row_ptr,
        *K_left_reduced
    );

    // reduce the right part of K
    // block starts at i = contact_left_size (first downshifted row)
    // block starts at j = contact_left_size + system_size (first column)
    row_reduce_K_off_diagonal_block_with_precomputing<<<blocks, threads>>>(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        system_size,
        contact_right_size,
        contact_left_size,
        contact_left_size + system_size,
        contact_right_col_indices,
        contact_right_row_ptr,
        *K_right_reduced
    );

    // add left and right part of K to the diagonal of the data array
    add_vector_to_diagonal<<<blocks, threads>>>(
        *A_data,
        *A_row_ptr,
        *A_col_indices,
        system_size,
        *K_left_reduced
    );
    add_vector_to_diagonal<<<blocks, threads>>>(
        *A_data,
        *A_row_ptr,
        *A_col_indices,
        system_size,
        *K_right_reduced
    );


    gpuErrchk(hipFree(contact_left_row_ptr));
    gpuErrchk(hipFree(contact_left_col_indices));
    gpuErrchk(hipFree(contact_right_row_ptr));
    gpuErrchk(hipFree(contact_right_col_indices));

}



void test_assemble_K(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double cutoff_radius,
                              const int num_metals, int kmc_step_count)
{


    double *K_og;
    K_og = (double *)malloc(N * N * sizeof(double));
    double *K_sparse_cpu_assemble;
    double *K_sparse_gpu_assemble;
    K_sparse_cpu_assemble = (double *)malloc(N * N * sizeof(double));
    K_sparse_gpu_assemble = (double *)malloc(N * N * sizeof(double));

    double reltol = 1e-12;
    double abstol = 1e-12;

    // original code for dense assemble on the gpu
    assemble_K_og(handle, gpubuf, N, N_left_tot, N_right_tot, Vd, pbc, d_high_G, d_low_G, cutoff_radius, num_metals, kmc_step_count, K_og);

    // load the data from the gpu for the sparse assemble
    double *posx = (double *)malloc(N * sizeof(double));
    double *posy = (double *)malloc(N * sizeof(double));
    double *posz = (double *)malloc(N * sizeof(double));
    double *lattice = (double *)malloc(3 * sizeof(double));
    //gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
    //const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    ELEMENT *metals = (ELEMENT *)malloc(num_metals * sizeof(ELEMENT));
    ELEMENT *element = (ELEMENT *)malloc(N * sizeof(ELEMENT));
    int *site_charge = (int *)malloc(N * sizeof(int));

    gpuErrchk(hipMemcpy(posx, gpubuf.site_x, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(posy, gpubuf.site_y, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(posz, gpubuf.site_z, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(lattice, gpubuf.lattice, 3 * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(metals, gpubuf.metal_types, num_metals * sizeof(ELEMENT), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(element, gpubuf.site_element, N * sizeof(ELEMENT), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(site_charge, gpubuf.site_charge, N * sizeof(int), hipMemcpyDeviceToHost));

    // count the nonzero elements of the dense K
    int nnz_og = count_nnz(K_og, N * N);

    // assemble the nonzero elements of K
    int nnz = calc_nnz(posx, posy, posz, lattice, pbc, cutoff_radius, N);

    double *data_h = (double *)malloc(nnz * sizeof(double));

    int nnz_cpu;
    int *row_ptr_h = NULL;
    int *col_indices_h = NULL;

    indices_creation_cpu(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        N,
        &col_indices_h,
        &row_ptr_h,
        &nnz_cpu
    );


    // nnz from counting
    if(nnz_og != nnz){
        std::cout << "nnz mismatch" << std::endl;
        std::cout << "nnz_og " << nnz_og << std::endl;
        std::cout << "nnz " << nnz << std::endl;
    }
    else{
        std::cout << "nnz match" << std::endl;
    }
    // nnz from reducing nnz per row
    if(nnz_og != nnz_cpu){
        std::cout << "nnz_cpu mismatch" << std::endl;
        std::cout << "nnz_og " << nnz_og << std::endl;
        std::cout << "nnz_cpu " << nnz_cpu << std::endl;
    }
    else{
        std::cout << "nnz_cpu match" << std::endl;
    }


    // test if the indices are correct
    // i.e. that in the dense matrix onlz elements
    // at positions given by indices and ptr are non zero
    bool right_indices = assert_nnz(
        K_og,
        row_ptr_h,
        col_indices_h,
        nnz,
        N);
    if(!right_indices){
        std::cout << "indices mismatch" << std::endl;
    }
    else{
        std::cout << "indices match" << std::endl;
    }


    assemble_K_cpu(
        metals, element, site_charge,
        num_metals,
        d_high_G, d_low_G,
        N,
        col_indices_h,
        row_ptr_h,
        data_h
    );
    sparse_to_dense<double>(K_sparse_cpu_assemble, data_h, col_indices_h, row_ptr_h, N);

    if(!assert_array_magnitude(K_sparse_cpu_assemble, K_og, abstol, reltol, N * N)){
        std::cout << "K_sparse_cpu_assemble and K_og mismatch" << std::endl;
    }
    else{
        std::cout << "K_sparse_cpu_assemble and K_og match" << std::endl;
    }

    // assemble the indices on the gpu

    int *col_indices_gpu_assemble_d;
    int *row_ptr_gpu_assemble_d;
    int nnz_gpu_assemble;

    indices_creation_gpu(
        gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.lattice, pbc,
        cutoff_radius,
        N,
        &col_indices_gpu_assemble_d,
        &row_ptr_gpu_assemble_d,
        &nnz_gpu_assemble
    );


    if(nnz_gpu_assemble != nnz){
        std::cout << "nnz_gpu_assemble mismatch" << std::endl;
        std::cout << "nnz_gpu_assemble " << nnz_gpu_assemble << std::endl;
        std::cout << "nnz " << nnz << std::endl;
    }
    else{
        std::cout << "nnz_gpu_assemble match" << std::endl;
    }



    // unload the indices and test if they are the same
    int col_indices_gpu_assemble_h[nnz];
    int row_ptr_gpu_assemble_h[N + 1];
    gpuErrchk( hipMemcpy(col_indices_gpu_assemble_h, col_indices_gpu_assemble_d, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(row_ptr_gpu_assemble_h, row_ptr_gpu_assemble_d, (N + 1) * sizeof(int), hipMemcpyDeviceToHost) );


    if(!assert_array_magnitude(col_indices_gpu_assemble_h, col_indices_h, abstol, reltol, nnz)){
        std::cout << "col_indices_gpu_assemble_h and col_indices_h mismatch" << std::endl;
    }
    else{
        std::cout << "col_indices_gpu_assemble_h and col_indices_h match" << std::endl;
    }
    if(!assert_array_magnitude(row_ptr_gpu_assemble_h, row_ptr_h, abstol, reltol, N + 1)){
        std::cout << "row_ptr_gpu_assemble_h and row_ptr_h mismatch" << std::endl;
    }
    else{
        std::cout << "row_ptr_gpu_assemble_h and row_ptr_h match" << std::endl;
    }


    double *data_d;
    int *col_indices_d;
    int *row_ptr_d;

    gpuErrchk( hipMalloc((void **)&data_d, nnz * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&col_indices_d, nnz * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&row_ptr_d, (N + 1) * sizeof(int)) );


    // copy the nonzero elements of K to the gpu
    gpuErrchk( hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(row_ptr_d, row_ptr_h, (N + 1) * sizeof(int), hipMemcpyHostToDevice) );


    assemble_K_gpu(
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        num_metals,
        d_high_G, d_low_G,
        N,
        col_indices_d,
        row_ptr_d,
        data_d
    );

    // unload sparse matrix
    gpuErrchk( hipMemcpy(data_h, data_d, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    sparse_to_dense<double>(K_sparse_gpu_assemble, data_h, col_indices_h, row_ptr_h, N);



    if(!assert_array_magnitude(K_sparse_gpu_assemble, K_og, abstol, reltol, N * N)){
        std::cout << "K_sparse_gpu_assemble and K_og mismatch" << std::endl;
    }
    else{
        std::cout << "K_sparse_gpu_assemble and K_og match" << std::endl;
    }

    int count_left_nnzs = 0;
    for(int i = 0; i < N; i++){
        for(int j = row_ptr_h[i]; j < row_ptr_h[i+1]; j++){
            if(i >= N_left_tot && i < N - N_right_tot && col_indices_h[j] < N_left_tot){
                count_left_nnzs += 1;
            }
        }
    }
    std::cout << "count_left_nnzs " << count_left_nnzs << std::endl;
    int count_right_nnzs = 0;
    for(int i = 0; i < N; i++){
        for(int j = row_ptr_h[i]; j < row_ptr_h[i+1]; j++){
            if(i >= N_left_tot && i < N - N_right_tot && col_indices_h[j] >= N - N_right_tot){
                count_right_nnzs += 1;
            }
        }
    }
    std::cout << "count_right_nnzs " << count_right_nnzs << std::endl;



    double *A_data_d = NULL;
    int *A_row_ptr_d = NULL;
    int *A_col_indices_d = NULL;
    int A_nnz;
    double *K_left_reduced_d = NULL;
    double *K_right_reduced_d = NULL;


    test_assemble_A_with_only_precomputing_indices(
        gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.lattice, pbc,
        cutoff_radius,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        num_metals,
        d_high_G, d_low_G,
        N,
        N_left_tot,
        N_right_tot,
        &A_data_d,
        &A_row_ptr_d,
        &A_col_indices_d,
        &A_nnz,
        &K_left_reduced_d,
        &K_right_reduced_d
    );

    std::cout << "A_nnz " << A_nnz << std::endl;
    std::cout << "nnz " << nnz << std::endl;
    double A_data_h[A_nnz];
    int A_row_ptr_h[N - N_left_tot - N_right_tot + 1];
    int A_col_indices_h[A_nnz];


    gpuErrchk( hipMemcpy(A_data_h, A_data_d, A_nnz * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(A_row_ptr_h, A_row_ptr_d, (N - N_left_tot - N_right_tot + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(A_col_indices_h, A_col_indices_d, A_nnz * sizeof(int), hipMemcpyDeviceToHost) );


    // sanity check if the indices are correct
    bool inside_nnz = true;
    for(int i = 0; i < N - N_left_tot - N_right_tot; i++){
        for(int j = A_row_ptr_h[i]; j < A_row_ptr_h[i+1]; j++){
            int row = i + N_left_tot;
            int col = A_col_indices_h[j] + N_left_tot;
            if(K_og[row * N + col] == 0.0){
                inside_nnz = false;
            }
    
        }
    }
    if(!inside_nnz){
        std::cout << "A_indices and K_og mismatch for nnz" << std::endl;
    }
    else{
        std::cout << "A_indices and K_og match for nnz" << std::endl;
    }

    bool outside_nnz = true;
    for(int i = 0; i < N - N_left_tot - N_right_tot; i++){
        for(int j = 0; j < N - N_left_tot - N_right_tot; j++){
            bool inside = false;
            for(int k = A_row_ptr_h[i]; k < A_row_ptr_h[i+1]; k++){
                if(A_col_indices_h[k] == j){
                    inside = true;
                }
            }
            int row = i + N_left_tot;
            int col = j + N_left_tot;            
            if(!inside){
                if(K_og[row * N + col] != 0.0){
                    outside_nnz = false;
                }
            }
            else{
                if(K_og[row * N + col] == 0.0){
                    outside_nnz = false;
                }
            }
        }
    }
    if(!outside_nnz){
        std::cout << "A_indices and K_og mismatch for zeros" << std::endl;
    }
    else{
        std::cout << "A_indices and K_og match for zeros" << std::endl;
    }




    double difference = 0.0;
    double sum_ref = 0.0;
    for(int i = 0; i < N - N_left_tot - N_right_tot; i++){
        for(int j = A_row_ptr_h[i]; j < A_row_ptr_h[i+1]; j++){
            int row = i + N_left_tot;
            int col = A_col_indices_h[j] + N_left_tot;

            difference += std::abs(A_data_h[j] - K_og[row * N + col]) * std::abs(A_data_h[j] - K_og[row * N + col]);
            sum_ref += std::abs(K_og[row * N + col]) * std::abs(K_og[row * N + col]);
        

        }
    }
    if(difference > abstol + reltol * sum_ref){
        std::cout << "A_data_h and K_og mismatch" << std::endl;

    }
    else{
        std::cout << "A_data_h and K_og match" << std::endl;
    }



    gpuErrchk(hipFree(A_data_d));
    gpuErrchk(hipFree(A_col_indices_d));
    gpuErrchk(hipFree(A_row_ptr_d));
    gpuErrchk(hipFree(K_left_reduced_d));
    gpuErrchk(hipFree(K_right_reduced_d));



    // // start of the benchmark
    // int num_measurents = 10;

    // double times_data_assemble_cpu[num_measurents];
    // double times_data_assemble_gpu[num_measurents];
    // double times_full_gpu_og[num_measurents];

    // double times_indices_assemble_gpu[num_measurents];
    // double times_indices_assemble_cpu[num_measurents];


    // for(int i = 0; i < num_measurents; i++){
    //     times_full_gpu_og[i] = assemble_K_og(handle, gpubuf, N, N_left_tot, N_right_tot, Vd, pbc, d_high_G, d_low_G, cutoff_radius, num_metals, kmc_step_count, K_og);
    //     std::cout << "times_full_gpu_og " << times_full_gpu_og[i] << std::endl;
    // }

    // for(int i = 0; i < num_measurents; i++){
    //     times_data_assemble_cpu[i] = omp_get_wtime();
    //     assemble_K_cpu(
    //         metals, element, site_charge,
    //         num_metals,
    //         d_high_G, d_low_G,
    //         N,
    //         col_indices_h,
    //         row_ptr_h,
    //         data_h
    //     );
    //     times_data_assemble_cpu[i] = omp_get_wtime() - times_data_assemble_cpu[i];
    //     std::cout << "times_data_assemble_cpu " << times_data_assemble_cpu[i] << std::endl;
    // }

    // for(int i = 0; i < num_measurents; i++){
    //     times_data_assemble_gpu[i] = omp_get_wtime();
    //     gpuErrchk(hipDeviceSynchronize());
    //     assemble_K_gpu(
    //         gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
    //         num_metals,
    //         d_high_G, d_low_G,
    //         N,
    //         col_indices_d,
    //         row_ptr_d,
    //         data_d
    //     );
    //     gpuErrchk(hipDeviceSynchronize());
    //     times_data_assemble_gpu[i] = omp_get_wtime() - times_data_assemble_gpu[i];
    //     std::cout << "times_data_assemble_gpu " << times_data_assemble_gpu[i] << std::endl;
    // }

    gpuErrchk( hipFree(col_indices_gpu_assemble_d) );
    gpuErrchk( hipFree(row_ptr_gpu_assemble_d) );
    free(col_indices_h);
    free(row_ptr_h);


    // for(int i = 0; i < num_measurents; i++){
    //     times_indices_assemble_cpu[i] = omp_get_wtime();
    //     indices_creation_cpu(
    //         posx, posy, posz,
    //         lattice, pbc,
    //         cutoff_radius,
    //         N,
    //         &col_indices_h,
    //         &row_ptr_h,
    //         &nnz_cpu
    //     );
    //     times_indices_assemble_cpu[i] = omp_get_wtime() - times_indices_assemble_cpu[i];
    //     free(col_indices_h);
    //     free(row_ptr_h);
    //     std::cout << "times_indices_assemble_cpu " << times_indices_assemble_cpu[i] << std::endl;
    // }
    // for(int i = 0; i < num_measurents; i++){
    //     times_indices_assemble_gpu[i] = omp_get_wtime();
    //     gpuErrchk(hipDeviceSynchronize());
    //     indices_creation_gpu(
    //         gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
    //         gpubuf.lattice, pbc,
    //         cutoff_radius,
    //         N,
    //         &col_indices_gpu_assemble_d,
    //         &row_ptr_gpu_assemble_d,
    //         &nnz_gpu_assemble
    //     );
    //     gpuErrchk(hipDeviceSynchronize());
    //     times_indices_assemble_gpu[i] = omp_get_wtime() - times_indices_assemble_gpu[i];
    //     hipFree(col_indices_gpu_assemble_d);
    //     hipFree(row_ptr_gpu_assemble_d);
    //     std::cout << "times_indices_assemble_gpu " << times_indices_assemble_gpu[i] << std::endl;
    // }


    gpuErrchk( hipFree(data_d) );
    gpuErrchk( hipFree(col_indices_d) );
    gpuErrchk( hipFree(row_ptr_d) );

    

    free(K_og);
    free(K_sparse_cpu_assemble);
    free(K_sparse_gpu_assemble);
    free(posx);
    free(posy);
    free(posz);
    free(lattice);
    free(metals);
    free(element);
    free(site_charge);
    free(data_h);

    std::cout << "K matrix assembled" << std::endl;




    std::cin.ignore();

    // TODO : calculate the RHS
    //  SOLVING FOR THE NEGATIVE INTERNAL POTENTIALS (KSUB)
    // prepare contact potentials

    // double *gpu_k_sub;
    // gpuErrchk( hipMalloc((void **)&gpu_k_sub, N_interface * sizeof(double)) ); 
    // gpuErrchk( hipMemset(gpu_k_sub, 0, N_interface * sizeof(double)) );
    // blocks_per_row = (N_left_tot - 1) / num_threads + 1;
    // num_blocks = blocks_per_row * N_interface;

    // thrust::device_ptr<double> VL_ptr = thrust::device_pointer_cast(VL);
    // thrust::fill(VL_ptr, VL_ptr + N_left_tot, -Vd/2);
    // thrust::device_ptr<double> VR_ptr = thrust::device_pointer_cast(VR);
    // thrust::fill(VR_ptr, VR_ptr + N_right_tot, Vd/2);


    // diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>
    //     (&gpu_k[N_left_tot * N], gpu_diag, VL, N, N_interface, N_left_tot);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>
    //     (&gpu_k[N_left_tot * N + N - N_right_tot], gpu_diag, VR, N, N_interface, N_right_tot);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // set_diag_K<<<blocks_per_row, num_threads>>>(gpu_k_sub, gpu_diag, N_interface);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    

    // hipFree(gpu_k_sub);


}

