#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include "gpu_buffers.h"
#include <iostream>
#include <omp.h>
#include <hipcub/hipcub.hpp>
#include <cstdlib>

#define NUM_THREADS 512

// returns true if thing is present in the array of things
template <typename T>
__device__ int is_in_array_gpu_og(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

template <typename T>
int is_in_array_cpu(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

__device__ double site_dist_gpu_og(double pos1x, double pos1y, double pos1z,
                                double pos2x, double pos2y, double pos2z,
                                double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

double site_dist_cpu(double pos1x, double pos1y, double pos1z,
                    double pos2x, double pos2y, double pos2z,
                    double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

__global__ void create_K_og(
    double *X,
    const double *posx, const double *posy, const double *posz,
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const double *lattice, const bool pbc, const double d_high_G, const double d_low_G,
    const double cutoff_radius, const int N, const int num_metals)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto idx = tid_total; idx < (size_t) N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        bool metal1 = is_in_array_gpu_og(metals, element[i], num_metals);
        bool metal2 = is_in_array_gpu_og(metals, element[j], num_metals);
        bool ischarged1 = site_charge[i] != 0;
        bool ischarged2 = site_charge[j] != 0;
        bool isVacancy1 = element[i] == VACANCY;
        bool isVacancy2 = element[j] == VACANCY;
        bool cvacancy1 = isVacancy1 && !ischarged1;
        bool cvacancy2 = isVacancy2 && !ischarged2;
        double dist = site_dist_gpu_og(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);

        bool neighbor = false;
        if (dist < cutoff_radius && i != j)
            neighbor = true;

        // direct terms:
        if (i != j && neighbor)
        {
            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                X[N * (i) + (j)] = -d_high_G;
            }
            else
            {
                X[N * (i) + (j)] = -d_low_G;
            }
        }
    }
}


template <int NTHREADS>
__global__ void diagonal_sum_og(double *A, double *diag, int N)
{

    int num_threads = blockDim.x;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int block_id = blockIdx.x;

    int tid = threadIdx.x;

    __shared__ double buf[NTHREADS];

    for (auto idx = block_id; idx < N * blocks_per_row; idx += gridDim.x)
    {

        int ridx = idx / blocks_per_row;
        int scol = (idx % blocks_per_row) * num_threads;
        int lcol = min(N, scol + num_threads);

        buf[tid] = 0.0;
        if (tid + scol < lcol)
        {
            buf[tid] = A[ridx * N + scol + tid];
        }

        int width = num_threads / 2;
        while (width != 0)
        {
            __syncthreads();
            if (tid < width)
            {
                buf[tid] += buf[tid + width];
            }
            width /= 2;
        }

        if (tid == 0)
        {
            atomicAdd(diag + ridx, buf[0]);
        }
    }
}

__global__ void set_diag_og(double *A, double *diag, int N)
{
    int didx = blockIdx.x * blockDim.x + threadIdx.x;
    if (didx < N)
    {
        double tmp = A[didx * N + didx];
        A[didx * N + didx] = 2 * tmp - diag[didx];
    }
}

template<typename T>
void sparse_to_dense(
    T *dense_matrix,
    T *data,
    int *col_indices,
    int *row_ptr,
    int matrix_size)
{

    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            // could not work for complex data type
            dense_matrix[i*matrix_size + j] = T(0);
        }
    }

    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            dense_matrix[i*matrix_size + col_indices[j]] = data[j];
        }
    }
}


template<typename T>
bool assert_array_magnitude(
    T *array_test,
    T *array_ref,
    double abstol,
    double reltol,
    int size)
{
    double sum_difference = 0.0;
    double sum_ref = 0.0;
    for (int i = 0; i < size; i++) {
        sum_difference += std::abs(array_test[i] - array_ref[i]) * std::abs(array_test[i] - array_ref[i]);
        sum_ref += std::abs(array_ref[i])*std::abs(array_ref[i]);

    }
    sum_difference = std::sqrt(sum_difference);
    sum_ref = std::sqrt(sum_ref);
    if (sum_difference > reltol * sum_ref + abstol) {
        std::printf("Arrays are in magnitude not the same\n");
        std::cout << "Difference " << sum_difference << std::endl;
        std::cout << "Relative " << sum_difference/sum_ref << std::endl;
        std::cout << "Mixed tolerance " << reltol * sum_ref + abstol << std::endl;
        return false;
    }

    return true;
}


double assemble_K_og(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double cutoff_radius,
                              const int num_metals, int kmc_step_count,
                              double *K_h)
{

    double *VL, *VR;
    gpuErrchk( hipMalloc((void **)&VL, N_left_tot * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&VR, N_right_tot * sizeof(double)) );

    double *gpu_k;
    double *gpu_diag;
    gpuErrchk( hipMalloc((void **)&gpu_k, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&gpu_diag, N * sizeof(double)) );
    gpuErrchk( hipMemset(gpu_k, 0, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    double time = -omp_get_wtime();
    //  BUILDING THE CONDUCTIVITY MATRIX
    int num_threads = 512;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * N;

    // compute the off-diagonal elements of K
    create_K_og<<<num_blocks, num_threads>>>(
        gpu_k, gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        gpubuf.lattice, pbc, d_high_G, d_low_G,
        cutoff_radius, N, num_metals);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Update the diagonal of K
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );
    diagonal_sum_og<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    num_blocks = (N - 1) / num_threads + 1;
    set_diag_og<<<num_blocks, num_threads>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    time += omp_get_wtime();

    gpuErrchk(hipMemcpy(K_h, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost));


    hipFree(gpu_diag);
    hipFree(VL);
    hipFree(VR);
    hipFree(gpu_k);

    return time;
}

int count_nnz(
    double *array,
    int size
)
{
    int count = 0;
    for (int i = 0; i < size; i++) {
        if (array[i] != 0.0) {
            count++;
        }
    }
    return count;
}


int calc_nnz(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size
){
    int nnz = 0;
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                nnz++;
            }
        }
    }
    return nnz;
}

void calc_nnz_per_row(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row
){
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row[i] = nnz_row;
    }
}

__global__ void calc_nnz_per_row_gpu(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row_d
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // TODO optimize this with a 2D grid instead of 1D
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row_d[i] = nnz_row;
    }

}


__global__ void calc_nnz_per_row_gpu_v2(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row_d
){

    // assume 1D grid, but now a block pre load the positions into shared memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // shared memory for position
    // shared memory needs 6 block size 
    extern __shared__ double buffer[];

    double *col_posx = &buffer[0];
    double *col_posy = &buffer[blockDim.x];
    double *col_posz = &buffer[2*blockDim.x];

    // load position into shared memory


    // now assumed enough threads for the full vector
    // problem for larger devices TODO
    for(int i = 0; i < matrix_size; i += blockDim.x * gridDim.x){
    
        int nnz_row = 0;

        double row_posx = 0.0;
        double row_posy = 0.0;
        double row_posz = 0.0;

        int iidx = i + idx;

        if(
            iidx < matrix_size
        ){
            row_posx = posx_d[iidx];
            row_posy = posy_d[iidx];
            row_posz = posz_d[iidx];
        }
    


        for(int k = 0; k < matrix_size; k += blockDim.x){
            // synchronize to not overwrite the shared memory
            // before every thread is finished
            __syncthreads();

            // stuff is loaded twice if blockIdx.x == k % gridDim.x
            // TODO optimize
            // problem we need all threads to load even for the last block row
            if(k + threadIdx.x < matrix_size){
                col_posx[threadIdx.x] = posx_d[k + threadIdx.x];
                col_posy[threadIdx.x] = posy_d[k + threadIdx.x];
                col_posz[threadIdx.x] = posz_d[k + threadIdx.x];
            }

            __syncthreads();

            int end = min(matrix_size - k, blockDim.x);
            // threads with iidx >= matrix_size still do this, but it is not used
            // better than branching
            for(int j = 0; j < end; j++){
                double dist = site_dist_gpu_og(row_posx, row_posy, row_posz,
                                            col_posx[j], col_posy[j], col_posz[j],
                                            lattice_d[0], lattice_d[1], lattice_d[2], pbc);
                if(dist < cutoff_radius){
                    nnz_row++;
                }
            }
        }

        if(
            iidx < matrix_size
        ){
            nnz_per_row_d[iidx] = nnz_row;
        }
    }

}




__global__ void calc_nnz_per_row_gpu_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int *nnz_per_row_d
){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // TODO optimize this with a 2D grid instead of 1D
    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int col = 0; col < block_size_j; col++){
            int i = block_start_i + row;
            int j = block_start_j + col;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                nnz_row++;
            }
        }
        nnz_per_row_d[row] = nnz_row;
    }

}



bool assert_nnz(
    double *matrix,
    int *row_ptr,
    int *col_indices,
    int nnz,
    int matrix_size
)
{
    bool nnz_match = true;

    // match that all the elements in csr indices are no zero
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(matrix[i*matrix_size + col_indices[j]] == 0.0){
                nnz_match = false;
            }
        }
    }

    // match that element not in csr indices are zero
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){

            bool in_csr = false;
            for(int k = row_ptr[i]; k < row_ptr[i+1]; k++){
                if(col_indices[k] == j){
                    in_csr = true;
                }
            }

            if(!in_csr && matrix[i*matrix_size + j] != 0.0){
                nnz_match = false;
            }

        }
    }

    return nnz_match;
}


template<typename T>
T reduce_array(
    T *array,
    int size
)
{
    T reduction = T(0);
    for (int i = 0; i < size; i++) {
        reduction += array[i];
    }
    return reduction;
}

template<typename T>
void modified_exclusive_scan(
    T *array,
    T *excl_scan,
    int size
)
{
    // saves additional the sum of all elements
    T reduction = T(0);
    for (int i = 0; i < size+1; i++) {
        excl_scan[i] = reduction;
        reduction += array[i];
    }
}



void assemble_K_indices(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row,
    int *row_ptr,
    int *col_indices)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    // loop first over rows, then over columns
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
        
            double dist = site_dist_cpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
            if(dist < cutoff_radius){
                col_indices[row_ptr[i] + nnz_row] = j;
                nnz_row++;
            }
        }
    }
}

__global__ void assemble_K_indices_gpu(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row_d,
    int *row_ptr_d,
    int *col_indices_d)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //TODO can be optimized with a 2D grid instead of 1D
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int j = 0; j < matrix_size; j++){
        
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                col_indices_d[row_ptr_d[i] + nnz_row] = j;
                nnz_row++;
            }
        }
    }
}


__global__ void assemble_K_indices_gpu_v2(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int matrix_size,
    int *nnz_per_row_d,
    int *row_ptr_d,
    int *col_indices_d)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // shared memory for position
    // shared memory needs 6 block size 
    extern __shared__ double buffer[];

    double *col_posx = &buffer[0];
    double *col_posy = &buffer[blockDim.x];
    double *col_posz = &buffer[2*blockDim.x];

    //TODO can be optimized with a 2D grid instead of 1D
    // now assumed enough threads for the full vector
    // problem for larger devices TODO
    for(int i = 0; i < matrix_size; i += blockDim.x * gridDim.x){
    
        int nnz_row = 0;

        double row_posx = 0.0;
        double row_posy = 0.0;
        double row_posz = 0.0;

        int iidx = i + idx;

        if(
            iidx < matrix_size
        ){
            row_posx = posx_d[iidx];
            row_posy = posy_d[iidx];
            row_posz = posz_d[iidx];
        }

        for(int k = 0; k < matrix_size; k += blockDim.x){
            __syncthreads();

            // stuff is loaded twice if blockIdx.x == k % gridDim.x
            // TODO optimize
            // problem we need all threads to load even for the last block row
            if(k + threadIdx.x < matrix_size){
                col_posx[threadIdx.x] = posx_d[k + threadIdx.x];
                col_posy[threadIdx.x] = posy_d[k + threadIdx.x];
                col_posz[threadIdx.x] = posz_d[k + threadIdx.x];
            }

            __syncthreads();

            int end = min(matrix_size - k, blockDim.x);

            for(int j = 0; j < end; j++){
                double dist = site_dist_gpu_og(row_posx, row_posy, row_posz,
                                            col_posx[j], col_posy[j], col_posz[j],
                                            lattice_d[0], lattice_d[1], lattice_d[2], pbc);
                if(iidx < matrix_size && dist < cutoff_radius){
                    col_indices_d[row_ptr_d[iidx] + nnz_row] = k+j;
                    nnz_row++;
                }
            }
        }

    }
}


__global__ void assemble_K_indices_gpu_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int *nnz_per_row_d,
    int *row_ptr_d,
    int *col_indices_d)
{
    // row ptr is already calculated
    // exclusive scam of nnz_per_row

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //TODO can be optimized with a 2D grid instead of 1D
    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        int nnz_row = 0;
        for(int col = 0; col < block_size_j; col++){
            int i = block_start_i + row;
            int j = block_start_j + col;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice_d[0], lattice_d[1], lattice_d[2], pbc);
            if(dist < cutoff_radius){
                col_indices_d[row_ptr_d[row] + nnz_row] = col;
                nnz_row++;
            }
        }
    }
}


void calc_off_diagonal_K_cpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    int num_metals,
    double d_high_G, double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                bool metal1 = is_in_array_cpu(metals, element[i], num_metals);
                bool metal2 = is_in_array_cpu(metals, element[col_indices[j]], num_metals);
                bool ischarged1 = site_charge[i] != 0;
                bool ischarged2 = site_charge[col_indices[j]] != 0;
                bool isVacancy1 = element[i] == VACANCY;
                bool isVacancy2 = element[col_indices[j]] == VACANCY;
                bool cvacancy1 = isVacancy1 && !ischarged1;
                bool cvacancy2 = isVacancy2 && !ischarged2;
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    data[j] = -d_high_G;
                }
                else
                {
                    data[j] = -d_low_G;
                }
            }
        }
    }
}


__global__ void calc_off_diagonal_K_gpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    int num_metals,
    double d_high_G, double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{
    // parallelize over rows
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                bool metal1 = is_in_array_gpu_og(metals, element[i], num_metals);
                bool metal2 = is_in_array_gpu_og(metals, element[col_indices[j]], num_metals);
                bool ischarged1 = site_charge[i] != 0;
                bool ischarged2 = site_charge[col_indices[j]] != 0;
                bool isVacancy1 = element[i] == VACANCY;
                bool isVacancy2 = element[col_indices[j]] == VACANCY;
                bool cvacancy1 = isVacancy1 && !ischarged1;
                bool cvacancy2 = isVacancy2 && !ischarged2;
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    data[j] = -d_high_G;
                }
                else
                {
                    data[j] = -d_low_G;
                }
            }
        }
    }
}



void calc_diagonal_K_cpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size
)
{
    #pragma omp parallel for
    for(int i = 0; i < matrix_size; i++){
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += data[j];
            }
        }
        //write the diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] = -tmp;
            }
        }
    }
}

__global__ void calc_diagonal_K_gpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += data[j];
            }
        }
        //write the diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] = -tmp;
            }
        }
    }
}

__global__ void reduce_K_gpu(
    int *col_indices,
    int *row_ptr,
    double *data,
    int matrix_size,
    double *rows_reduced
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += -data[j];
            }
        }
        rows_reduced[i] = tmp;
    }

}


void assemble_K_cpu(
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int matrix_size,
    int *col_indices,
    int *row_ptr,
    double *data
)
{

    calc_off_diagonal_K_cpu(
        metals, element, site_charge,
        num_metals,
        d_high_G, d_low_G,
        matrix_size,
        col_indices,
        row_ptr,
        data);

    calc_diagonal_K_cpu(col_indices, row_ptr, data, matrix_size);


}


void assemble_K_gpu(
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int matrix_size,
    int *col_indices_d,
    int *row_ptr_d,
    double *data_d
)
{
    int threads = 256;
    int blocks = (matrix_size + threads - 1) / threads;

    calc_off_diagonal_K_gpu<<<blocks, threads>>>(
        metals_d, element_d, site_charge_d,
        num_metals,
        d_high_G, d_low_G,
        matrix_size,
        col_indices_d,
        row_ptr_d,
        data_d);

    calc_diagonal_K_gpu<<<blocks, threads>>>(col_indices_d, row_ptr_d, data_d, matrix_size);
}


void indices_creation_cpu(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    const int matrix_size,
    int **col_indices,
    int **row_ptr,
    int *nnz
)
{
    int nnz_per_row[matrix_size];
    *row_ptr = (int *)malloc((matrix_size + 1) * sizeof(int));

    // calculate the nnz per row
    calc_nnz_per_row(posx, posy, posz, lattice, pbc, cutoff_radius, matrix_size, nnz_per_row);
    
    // exclusive sum to get the row ptr
    modified_exclusive_scan<int>(nnz_per_row, (*row_ptr), matrix_size);

    // by convention the last element of the row ptr is the nnz
    nnz[0] = (*row_ptr)[matrix_size];

    *col_indices = (int *)malloc(nnz[0] * sizeof(int));
    

    // assemble the indices of K
    assemble_K_indices(
        posx, posy, posz,
        lattice, pbc,
        cutoff_radius,
        matrix_size,
        nnz_per_row,
        (*row_ptr),
        (*col_indices)
    );
}

void indices_creation_gpu(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const int matrix_size,
    int **col_indices_d,
    int **row_ptr_d,
    int *nnz
)
{
    // parallelize over rows
    int threads = 256;
    int blocks = (matrix_size + threads - 1) / threads;

    int *nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)row_ptr_d, (matrix_size + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&nnz_per_row_d, matrix_size * sizeof(int)) );
    gpuErrchk(hipMemset((*row_ptr_d), 0, (matrix_size + 1) * sizeof(int)) );

    // calculate the nnz per row
    calc_nnz_per_row_gpu<<<blocks, threads>>>(posx_d, posy_d, posz_d, lattice_d, pbc, cutoff_radius, matrix_size, nnz_per_row_d);

    void     *temp_storage_d = NULL;
    size_t   temp_storage_bytes = 0;
    // determines temporary device storage requirements for inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, matrix_size);
    // Allocate temporary storage for inclusive prefix sum
    gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
    // Run inclusive prefix sum
    // inclusive sum starting at second value to get the row ptr
    // which is the same as inclusive sum starting at first value and last value filled with nnz
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, matrix_size);
    
    // nnz is the same as (*row_ptr_d)[matrix_size]
    gpuErrchk( hipMemcpy(nnz, (*row_ptr_d) + matrix_size, sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMalloc((void **)col_indices_d, nnz[0] * sizeof(int)) );

    // assemble the indices of K
    assemble_K_indices_gpu<<<blocks, threads>>>(
        posx_d, posy_d, posz_d,
        lattice_d, pbc,
        cutoff_radius,
        matrix_size,
        nnz_per_row_d,
        (*row_ptr_d),
        (*col_indices_d)
    );

    hipFree(temp_storage_d);
    hipFree(nnz_per_row_d);
}


void indices_creation_gpu_v2(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const int matrix_size,
    int **col_indices_d,
    int **row_ptr_d,
    int *nnz
)
{
    // parallelize over rows
    int threads = 256;
    int blocks = (matrix_size + threads - 1) / threads;

    int *nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)row_ptr_d, (matrix_size + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&nnz_per_row_d, matrix_size * sizeof(int)) );
    gpuErrchk(hipMemset((*row_ptr_d), 0, (matrix_size + 1) * sizeof(int)) );

    // calculate the nnz per row

    calc_nnz_per_row_gpu_v2<<<blocks, threads, 3*sizeof(double)*threads>>>(posx_d, posy_d, posz_d, lattice_d,
                                                pbc, cutoff_radius, matrix_size, nnz_per_row_d);


    void     *temp_storage_d = NULL;
    size_t   temp_storage_bytes = 0;
    // determines temporary device storage requirements for inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, matrix_size);
    // Allocate temporary storage for inclusive prefix sum
    gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
    // Run inclusive prefix sum
    // inclusive sum starting at second value to get the row ptr
    // which is the same as inclusive sum starting at first value and last value filled with nnz
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, matrix_size);
    
    // nnz is the same as (*row_ptr_d)[matrix_size]
    gpuErrchk( hipMemcpy(nnz, (*row_ptr_d) + matrix_size, sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMalloc((void **)col_indices_d, nnz[0] * sizeof(int)) );

    // assemble the indices of K
    assemble_K_indices_gpu_v2<<<blocks, threads, 3*sizeof(double)*threads>>>(
        posx_d, posy_d, posz_d,
        lattice_d, pbc,
        cutoff_radius,
        matrix_size,
        nnz_per_row_d,
        (*row_ptr_d),
        (*col_indices_d)
    );

    hipFree(temp_storage_d);
    hipFree(nnz_per_row_d);
}





void indices_creation_gpu_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int **col_indices_d,
    int **row_ptr_d,
    int *nnz
)
{
    // parallelize over rows
    int threads = 256;
    int blocks = (block_size_i + threads - 1) / threads;

    int *nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)row_ptr_d, (block_size_i + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&nnz_per_row_d, block_size_i * sizeof(int)) );
    gpuErrchk(hipMemset((*row_ptr_d), 0, (block_size_i + 1) * sizeof(int)) );

    // calculate the nnz per row
    calc_nnz_per_row_gpu_off_diagonal_block<<<blocks, threads>>>(posx_d, posy_d, posz_d, lattice_d, pbc, cutoff_radius,
        block_size_i, block_size_j, block_start_i, block_start_j, nnz_per_row_d);

    void     *temp_storage_d = NULL;
    size_t   temp_storage_bytes = 0;
    // determines temporary device storage requirements for inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, block_size_i);
    // Allocate temporary storage for inclusive prefix sum
    gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
    // Run inclusive prefix sum
    // inclusive sum starting at second value to get the row ptr
    // which is the same as inclusive sum starting at first value and last value filled with nnz
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, nnz_per_row_d, (*row_ptr_d)+1, block_size_i);
    
    // nnz is the same as (*row_ptr_d)[block_size_i]
    gpuErrchk( hipMemcpy(nnz, (*row_ptr_d) + block_size_i, sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMalloc((void **)col_indices_d, nnz[0] * sizeof(int)) );

    // assemble the indices of K
    assemble_K_indices_gpu_off_diagonal_block<<<blocks, threads>>>(
        posx_d, posy_d, posz_d,
        lattice_d, pbc,
        cutoff_radius,
        block_size_i,
        block_size_j,
        block_start_i,
        block_start_j,
        nnz_per_row_d,
        (*row_ptr_d),
        (*col_indices_d)
    );

    hipFree(temp_storage_d);
    hipFree(nnz_per_row_d);
}

__global__ void row_reduce_K_off_diagonal_block(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    double *rows_reduced_d
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        double tmp = 0.0;
        for(int col = 0; col < block_size_j; col++){
            int i = block_start_i + row;
            int j = block_start_j + col;

            bool metal1 = is_in_array_gpu_og(metals_d, element_d[i], num_metals);
            bool metal2 = is_in_array_gpu_og(metals_d, element_d[j], num_metals);
            bool ischarged1 = site_charge_d[i] != 0;
            bool ischarged2 = site_charge_d[j] != 0;
            bool isVacancy1 = element_d[i] == VACANCY;
            bool isVacancy2 = element_d[j] == VACANCY;
            bool cvacancy1 = isVacancy1 && !ischarged1;
            bool cvacancy2 = isVacancy2 && !ischarged2;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i], posx_d[j], posy_d[j], posz_d[j], lattice_d[0], lattice_d[1], lattice_d[2], pbc);

            if (dist < cutoff_radius)
            {
                // sign is switched since the diagonal is positive
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    tmp += d_high_G;
                }
                else
                {
                    tmp += d_low_G;
                }
            }            
        }
        rows_reduced_d[row] = tmp;

    }

}


__global__ void row_reduce_K_off_diagonal_block_with_precomputing(
    const double *posx_d, const double *posy_d, const double *posz_d,
    const double *lattice_d, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int block_size_i,
    int block_size_j,
    int block_start_i,
    int block_start_j,
    int *col_indices_d,
    int *row_ptr_d,
    double *rows_reduced_d
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){
        double tmp = 0.0;
        for(int col = row_ptr_d[row]; col < row_ptr_d[row+1]; col++){
            int i = block_start_i + row;
            int j = block_start_j + col_indices_d[col];

            bool metal1 = is_in_array_gpu_og(metals_d, element_d[i], num_metals);
            bool metal2 = is_in_array_gpu_og(metals_d, element_d[j], num_metals);
            bool ischarged1 = site_charge_d[i] != 0;
            bool ischarged2 = site_charge_d[j] != 0;
            bool isVacancy1 = element_d[i] == VACANCY;
            bool isVacancy2 = element_d[j] == VACANCY;
            bool cvacancy1 = isVacancy1 && !ischarged1;
            bool cvacancy2 = isVacancy2 && !ischarged2;
            double dist = site_dist_gpu_og(posx_d[i], posy_d[i], posz_d[i], posx_d[j], posy_d[j], posz_d[j], lattice_d[0], lattice_d[1], lattice_d[2], pbc);

            if (dist < cutoff_radius)
            {
                // sign is switched since the diagonal is positive
                if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                {
                    tmp += d_high_G;
                }
                else
                {
                    tmp += d_low_G;
                }
            }            
        }
        rows_reduced_d[row] = tmp;

    }

}


__global__ void add_vector_to_diagonal(
    double *data,
    int *row_ptr,
    int *col_indices,
    int matrix_size,
    double *vector
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] += vector[i];
            }
        }
    }
}


__global__ void set_diagonal_to_zero_gpu(
    double *data,
    int *row_ptr,
    int *col_indices,
    int matrix_size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] = 0.0;
            }
        }
    }
}

__global__ void reduced_three_vectors(
    const double * const vec1,
    const double * const vec2,
    const double * const vec3,
    double *vec_out,
    int size
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < size; i += blockDim.x * gridDim.x){
        vec_out[i] = vec1[i] + vec2[i] + vec3[i];
    }
}




template <typename T>
void writeArrayToBinFile(T* array, int numElements, const std::string& filename) {
    std::ofstream file(filename, std::ios::binary);
    if (file.is_open()) {
        file.write(reinterpret_cast<char*>(array), numElements*sizeof(T));
        file.close();
        std::cout << "Array data written to file: " << filename << std::endl;
    } else {
        std::cerr << "Unable to open the file for writing." << std::endl;
    }
}
void test_assemble_A_with_only_precomputing_indices(
    const double *posx, const double *posy, const double *posz,
    const double *lattice, const bool pbc,
    const double cutoff_radius,
    const ELEMENT *metals_d, const ELEMENT *element_d, const int *site_charge_d,
    const int num_metals,
    const double d_high_G, const double d_low_G,
    int K_size,
    int contact_left_size,
    int contact_right_size,
    double **A_data,
    int **A_row_ptr,
    int **A_col_indices,
    int *A_nnz,
    double **K_left_reduced,
    double **K_right_reduced
)
{


    int system_size = K_size - contact_left_size - contact_right_size;
    std::cout << "system size " << system_size << std::endl;

    gpuErrchk(hipMalloc((void **)K_left_reduced, system_size * sizeof(double)));
    gpuErrchk(hipMalloc((void **)K_right_reduced, system_size * sizeof(double)));


    int number_of_measurements = 1;

    // parallelize over rows
    int threads = 256;
    int blocks = (system_size + threads - 1) / threads;

    double *times_device_indices = (double *)malloc(number_of_measurements * sizeof(double));
    double *times_off_diagonal = (double *)malloc(number_of_measurements * sizeof(double));
    double *times_contact_indices = (double *)malloc(number_of_measurements * sizeof(double));
    double *times_reduction = (double *)malloc(number_of_measurements * sizeof(double));
    double *times_add_to_diagonal = (double *)malloc(number_of_measurements * sizeof(double));


    // shift site position to the device
    // reduce the matrix size to the system size
    // works since the positions are ordered
    double time_device_indices;

    for(int i = 0; i < number_of_measurements; i++){
        time_device_indices = -omp_get_wtime();
        gpuErrchk(hipDeviceSynchronize());
        indices_creation_gpu_v2(
            posx + contact_left_size,
            posy + contact_left_size,
            posz + contact_left_size,
            lattice, pbc,
            cutoff_radius,
            system_size,
            A_col_indices,
            A_row_ptr,
            A_nnz
        );
        gpuErrchk(hipDeviceSynchronize());
        time_device_indices += omp_get_wtime();   
        times_device_indices[i] = time_device_indices;
        std::cout << "time_device_indices " << time_device_indices << std::endl;
        if(i < number_of_measurements-1){
            gpuErrchk(hipFree(*A_col_indices));
            gpuErrchk(hipFree(*A_row_ptr));
        }
    }


    // dealocate in measurement loop to not create memory leaks


    // allocate the data array
    gpuErrchk(hipMalloc((void **)A_data, A_nnz[0] * sizeof(double)));

    double time_off_diagonal;

    for(int i = 0; i < number_of_measurements; i++){
        gpuErrchk(hipMemset((*A_data), 0, A_nnz[0] * sizeof(double)));

        time_off_diagonal= -omp_get_wtime();
        gpuErrchk(hipDeviceSynchronize());
        calc_off_diagonal_K_gpu<<<blocks, threads>>>(
            metals_d,
            element_d + contact_left_size,
            site_charge_d + contact_left_size,
            num_metals,
            d_high_G, d_low_G,
            system_size,
            *A_col_indices,
            *A_row_ptr,
            *A_data);
        gpuErrchk(hipDeviceSynchronize());
        time_off_diagonal += omp_get_wtime();
        std::cout << "time_off_diagonal " << time_off_diagonal << std::endl;
        times_off_diagonal[i] = time_off_diagonal;

    }

    // TODO possible faster to calculate once the off diagonal block indices
    // then do the reduction from these given indices
    int *contact_left_row_ptr = NULL;
    int *contact_left_col_indices = NULL;
    int contact_left_nnz;
    int *contact_right_row_ptr = NULL;
    int *contact_right_col_indices = NULL;
    int contact_right_nnz;


    double time_contact_indices;
    for(int i = 0; i < number_of_measurements; i++){
        time_contact_indices = -omp_get_wtime();
        gpuErrchk(hipDeviceSynchronize());
        indices_creation_gpu_off_diagonal_block(
            posx, posy, posz,
            lattice, pbc,
            cutoff_radius,
            system_size,
            contact_left_size,
            contact_left_size,
            0,
            &contact_left_col_indices,
            &contact_left_row_ptr,
            &contact_left_nnz
        );

        indices_creation_gpu_off_diagonal_block(
            posx, posy, posz,
            lattice, pbc,
            cutoff_radius,
            system_size,
            contact_right_size,
            contact_left_size,
            contact_left_size + system_size,
            &contact_right_col_indices,
            &contact_right_row_ptr,
            &contact_right_nnz
        );
        gpuErrchk(hipDeviceSynchronize());
        time_contact_indices += omp_get_wtime();
        std::cout << "time_contact_indices " << time_contact_indices << std::endl;
        times_contact_indices[i] = time_contact_indices;
        // dealocate contact indices to not create memory leaks in the measurement
        if(i < number_of_measurements-1){
            gpuErrchk(hipFree(contact_left_col_indices));
            gpuErrchk(hipFree(contact_left_row_ptr));
            gpuErrchk(hipFree(contact_right_col_indices));
            gpuErrchk(hipFree(contact_right_row_ptr));
        }

    }

    double *A_reduced;
    gpuErrchk(hipMalloc((void **)&A_reduced, system_size * sizeof(double)));
    double time_reduce;

    for(int i = 0; i < number_of_measurements; i++){
        gpuErrchk(hipMemset(A_reduced, 0, system_size * sizeof(double)));
        gpuErrchk(hipMemset((*K_left_reduced), 0, system_size * sizeof(double))); 
        gpuErrchk(hipMemset((*K_right_reduced), 0, system_size * sizeof(double)));
        // reduce the left part of K
        // block starts at i = contact_left_size (first downshifted row)
        // block starts at j = 0 (first column)
        time_reduce = -omp_get_wtime();
        gpuErrchk(hipDeviceSynchronize());

        // reduce the diagonal
        reduce_K_gpu<<<blocks, threads>>>(
            *A_col_indices,
            *A_row_ptr,
            *A_data,
            system_size,
            A_reduced
        );

        row_reduce_K_off_diagonal_block_with_precomputing<<<blocks, threads>>>(
            posx, posy, posz,
            lattice, pbc,
            cutoff_radius,
            metals_d, element_d, site_charge_d,
            num_metals,
            d_high_G, d_low_G,
            system_size,
            contact_left_size,
            contact_left_size,
            0,
            contact_left_col_indices,
            contact_left_row_ptr,
            *K_left_reduced
        );

        // reduce the right part of K
        // block starts at i = contact_left_size (first downshifted row)
        // block starts at j = contact_left_size + system_size (first column)
        row_reduce_K_off_diagonal_block_with_precomputing<<<blocks, threads>>>(
            posx, posy, posz,
            lattice, pbc,
            cutoff_radius,
            metals_d, element_d, site_charge_d,
            num_metals,
            d_high_G, d_low_G,
            system_size,
            contact_right_size,
            contact_left_size,
            contact_left_size + system_size,
            contact_right_col_indices,
            contact_right_row_ptr,
            *K_right_reduced
        );
        gpuErrchk(hipDeviceSynchronize());
        time_reduce += omp_get_wtime();
        std::cout << "time_reduce " << time_reduce << std::endl;

        times_reduction[i] = time_reduce;

    }

    double *A_diag;
    gpuErrchk(hipMalloc((void **)&A_diag, system_size * sizeof(double)));
    double time_add_to_diagonal;

    for(int i = 0; i < number_of_measurements; i++){
        gpuErrchk(hipMemset(A_diag, 0, system_size * sizeof(double))); 
        

        set_diagonal_to_zero_gpu<<<blocks, threads>>>(
            *A_data,
            *A_row_ptr,
            *A_col_indices,
            system_size
        );


        time_add_to_diagonal = -omp_get_wtime();
        gpuErrchk(hipDeviceSynchronize());
        reduced_three_vectors<<<blocks, threads>>>(
            A_reduced,
            *K_left_reduced,
            *K_right_reduced,
            A_diag,
            system_size
        );
        add_vector_to_diagonal<<<blocks, threads>>>(
            *A_data,
            *A_row_ptr,
            *A_col_indices,
            system_size,
            A_diag
        );
        gpuErrchk(hipDeviceSynchronize());
        time_add_to_diagonal += omp_get_wtime();
        std::cout << "time_add_to_diagonal " << time_add_to_diagonal << std::endl;
        times_add_to_diagonal[i] = time_add_to_diagonal;
    }
    // //save times
    // if(system_size < 14000){
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_7k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices, number_of_measurements, base_path + "/times_device_indices.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal, number_of_measurements, base_path + "/times_off_diagonal.bin");
    //     writeArrayToBinFile<double>(times_contact_indices, number_of_measurements, base_path + "/times_contact_indices.bin");
    //     writeArrayToBinFile<double>(times_reduction, number_of_measurements, base_path + "/times_reduction.bin");
    //     writeArrayToBinFile<double>(times_add_to_diagonal, number_of_measurements, base_path + "/times_add_to_diagonal.bin");

    // }
    // else if(system_size > 14000 && system_size < 40000){
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_28k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices, number_of_measurements, base_path + "/times_device_indices.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal, number_of_measurements, base_path + "/times_off_diagonal.bin");
    //     writeArrayToBinFile<double>(times_contact_indices, number_of_measurements, base_path + "/times_contact_indices.bin");
    //     writeArrayToBinFile<double>(times_reduction, number_of_measurements, base_path + "/times_reduction.bin");
    //     writeArrayToBinFile<double>(times_add_to_diagonal, number_of_measurements, base_path + "/times_add_to_diagonal.bin");
    // }
    // else if(system_size > 40000 && system_size < 100000){
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_80k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices, number_of_measurements, base_path + "/times_device_indices.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal, number_of_measurements, base_path + "/times_off_diagonal.bin");
    //     writeArrayToBinFile<double>(times_contact_indices, number_of_measurements, base_path + "/times_contact_indices.bin");
    //     writeArrayToBinFile<double>(times_reduction, number_of_measurements, base_path + "/times_reduction.bin");
    //     writeArrayToBinFile<double>(times_add_to_diagonal, number_of_measurements, base_path + "/times_add_to_diagonal.bin");
    // }
    // else{
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_144k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices, number_of_measurements, base_path + "/times_device_indices.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal, number_of_measurements, base_path + "/times_off_diagonal.bin");
    //     writeArrayToBinFile<double>(times_contact_indices, number_of_measurements, base_path + "/times_contact_indices.bin");
    //     writeArrayToBinFile<double>(times_reduction, number_of_measurements, base_path + "/times_reduction.bin");
    //     writeArrayToBinFile<double>(times_add_to_diagonal, number_of_measurements, base_path + "/times_add_to_diagonal.bin");
    // }

    gpuErrchk(hipFree(A_diag));

    gpuErrchk(hipFree(contact_left_row_ptr));
    gpuErrchk(hipFree(contact_left_col_indices));
    gpuErrchk(hipFree(contact_right_row_ptr));
    gpuErrchk(hipFree(contact_right_col_indices));
    gpuErrchk(hipFree(A_reduced));


    free(times_device_indices);
    free(times_off_diagonal);
    free(times_contact_indices);
    free(times_reduction);
    free(times_add_to_diagonal);
    
    // exit(1);

}



void test_assemble_K(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double cutoff_radius,
                              const int num_metals, int kmc_step_count)
{


    double *K_og;
    K_og = (double *)malloc(N * N * sizeof(double));

    double reltol = 1e-12;
    double abstol = 1e-12;

//     // original code for dense assemble on the gpu
    assemble_K_og(handle, gpubuf, N, N_left_tot, N_right_tot, Vd, pbc, d_high_G, d_low_G, cutoff_radius, num_metals, kmc_step_count, K_og);
    // // count the nonzero elements of the dense K
    int nnz_og = count_nnz(K_og, N * N);

    // // assemble the nonzero elements of K
    // int nnz = calc_nnz(posx, posy, posz, lattice, pbc, cutoff_radius, N);

    // double *data_h = (double *)malloc(nnz * sizeof(double));

    // load the data from the gpu for the sparse assemble
    double *posx = (double *)malloc(N * sizeof(double));
    double *posy = (double *)malloc(N * sizeof(double));
    double *posz = (double *)malloc(N * sizeof(double));
    double *lattice = (double *)malloc(3 * sizeof(double));
    //gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
    //const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    ELEMENT *metals = (ELEMENT *)malloc(num_metals * sizeof(ELEMENT));
    ELEMENT *element = (ELEMENT *)malloc(N * sizeof(ELEMENT));
    int *site_charge = (int *)malloc(N * sizeof(int));

    gpuErrchk(hipMemcpy(posx, gpubuf.site_x, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(posy, gpubuf.site_y, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(posz, gpubuf.site_z, N * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(lattice, gpubuf.lattice, 3 * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(metals, gpubuf.metal_types, num_metals * sizeof(ELEMENT), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(element, gpubuf.site_element, N * sizeof(ELEMENT), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(site_charge, gpubuf.site_charge, N * sizeof(int), hipMemcpyDeviceToHost));



    int number_of_measurements = 1;
    double *times_device_indices_cpu = (double *)malloc(number_of_measurements * sizeof(double));
    double *times_off_diagonal_cpu = (double *)malloc(number_of_measurements * sizeof(double));

    int nnz_cpu;
    int *row_ptr_h = NULL;
    int *col_indices_h = NULL;
    double time_device_indices_cpu;
    double time_off_diagonal_cpu;
    omp_set_num_threads(14);

    for(int i = 0; i < number_of_measurements; i++){
        time_device_indices_cpu = -omp_get_wtime();
        indices_creation_cpu(
            posx + N_left_tot,
            posy + N_left_tot,
            posz + N_left_tot,
            lattice, pbc,
            cutoff_radius,
            N - N_left_tot - N_right_tot,
            &col_indices_h,
            &row_ptr_h,
            &nnz_cpu
        );
        time_device_indices_cpu += omp_get_wtime();
        times_device_indices_cpu[i] = time_device_indices_cpu;
        if(i < number_of_measurements-1){
            free(col_indices_h);
            free(row_ptr_h);
        }
        std::cout << "time_device_indices_cpu " << time_device_indices_cpu << std::endl;
    }

    std::cout << "nnz_cpu " << nnz_cpu << std::endl;

    double *data_h = (double *)malloc(nnz_cpu * sizeof(double));


    for(int i = 0; i < number_of_measurements; i++){
        // set data_h to zero
        #pragma omp parallel for
        for(int j = 0; j < nnz_cpu; j++){
            data_h[j] = 0.0;
        }
        time_off_diagonal_cpu = -omp_get_wtime();
        calc_off_diagonal_K_cpu(
            metals,
            element + N_left_tot,
            site_charge + N_left_tot,
            num_metals,
            d_high_G, d_low_G,
            N - N_left_tot - N_right_tot,
            col_indices_h,
            row_ptr_h,
            data_h);
        time_off_diagonal_cpu += omp_get_wtime();
        times_off_diagonal_cpu[i] = time_off_diagonal_cpu;
        std::cout << "time_off_diagonal_cpu " << time_off_diagonal_cpu << std::endl;
    }

    // // save times
    // if(N - N_left_tot - N_right_tot < 14000){
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_7k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices_cpu, number_of_measurements, base_path + "/times_device_indices_cpu.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal_cpu, number_of_measurements, base_path + "/times_off_diagonal_cpu.bin");
    // }
    // else if(N - N_left_tot - N_right_tot > 14000 && N - N_left_tot - N_right_tot < 40000){
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_28k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices_cpu, number_of_measurements, base_path + "/times_device_indices_cpu.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal_cpu, number_of_measurements, base_path + "/times_off_diagonal_cpu.bin");
    // }
    // else if(N - N_left_tot - N_right_tot > 40000 && N - N_left_tot - N_right_tot < 100000){
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_80k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices_cpu, number_of_measurements, base_path + "/times_device_indices_cpu.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal_cpu, number_of_measurements, base_path + "/times_off_diagonal_cpu.bin");
    // }
    // else{
    //     std::string base_path = "/usr/scratch/mont-fort17/almaeder/kmc_144k/system_K/results";
    //     writeArrayToBinFile<double>(times_device_indices_cpu, number_of_measurements, base_path + "/times_device_indices_cpu.bin");
    //     writeArrayToBinFile<double>(times_off_diagonal_cpu, number_of_measurements, base_path + "/times_off_diagonal_cpu.bin");
    // }

    free(times_device_indices_cpu);
    free(times_off_diagonal_cpu);
    free(data_h);
    free(row_ptr_h);
    free(col_indices_h);



    double *A_data_d = NULL;
    int *A_row_ptr_d = NULL;
    int *A_col_indices_d = NULL;
    int A_nnz;
    double *K_left_reduced_d = NULL;
    double *K_right_reduced_d = NULL;


    test_assemble_A_with_only_precomputing_indices(
        gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.lattice, pbc,
        cutoff_radius,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        num_metals,
        d_high_G, d_low_G,
        N,
        N_left_tot,
        N_right_tot,
        &A_data_d,
        &A_row_ptr_d,
        &A_col_indices_d,
        &A_nnz,
        &K_left_reduced_d,
        &K_right_reduced_d
    );

    if(nnz_cpu != A_nnz){
        std::cout << "nnz mismatch" << std::endl;
    }
    else{
        std::cout << "nnz match" << std::endl;
    }

    std::cout << "A_nnz " << A_nnz << std::endl;
    std::cout << "nnz " << nnz_og << std::endl;
    double *A_data_h = (double *)malloc(A_nnz * sizeof(double));
    int *A_row_ptr_h = (int *)malloc((N - N_left_tot - N_right_tot + 1) * sizeof(int));
    int *A_col_indices_h = (int *)malloc(A_nnz * sizeof(int));



    gpuErrchk( hipMemcpy(A_data_h, A_data_d, A_nnz * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(A_row_ptr_h, A_row_ptr_d, (N - N_left_tot - N_right_tot + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(A_col_indices_h, A_col_indices_d, A_nnz * sizeof(int), hipMemcpyDeviceToHost) );


    // sanity check if the indices are correct
    bool inside_nnz = true;

    for(int i = 0; i < N - N_left_tot - N_right_tot; i++){
        for(int j = A_row_ptr_h[i]; j < A_row_ptr_h[i+1]; j++){
            int row = i + N_left_tot;
            int col = A_col_indices_h[j] + N_left_tot;
            if(K_og[row * N + col] == 0.0){
                inside_nnz = false;
            }
    
        }
    }
    if(!inside_nnz){
        std::cout << "A_indices and K_og mismatch for nnz" << std::endl;
    }
    else{
        std::cout << "A_indices and K_og match for nnz" << std::endl;
    }

    bool outside_nnz = true;
    for(int i = 0; i < N - N_left_tot - N_right_tot; i++){
        for(int j = 0; j < N - N_left_tot - N_right_tot; j++){
            bool inside = false;
            for(int k = A_row_ptr_h[i]; k < A_row_ptr_h[i+1]; k++){
                if(A_col_indices_h[k] == j){
                    inside = true;
                }
            }
            int row = i + N_left_tot;
            int col = j + N_left_tot;            
            if(!inside){
                if(K_og[row * N + col] != 0.0){
                    outside_nnz = false;
                }
            }
            else{
                if(K_og[row * N + col] == 0.0){
                    outside_nnz = false;
                }
            }
        }
    }
    if(!outside_nnz){
        std::cout << "A_indices and K_og mismatch for zeros" << std::endl;
    }
    else{
        std::cout << "A_indices and K_og match for zeros" << std::endl;
    }




    double difference = 0.0;
    double sum_ref = 0.0;
    for(int i = 0; i < N - N_left_tot - N_right_tot; i++){
        for(int j = A_row_ptr_h[i]; j < A_row_ptr_h[i+1]; j++){
            int row = i + N_left_tot;
            int col = A_col_indices_h[j] + N_left_tot;

            difference += std::abs(A_data_h[j] - K_og[row * N + col]) * std::abs(A_data_h[j] - K_og[row * N + col]);
            sum_ref += std::abs(K_og[row * N + col]) * std::abs(K_og[row * N + col]);
        

        }
    }
    if(difference > abstol + reltol * sum_ref){
        std::cout << "A_data_h and K_og mismatch" << std::endl;

    }
    else{
        std::cout << "A_data_h and K_og match" << std::endl;
    }
    std::cout << difference / sum_ref << std::endl;


    gpuErrchk(hipFree(A_data_d));
    gpuErrchk(hipFree(A_col_indices_d));
    gpuErrchk(hipFree(A_row_ptr_d));
    gpuErrchk(hipFree(K_left_reduced_d));
    gpuErrchk(hipFree(K_right_reduced_d));


    free(A_data_h);
    free(A_col_indices_h);
    free(A_row_ptr_h);


    free(K_og);
    free(posx);
    free(posy);
    free(posz);
    free(lattice);
    free(metals);
    free(element);
    free(site_charge);

    std::cout << "K matrix assembled" << std::endl;




    std::cin.ignore();


}

