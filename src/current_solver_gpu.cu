#include "hip/hip_runtime.h"
#include "gpu_solvers.h"
#define NUM_THREADS 512

// Constants needed:
const double eV_to_J = 1.60217663e-19;          // [C]
const double h_bar = 1.054571817e-34;           // [Js]

struct is_defect
{
    __host__ __device__ bool operator()(const ELEMENT element)
    {
        return ((element != DEFECT) && (element != OXYGEN_DEFECT));
    }
};

struct is_not_zero
{
    __host__ __device__ bool operator()(const int integer)
    {
        return (integer != 0);
    }
};

// Collect the indices of the contacts and the vacancies
__global__ void get_is_tunnel(int *is_tunnel, int *tunnel_indices, const ELEMENT *element, 
                              int N_atom, int num_layers_contact, int num_source_inj, int num_ground_ext)
{
    int total_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int idx = total_tid; idx < N_atom; idx += total_threads)
    {
        int yes = 0; 

        // vacancies and contacts have states within the bandgap which are included in the tunneling model
        // include the first layer of the contacts, as the rest are directly connected to it
        // METALS ARE HARDCODED

        if ( element[idx] == VACANCY || 
           ( (element[idx] == Ti_EL || element[idx] == N_EL) &&  (idx > (num_layers_contact - 1)*num_source_inj) && (idx < (N_atom - (num_layers_contact - 1)*num_ground_ext)) )) 
        {
            yes = 1;
        }

        is_tunnel[idx] = yes;
        tunnel_indices[idx] = yes * idx;
    }
}

// Compute the number of nonzeros per row of the matrix including the injection, extraction, and device nodes (excluding the ground). 
// Has dimensions of Nsub by Nsub (by the cpu code)
__global__ void calc_nnz_per_row_T_neighbor( const double *posx_d, const double *posy_d, const double *posz_d,
                                            const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge,
                                            const double *lattice, bool pbc, double nn_dist, const double tol,
                                            int num_source_inj, int num_ground_ext, const int num_layers_contact,
                                            int num_metals, int matrix_size, int *nnz_per_row_d){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int Natom = matrix_size - 2; 
    
    // TODO optimize this with a 2D grid instead of 1D
    for(int i = idx; i < Natom - 1; i += blockDim.x * gridDim.x){  // N_atom - 1 to exclude the ground node

        int nnz_row = 0;

        for(int j = 0; j < Natom - 1; j++){ // N_atom - 1 to exclude the ground node

            double dist = site_dist_gpu(posx_d[i], posy_d[i], posz_d[i],
                                        posx_d[j], posy_d[j], posz_d[j],
                                        lattice[0], lattice[1], lattice[2], pbc);
            
            // diagonal terms
            if ( i == j )
            {
                nnz_row++;
            }

            // direct terms 
            else if ( i != j && dist < nn_dist )
            {
                nnz_row++;
            }
        }

        // this can be memset outside the kernel instead
        // source/ground connections
        if ( i < num_source_inj )
        {
            atomicAdd(&nnz_per_row_d[1], 1);
            nnz_row++;
        }
        if ( i > (Natom - num_ground_ext) )
        {
            atomicAdd(&nnz_per_row_d[0], 1);
            nnz_row++;
        }

        nnz_per_row_d[i+2] = nnz_row;

        if ( i == 0 )
        {
            atomicAdd(&nnz_per_row_d[0], 2); // loop connection and diagonal element
            atomicAdd(&nnz_per_row_d[1], 2); // loop connection and diagonal element
        }
    }

}

__global__ void calc_col_idx_T_neighbor(const double *posx_d, const double *posy_d, const double *posz_d,
                                        const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge,
                                        const double *lattice, bool pbc, double nn_dist, const double tol,
                                        int num_source_inj, int num_ground_ext, const int num_layers_contact,
                                        int num_metals, int matrix_size, int *nnz_per_row_d, int *row_ptr_d, int *col_indices_d)
{
    // row ptr is already calculated
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N_full = matrix_size;
    
    // INDEXED OVER NFULL
    for(int i = idx; i < N_full - 1; i += blockDim.x * gridDim.x){                      // exclude ground node with Nfull - 1

        int nnz_row = 0;

        // loop connection and injection row
        if ( i == 0 )
        {
            for (int j = 0; j < N_full - 1; j++)                                        // exclude ground node with Nfull - 1
            {
                if ( (j < 2) || j > (N_full - num_ground_ext) )
                {
                    col_indices_d[row_ptr_d[i] + nnz_row] = j;
                    nnz_row++;
                }
            }
        }
        // loop connection and extraction row
        if ( i == 1 )
        {
            for (int j = 0; j < num_source_inj + 2; j++)
            {
                col_indices_d[row_ptr_d[i] + nnz_row] = j;
                nnz_row++;
            }
        }

        // inner matrix terms
        if (i >= 2)
        {
            for(int j = 0; j < N_full - 1; j++){                                        // exclude ground node with Nfull - 1

                // add injection term for this row
                if ( (j == 1) && (i < num_source_inj + 2) )
                {
                    col_indices_d[row_ptr_d[i] + nnz_row] = 1;
                    nnz_row++;
                }

                // add extraction term for this row
                if ( (j == 0) && (i > N_full - num_ground_ext) )
                {
                    col_indices_d[row_ptr_d[i] + nnz_row] = 0;
                    nnz_row++;
                }

                if ( j >= 2 ) 
                {
                    double dist = site_dist_gpu(posx_d[i - 2], posy_d[i - 2], posz_d[i - 2],
                                                  posx_d[j - 2], posy_d[j - 2], posz_d[j - 2],
                                                  lattice[0], lattice[1], lattice[2], pbc);
                    
                    // diagonal terms
                    if ( i == j )
                    {
                        col_indices_d[row_ptr_d[i] + nnz_row] = j;
                        nnz_row++;
                    }

                    // direct terms 
                    else if ( i != j && dist < nn_dist )
                    {
                        col_indices_d[row_ptr_d[i] + nnz_row] = j;
                        nnz_row++;
                    }
                }
            }
        }

    }
}


// assemble the data for the T matrix - 1D distribution over rows
__global__ void populate_data_T_neighbor(const double *posx_d, const double *posy_d, const double *posz_d,
                                         const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge,
                                         const double *lattice, bool pbc, double nn_dist, const double tol,
                                         const double high_G, const double low_G, const double loop_G, 
                                         const double Vd, const double m_e, const double V0,
                                         int num_source_inj, int num_ground_ext, const int num_layers_contact,
                                         int num_metals, int matrix_size, int *row_ptr_d, int *col_indices_d, double *data_d)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N_full = matrix_size;
    int N_atom = matrix_size - 2;
    
    for(int i = idx; i < N_full - 1; i += blockDim.x * gridDim.x){

        for( int j = row_ptr_d[i]; j < row_ptr_d[i+1]; j++ )
        {
            // col_indices_d[j] is the index of j in the matrix. j is the index of the data vector
            // if dealing with a diagonal element, we add the positive value from i = i and j = N_full to include the ground node

            // extraction boundary (row)
            if(i == 0)
            {
                // diagonal element (0, 0) --> add the value from (0, N_full)
                if (col_indices_d[j] == 0)
                {
                    data_d[j] = +high_G;
                }
                // loop connection (0, 1)
                if (col_indices_d[j] == 1)
                {
                    data_d[j] = -loop_G;
                }
                // extraction connections from the device
                if ( col_indices_d[j] > N_full - num_ground_ext )
                {
                    data_d[j] = -high_G;
                } 
            }

            // injection boundary (row)
            if(i == 1)
            {
                // loop connection (1, 0)
                if (col_indices_d[j] == 0)
                {
                    data_d[j] = -loop_G;
                }
                // injection connections to the device
                if ( col_indices_d[j] >= 2 || (col_indices_d[j] > N_full - num_ground_ext) )
                {
                    data_d[j] = -high_G;
                } 
            }

            // inner matrix terms
            if (i >= 2)
            {
                // diagonal elements --> add the value from (i - 2, N_full - 2) if site i - 2 neighbors the ground node
                if (i == col_indices_d[j])
                {
                    double dist_angstrom = site_dist_gpu(posx_d[i - 2], posy_d[i - 2], posz_d[i - 2],
                                                         posx_d[N_atom-1], posy_d[N_atom-1], posz_d[N_atom-1], 
                                                         lattice[0], lattice[1], lattice[2], pbc);                                   
                    bool neighboring_ground = (dist_angstrom < nn_dist);
                    
                    if (neighboring_ground) 
                    {
                        data_d[j] = +high_G;     // assuming all the connections to ground come from the right contact
                    } 
                }

                // extraction boundary (column)
                if ( (col_indices_d[j] == 0) && (i > N_full - num_ground_ext) )
                {
                    data_d[j] = -high_G;
                }

                // injection boundary (column)
                if ( (col_indices_d[j] == 1) && (i < num_source_inj + 2) )
                {
                    data_d[j] = -high_G;
                }

                // off-diagonal inner matrix elements
                if ( (col_indices_d[j] >= 2) && (col_indices_d[j] != i)) 
                {

                    double dist_angstrom = site_dist_gpu(posx_d[i - 2], posy_d[i - 2], posz_d[i - 2],
                                                         posx_d[col_indices_d[j] - 2], posy_d[col_indices_d[j] - 2], posz_d[col_indices_d[j] - 2], 
                                                         lattice[0], lattice[1], lattice[2], pbc);                                       
                        
                    bool neighbor = (dist_angstrom < nn_dist);                                                      

                    // direct terms (neighbor connections)
                    if ( neighbor )
                    {
                        // contacts
                        bool metal1 = is_in_array_gpu<ELEMENT>(metals, element[i - 2], num_metals);
                        bool metal2 = is_in_array_gpu<ELEMENT>(metals, element[col_indices_d[j] - 2], num_metals);

                        // conductive vacancy sites
                        bool cvacancy1 = (element[i - 2] == VACANCY) && (atom_charge[i - 2] == 0);
                        bool cvacancy2 = (element[col_indices_d[j] - 2] == VACANCY) && (atom_charge[col_indices_d[j] - 2] == 0);
                        
                        if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
                        {
                            data_d[j] = -high_G;
                        }
                        else
                        {
                            data_d[j] = -low_G;
                        }
                    }

                }
            }
        }
    }
}


__global__ void populate_data_T_tunnel(double *X, const double *posx, const double *posy, const double *posz,
                                       const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge,
                                       const double *lattice, bool pbc, double high_G, double low_G, double loop_G,
                                       double nn_dist, double m_e, double V0, int num_source_inj, int num_ground_ext, const int num_layers_contact,
                                       int N_atom, int num_tunnel_points, const int *tunnel_indices, int num_metals, const double Vd, const double tol)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    int N = num_tunnel_points;

    for (auto idx = tid_total; idx < N * N; idx += num_threads_total)
    {    

        int i = idx / N;
        int j = idx % N;

        double posx_i = posx[tunnel_indices[i]];
        double posx_j = posx[tunnel_indices[j]];

        double posy_i = posy[tunnel_indices[i]];
        double posy_j = posy[tunnel_indices[j]];

        double posz_i = posz[tunnel_indices[i]];
        double posz_j = posz[tunnel_indices[j]];

        double atom_CB_edge_i = atom_CB_edge[tunnel_indices[i]];
        double atom_CB_edge_j = atom_CB_edge[tunnel_indices[j]];

        ELEMENT element_i = element[tunnel_indices[i]];
        ELEMENT element_j = element[tunnel_indices[j]];

        double dist_angstrom = site_dist_gpu(posx_i, posy_i, posz_i, 
                                             posx_j, posy_j, posz_j, 
                                             lattice[0], lattice[1], lattice[2], pbc);

        bool neighbor = (dist_angstrom < nn_dist) && (i != j);

        // tunneling terms occur between not-neighbors
        if (i != j && !neighbor)
        { 
            bool any_vacancy1 = element_i == VACANCY;
            bool any_vacancy2 = element_j == VACANCY;

            // contacts, the last layer has already been excluded when creating the tunnel indices
            bool metal1p = is_in_array_gpu(metals, element_i, num_metals);
            bool metal2p = is_in_array_gpu(metals, element_j, num_metals);

            // types of tunnelling conditions considered
            bool trap_to_trap = (any_vacancy1 && any_vacancy2);
            bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
            bool contact_to_contact = (metal1p && metal2p);

            double local_E_drop = atom_CB_edge_i - atom_CB_edge_j;                // [eV] difference in energy between the two atoms

            // compute the WKB tunneling coefficients for all the tunnelling conditions
            if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
            {
                
                double prefac = -(sqrt( 2 * m_e ) / h_bar) * (2.0 / 3.0);           // [s/(kg^1/2 * m^2)] coefficient inside the exponential
                double dist = (1e-10)*dist_angstrom;                                // [m] 3D distance between atoms i and j

                if (contact_to_trap)
                {
                    double energy_window = fabs(local_E_drop);                      // [eV] energy window for tunneling from the contacts
                    double dV = 0.01;                                               // [V] energy spacing for numerical integration
                    double dE = eV_to_J * dV;                                       // [eV] energy spacing for numerical integration
                        
                    // integrate over all the occupied energy levels in the contact
                    double T = 0.0;
                    for (double iv = 0; iv < energy_window; iv += dE)
                    {
                        double E1 = eV_to_J * V0 + iv;                                  // [J] Energy distance to CB before tunnelling
                        double E2 = E1 - fabs(local_E_drop);                            // [J] Energy distance to CB after tunnelling

                        if (E2 > 0)                                                     // trapezoidal potential barrier (low field)                 
                        {                                                           
                            T += exp(prefac * (dist / fabs(local_E_drop)) * ( pow(E1, 1.5) - pow(E2, 1.5) ) );
                        }

                        if (E2 < 0)                                                      // triangular potential barrier (high field)                               
                        {
                            T += exp(prefac * (dist / fabs(local_E_drop)) * ( pow(E1, 1.5) )); 
                        } 
                    }
                    X[N * i + j] = -T;      
                } 
                else 
                {
                    double E1 = eV_to_J * V0;                                        // [J] Energy distance to CB before tunnelling
                    double E2 = E1 - fabs(local_E_drop);                             // [J] Energy distance to CB after tunnelling
                          
                    if (E2 > 0)                                                      // trapezoidal potential barrier (low field)
                    {                                                           
                        double T = exp(prefac * (dist / fabs(E1 - E2)) * ( pow(E1, 1.5) - pow(E2, 1.5) ) );
                        X[N * i + j] = -T; 
                    }

                    if (E2 < 0)                                                        // triangular potential barrier (high field)
                    {
                        double T = exp(prefac * (dist / fabs(E1 - E2)) * ( pow(E1, 1.5) ));
                        X[N * i + j] = -T; 
                    }
                }
            }
        }
        
    }
}


__global__ void calc_diagonal_T_gpu( int *col_indices, int *row_ptr, double *data, int matrix_size, double *diagonal)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size - 1; i += blockDim.x * gridDim.x){ // MINUS ONE
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i != col_indices[j]){
                tmp += data[j];
            }
        }
        diagonal[i] = -tmp;
        //write the sum of the off-diagonals onto the existing diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i == col_indices[j]){
                data[j] += -tmp;
            }
        }
    }
}


__global__ void update_m(double *m, long minidx, int np2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int bid = blockIdx.x;

    if (idx < np2)
    {
        double minm = m[minidx];
        m[idx] += abs(minm);
    }
}


__global__ void copy_pdisp(double *site_power, ELEMENT *element, const ELEMENT *metals, double *pdisp, int *atom_gpu_index, int N_atom,
                           const int num_metals, const double alpha)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int idx = tid; idx < N_atom; idx += total_threads)
    {
        bool metal = is_in_array_gpu(metals, element[atom_gpu_index[idx]], num_metals);
        if (!metal)
            site_power[atom_gpu_index[idx]] = -1 * alpha * pdisp[idx];
    }
}

//extracts the diagonal of the dense submatrix into a global vector
__global__ void extract_diag_tunnel(
    double *tunnel_matrix,
    int *tunnel_indices, 
    int num_tunnel_points,
    double *diagonal
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < num_tunnel_points; i += blockDim.x * gridDim.x)
    {
        // +2 since first two indices are the ground and injection nodes
        diagonal[tunnel_indices[i] + 2] += tunnel_matrix[i * num_tunnel_points + i];
    }
}

__global__ void inverse_vector(double *vec, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < N; i += blockDim.x * gridDim.x)
    {
        vec[i] = 1.0 / vec[i];
    }
}

template <int NTHREADS>
__global__ void get_imacro_sparse(const double *x_values, const int *x_row_ptr, const int *x_col_ind,
                                  const double *m, double *imacro)
{
    int num_threads = blockDim.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int total_tid = bid * num_threads + tid;
    int total_threads = num_threads * gridDim.x;

    int row_start = x_row_ptr[1] + 2;
    int row_end = x_row_ptr[2];

    __shared__ double buf[NTHREADS];
    buf[tid] = 0.0;
 
    for (int idx = row_start + total_tid; idx < row_end; idx += total_threads)
    {
        int col_index = x_col_ind[idx];
        if (col_index >= 2) 
        {
            // buf[tid] += x_values[idx] * (m[0] - m[col_index]);               // extracted (= injected when including ground node)
            buf[tid] += x_values[idx] * (m[col_index] - m[1]);                  // injected
        }
    }

    int width = num_threads / 2;
    while (width != 0)
    {
        __syncthreads();
        if (tid < width)
        {
            buf[tid] += buf[tid + width];
        }
        width /= 2;
    }

    if (tid == 0)
    {
        atomicAdd(imacro, buf[0]);
    }
}

// new version with split matrix for neighbor/tunnel connections
void update_power_gpu_split(hipblasHandle_t handle, hipsolverHandle_t handle_cusolver, GPUBuffers &gpubuf, 
                            const int num_source_inj, const int num_ground_ext, const int num_layers_contact,
                            const double Vd, const int pbc, const double high_G, const double low_G, const double loop_G, const double G0, const double tol,
                            const double nn_dist, const double m_e, const double V0, int num_metals, double *imacro,
                            const bool solve_heating_local, const bool solve_heating_global, const double alpha_disp)
{
    auto t0 = std::chrono::steady_clock::now();

    // ***************************************************************************************
    // 1. Update the atoms array from the sites array using copy_if with is_defect as a filter
    int *gpu_index;
    int *atom_gpu_index;
    gpuErrchk( hipMalloc((void **)&gpu_index, gpubuf.N_ * sizeof(int)) );                                           // indices of the site array
    gpuErrchk( hipMalloc((void **)&atom_gpu_index, gpubuf.N_ * sizeof(int)) );                                      // indices of the atom array

    thrust::device_ptr<int> gpu_index_ptr = thrust::device_pointer_cast(gpu_index);
    thrust::sequence(gpu_index_ptr, gpu_index_ptr + gpubuf.N_, 0);

    // do these in parallel with a kernel! - check that the positions dont change
    // check if there's some buffer which can be allocated and reused for all of these
    double *last_atom = thrust::copy_if(thrust::device, gpubuf.site_x, gpubuf.site_x + gpubuf.N_, gpubuf.site_element, gpubuf.atom_x, is_defect());
    int N_atom = last_atom - gpubuf.atom_x;
    thrust::copy_if(thrust::device, gpubuf.site_y, gpubuf.site_y + gpubuf.N_, gpubuf.site_element, gpubuf.atom_y, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_z, gpubuf.site_z + gpubuf.N_, gpubuf.site_element, gpubuf.atom_z, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_charge, gpubuf.site_charge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_charge, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_element, gpubuf.site_element + gpubuf.N_, gpubuf.site_element, gpubuf.atom_element, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_CB_edge, gpubuf.site_CB_edge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_CB_edge, is_defect());
    thrust::copy_if(thrust::device, gpu_index, gpu_index + gpubuf.N_, gpubuf.site_element, atom_gpu_index, is_defect());

    // auto t1 = std::chrono::steady_clock::now();
    // std::chrono::duration<double> dt = t1 - t0;
    // std::cout << "time to update atom arrays: " << dt.count() << "\n";

    // ***************************************************************************************
    // 2. Collect the indices of the contacts and the vacancies    
    int num_threads = 1024;
    int blocks_per_row = (N_atom - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * N_atom;

    // indices of the tunneling connections (contacts and vacancies) in the Natom array
    int *is_tunnel; // [0, 1, 0, 0, 1...] where 1 indicates a tunnel connection
    int *is_tunnel_indices; // [0, 1, 0, 0, 4...] storing the indices of the tunnel connections
    gpuErrchk( hipMalloc((void **)&is_tunnel, N_atom * sizeof(int)) );    
    gpuErrchk( hipMalloc((void **)&is_tunnel_indices, N_atom * sizeof(int)) );                                         
    get_is_tunnel<<<num_blocks, num_threads>>>(is_tunnel, is_tunnel_indices, gpubuf.atom_element, N_atom, num_layers_contact, num_source_inj, num_ground_ext);
    gpuErrchk( hipPeekAtLastError() );
    // check if global counter could be faster

    // auto tx1 = std::chrono::steady_clock::now();
    // std::chrono::duration<double> dtx1 = tx1 - t1;
    // std::cout << "getting the bool array: " << dtx1.count() << "\n";

    // boolean array of whether this location in Natoms is a tunnel connection or not
    int num_tunnel_points = thrust::reduce(thrust::device, is_tunnel, is_tunnel + N_atom, 0); // sum([0, 1, 0, 0, 1...])
    gpuErrchk( hipPeekAtLastError() );
    std::cout << "size of tunneling submatrix: " << num_tunnel_points << "\n";

    int *tunnel_indices; // [1, 4...]
    gpuErrchk( hipMalloc((void **)&tunnel_indices, num_tunnel_points * sizeof(int)) ); 
    thrust::copy_if(thrust::device, is_tunnel_indices, is_tunnel_indices + gpubuf.N_, tunnel_indices, is_not_zero());
    
    // // debug
    // int *check_tunnel_inds = new int[num_tunnel_points];
    // gpuErrchk( hipMemcpy(check_tunnel_inds, tunnel_indices, num_tunnel_points * sizeof(int), hipMemcpyDeviceToHost) );
    // for (int i = 0; i < num_tunnel_points; i++)
    // {
    //     std::cout << check_tunnel_inds[i] << " ";
    // }
    // exit(1);
    // // end debug

    // **************************************************************************
    // 3. Assemble the sparsity pattern of the sparse neighbor matrix
    int Nfull = N_atom + 2;
    int matrix_size = Nfull; 

    // get the number of nonzeros per row
    int *neighbor_nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)&neighbor_nnz_per_row_d, matrix_size * sizeof(int)) );
    gpuErrchk( hipMemset(neighbor_nnz_per_row_d, 0, matrix_size * sizeof(int)) );

    num_threads = 512;
    num_blocks = (matrix_size + num_threads - 1) / num_threads;
    calc_nnz_per_row_T_neighbor<<<num_blocks, num_threads>>>(gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                                                             gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                                                             gpubuf.lattice, pbc, nn_dist, tol,
                                                             num_source_inj, num_ground_ext, num_layers_contact,
                                                             num_metals, matrix_size, neighbor_nnz_per_row_d);
    gpuErrchk( hipPeekAtLastError() );

    // compute the row pointers with an inclusive sum:
    int *neighbor_row_ptr_d;
    gpuErrchk( hipMalloc((void **)&neighbor_row_ptr_d, (matrix_size + 1 - 1) * sizeof(int)) );
    gpuErrchk( hipMemset(neighbor_row_ptr_d, 0, (matrix_size + 1 - 1) * sizeof(int)) );
    
    void     *temp_storage_d = NULL;                                                          // determines temporary device storage requirements for inclusive prefix sum
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, neighbor_nnz_per_row_d, neighbor_row_ptr_d+1, matrix_size - 1); // subtract 1 to ignore the ground node
    gpuErrchk( hipMalloc(&temp_storage_d, temp_storage_bytes) );                             // inclusive sum starting at second value to get the row ptr, which is the same as inclusive sum starting at first value and last value filled with nnz
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes, neighbor_nnz_per_row_d, neighbor_row_ptr_d+1, matrix_size - 1);
    
    // get the number of nonzero elements:
    int neighbor_nnz;
    gpuErrchk( hipMemcpy(&neighbor_nnz, neighbor_row_ptr_d + matrix_size - 1, sizeof(int), hipMemcpyDeviceToHost) );
    std::cout << "\nsparse nnz: " << neighbor_nnz << std::endl;

    // assemble the column indices from 0 to Nsub (excluding the ground node)
    int *neighbor_col_indices_d;
    gpuErrchk( hipMalloc((void **)&neighbor_col_indices_d, neighbor_nnz * sizeof(int)) );
    calc_col_idx_T_neighbor<<<num_blocks, num_threads>>>(gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                                                         gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                                                         gpubuf.lattice, pbc, nn_dist, tol,
                                                         num_source_inj, num_ground_ext, num_layers_contact,
                                                         num_metals, matrix_size, neighbor_nnz_per_row_d,
                                                         neighbor_row_ptr_d, neighbor_col_indices_d);
    gpuErrchk( hipPeekAtLastError() );

     // **************************************************************************
    // 4. Populate the entries of the sparse Natom matrix

    double *neighbor_data_d;
    gpuErrchk(hipMalloc((void **)&neighbor_data_d, neighbor_nnz * sizeof(double)));
    gpuErrchk(hipMemset(neighbor_data_d, 0, neighbor_nnz * sizeof(double)));

    num_threads = 512;
    num_blocks = (Nfull + num_threads - 1) / num_threads;
    populate_data_T_neighbor<<<num_blocks, num_threads>>>(gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                                                          gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                                                          gpubuf.lattice, pbc, nn_dist, tol, high_G, low_G, loop_G,
                                                          Vd, m_e, V0,
                                                          num_source_inj, num_ground_ext, num_layers_contact,
                                                          num_metals, Nfull, neighbor_row_ptr_d, neighbor_col_indices_d, neighbor_data_d);
    gpuErrchk( hipPeekAtLastError() );

    // output sparsity of neighbor connections
    int submatrix_size = Nfull - 1;
    dump_csr_matrix_txt(submatrix_size, neighbor_nnz, neighbor_row_ptr_d, neighbor_col_indices_d, neighbor_data_d, 0);
    std::cout << "dumped sparse neighbor matrix\n";

    // the Nsub matrix of just the sparse neighbor connections is contained in [neighbor_row_ptr_d, neighbor_col_indices_d, neighbor_data_d]

    // *************************************************************************************************************************************
    // 5. Populate the dense matrix corresponding to all of the tunnel connections, using tunnel_indices to index the atom attributes arrays

    double *tunnel_matrix_d;
    gpuErrchk(hipMalloc((void **)&tunnel_matrix_d, num_tunnel_points * num_tunnel_points * sizeof(double)));
    gpuErrchk(hipMemset(tunnel_matrix_d, 0, num_tunnel_points * num_tunnel_points * sizeof(double)));

    num_threads = 512;
    num_blocks = (num_tunnel_points + num_threads - 1) / num_threads;
    populate_data_T_tunnel<<<num_blocks, num_threads>>>(tunnel_matrix_d, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                                                        gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                                                        gpubuf.lattice, pbc, high_G, low_G, loop_G, nn_dist, m_e, V0,
                                                        num_source_inj, num_ground_ext, num_layers_contact, N_atom, num_tunnel_points, tunnel_indices,
                                                        num_metals, Vd, tol);
    gpuErrchk( hipPeekAtLastError() );

    // **************************************************************************
    // 6. Reduce the diagonals
    double *diagonal_d;
    gpuErrchk( hipMalloc((void **)&diagonal_d, Nfull * sizeof(double)) );
    gpuErrchk( hipMemset(diagonal_d, 0, Nfull * sizeof(double) ) );


    // reduce the diagonal for the sparse banded matrix
    num_threads = 512;
    num_blocks = (Nfull + num_threads - 1) / num_threads;
    calc_diagonal_T_gpu<<<num_blocks, num_threads>>>(neighbor_col_indices_d, neighbor_row_ptr_d, neighbor_data_d, Nfull, diagonal_d);
    gpuErrchk( hipPeekAtLastError() );

    // reduce the diagonal for the dense tunnel matrix
    double *tunnel_diag_d;
    gpuErrchk( hipMalloc((void **)&tunnel_diag_d, num_tunnel_points * sizeof(double)) );                              // diagonal elements of the transmission matrix
    gpuErrchk( hipMemset(tunnel_diag_d, 0, num_tunnel_points * sizeof(double)) );

    num_threads = 512;
    blocks_per_row = (num_tunnel_points - 1) / num_threads + 1;
    num_blocks = blocks_per_row * (N_atom + 2);

    row_reduce<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(tunnel_matrix_d, tunnel_diag_d, num_tunnel_points);
    gpuErrchk( hipPeekAtLastError() );

    write_to_diag<<<blocks_per_row, num_threads>>>(tunnel_matrix_d, tunnel_diag_d, num_tunnel_points);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    //diagonal_d contains already the diagonal of the neighbor matrix
    extract_diag_tunnel<<<blocks_per_row, num_threads>>>(
        tunnel_matrix_d,
        tunnel_indices, 
        num_tunnel_points,
        diagonal_d);
    inverse_vector<<<blocks_per_row, num_threads>>>(diagonal_d, Nfull);

    double *diagonal_inv_d = diagonal_d;


    // double *diagonal_inv_h = (double *)calloc(Nfull, sizeof(double));
    // gpuErrchk( hipMemcpy(diagonal_inv_h, diagonal_inv_d, Nfull * sizeof(double), hipMemcpyDeviceToHost) );
    // for (int i = 0; i < Nfull; i++){
    //     std::cout << diagonal_inv_h[i] << " ";
    // }   
    // std::cout << "\n";
    // exit(1);


    // the sparse matrix of the neighbor connectivity is contained in [neighbor_row_ptr_d, neighbor_col_indices_d, neighbor_data_d]
    // the dense matrix of the non-neighbor connectivity is contained in [tunnel_matrix_d] with size num_tunnel_points
    // To build the full matrix, row i and column i of tunnel_matrix_d should be added to row tunnel_indices[i] and col tunnel_indices[i] of the neighbor matrix

    //debug
    // double *cpu_T = new double[num_tunnel_points * num_tunnel_points];
    // hipMemcpy(cpu_T, tunnel_matrix_d, sizeof(double) * num_tunnel_points * num_tunnel_points, hipMemcpyDeviceToHost);
    // std::cout << "printing tunnel matrix\n";
    // std::ofstream fout2("T.txt");
    // int row, col;
    // for (row = 0; row < num_tunnel_points; row++) {
    // for (col = 0; col < num_tunnel_points; col++) {
    //     fout2 << cpu_T[row * num_tunnel_points + col] << ' ';
    // }
    // fout2 << '\n';
    // }
    // fout2.close(); 
    //debug end

    //debug
    // int *check_tunnel_inds = new int[num_tunnel_points];
    // gpuErrchk( hipMemcpy(check_tunnel_inds, tunnel_indices, num_tunnel_points * sizeof(int), hipMemcpyDeviceToHost) );
    // std::cout << "printing tunnel indices\n";
    // std::ofstream fout("insertion_indices.txt");
    // for (int i = 0; i < num_tunnel_points; i++)
    // {
    //     fout << check_tunnel_inds[i] << ' ';
    // }
    // fout.close(); 
    //debug end

    // results of debug: checked against the full sparse assembly by reassembling the matrix in a python script 

    std::cout << "matrix population is done\n";

    // **************************************************************************
    // 7. Prepare the RHS vector

    double *gpu_m;
    gpuErrchk( hipMalloc((void **)&gpu_m, (N_atom + 2) * sizeof(double)) );                                 // [] current injection vector
    gpuErrchk( hipMemset(gpu_m, 0, (N_atom + 2) * sizeof(double)) );                                                                         
    thrust::device_ptr<double> m_ptr = thrust::device_pointer_cast(gpu_m);
    thrust::fill(m_ptr, m_ptr + 1, -loop_G * Vd);                                                            // max Current extraction (ground)                          
    thrust::fill(m_ptr + 1, m_ptr + 2, loop_G * Vd);                                                         // max Current injection (source)
    hipDeviceSynchronize();

    // ************************************************************
    // 8. Solve the system of linear equations 
    
    // the initial guess for the solution is the current site-resolved potential inside the device
    double *gpu_virtual_potentials;
    gpuErrchk( hipMalloc((void **)&gpu_virtual_potentials, (N_atom + 2) * sizeof(double)) );                   // [V] Virtual potential vector  
    gpuErrchk( hipMemset(gpu_virtual_potentials, 0, (N_atom + 2) * sizeof(double)) );                          // initialize the rhs for solving the system                                    
    
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);
    hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_DEVICE);

    // sparse solver without preconditioning:
    int Nsub = Nfull - 1;
    solve_sparse_CG_splitmatrix(handle, cusparseHandle, tunnel_matrix_d, num_tunnel_points, 
                                neighbor_data_d, neighbor_row_ptr_d, neighbor_col_indices_d, neighbor_nnz, 
                                Nsub, tunnel_indices, gpu_m, gpu_virtual_potentials, diagonal_inv_d);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    double check_element;
    gpuErrchk( hipMemcpy(&check_element, gpu_virtual_potentials + num_source_inj, sizeof(double), hipMemcpyDeviceToHost) );
    if (std::abs(check_element - Vd) > 0.1)
    {
        std::cout << "WARNING: non-negligible potential drop of " << std::abs(check_element - Vd) <<
                    " across the contact at VD = " << Vd << "\n";
    }

    std::cout << "done system solve\n";
    // exit(1);

    // auto t4 = std::chrono::steady_clock::now();
    // std::chrono::duration<double> dt3 = t4 - t3;
    // std::cout << "time to solve linear system: " << dt3.count() << "\n";


    // // ****************************************************
    // // 3. Calculate the net current flowing into the device
    double *gpu_imacro;
    gpuErrchk( hipMalloc((void **)&gpu_imacro, 1 * sizeof(double)) );                                       // [A] The macroscopic device current
    hipDeviceSynchronize();


    // // scale the virtual potentials by G0 (conductance quantum) instead of multiplying inside the X matrix
    thrust::device_ptr<double> gpu_virtual_potentials_ptr = thrust::device_pointer_cast(gpu_virtual_potentials);
    thrust::transform(gpu_virtual_potentials_ptr, gpu_virtual_potentials_ptr + N_atom + 2, gpu_virtual_potentials_ptr, thrust::placeholders::_1 * G0);

    // // macroscopic device current
    gpuErrchk( hipMemset(gpu_imacro, 0, sizeof(double)) ); 
    hipDeviceSynchronize();

    // // dot product of first row of X[i] times M[0] - M[i]
    num_threads = 512;
    num_blocks = (N_atom - 1) / num_threads + 1;
    get_imacro_sparse<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(
        neighbor_data_d, neighbor_row_ptr_d, neighbor_col_indices_d, gpu_virtual_potentials, gpu_imacro);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    gpuErrchk( hipMemcpy(imacro, gpu_imacro, sizeof(double), hipMemcpyDeviceToHost) );

    std::cout << solve_heating_local  << "\n";
    std::cout << solve_heating_global << "\n";

    //TODO does not work now
    // implement the heating calculation (possible from the splitting)
    // ineg would be possible the following way: -aij*xij so -aij xsparseij - aij xdenseij

    exit(1);

    // std::cout << "I_macro: " << *imacro * (1e6) << "\n";
    // std::cout << "exiting after I_macro\n"; exit(1);

    // hipFree(X_data);
    // hipFree(X_data_copy);
    // hipFree(X_row_ptr);
    // hipFree(X_col_indices);
    // hipFree(gpu_virtual_potentials);
    // hipFree(gpu_imacro);
    // hipFree(gpu_m);
    // hipFree(gpu_index);
    // hipFree(atom_gpu_index);
}

// *** FULL SPARSE MATRIX VERSION ***



// does not assume that the column indices are sorted
__global__ void set_ineg_sparse(double *ineg_values, int *ineg_row_ptr, int *ineg_col_indices, const double *x_values, const int *x_row_ptr, const int *x_col_indices, const double *m, double Vd, int N)
{
    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto i = tid_total; i < N; i += num_threads_total)
    {
        for( int j = ineg_row_ptr[i]; j < ineg_row_ptr[i+1]; j++ )
        {
            if (ineg_col_indices[j] >= 2)
            {
                ineg_values[j] = 0.0;

                double ical = x_values[j] * (m[i + 2] - m[ineg_col_indices[j] + 2]);

                if (ical < 0 && Vd > 0)
                {
                    ineg_values[j] = -ical;
                }
                else if (ical > 0 && Vd < 0)
                {
                    ineg_values[j] = -ical;
                }
            }
        }
    }
}


// full sparse matrix assembly
void update_power_gpu_sparse(hipblasHandle_t handle, hipsolverHandle_t handle_cusolver, GPUBuffers &gpubuf, 
                             const int num_source_inj, const int num_ground_ext, const int num_layers_contact,
                             const double Vd, const int pbc, const double high_G, const double low_G, const double loop_G, const double G0, const double tol,
                             const double nn_dist, const double m_e, const double V0, int num_metals, double *imacro,
                             const bool solve_heating_local, const bool solve_heating_global, const double alpha_disp)
{
    auto t0 = std::chrono::steady_clock::now();

    // ***************************************************************************************
    // 1. Update the atoms array from the sites array using copy_if with is_defect as a filter
    int *gpu_index;
    int *atom_gpu_index;
    gpuErrchk( hipMalloc((void **)&gpu_index, gpubuf.N_ * sizeof(int)) );                                           // indices of the site array
    gpuErrchk( hipMalloc((void **)&atom_gpu_index, gpubuf.N_ * sizeof(int)) );                                      // indices of the atom array

    thrust::device_ptr<int> gpu_index_ptr = thrust::device_pointer_cast(gpu_index);
    thrust::sequence(gpu_index_ptr, gpu_index_ptr + gpubuf.N_, 0);

    double *last_atom = thrust::copy_if(thrust::device, gpubuf.site_x, gpubuf.site_x + gpubuf.N_, gpubuf.site_element, gpubuf.atom_x, is_defect());
    int N_atom = last_atom - gpubuf.atom_x;
    thrust::copy_if(thrust::device, gpubuf.site_y, gpubuf.site_y + gpubuf.N_, gpubuf.site_element, gpubuf.atom_y, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_z, gpubuf.site_z + gpubuf.N_, gpubuf.site_element, gpubuf.atom_z, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_charge, gpubuf.site_charge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_charge, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_element, gpubuf.site_element + gpubuf.N_, gpubuf.site_element, gpubuf.atom_element, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_CB_edge, gpubuf.site_CB_edge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_CB_edge, is_defect());
    thrust::copy_if(thrust::device, gpu_index, gpu_index + gpubuf.N_, gpubuf.site_element, atom_gpu_index, is_defect());

    auto t1 = std::chrono::steady_clock::now();
    std::chrono::duration<double> dt = t1 - t0;
    std::cout << "time to update atom arrays: " << dt.count() << "\n";

    // ***************************************************************************************
    // 2. Assemble the transmission matrix (X) with both direct and tunnel connections and the
    // solution vector (M) which represents the current inflow/outflow
    // int N_full = N_atom + 2;                                                                               // number of atoms + injection node + extraction node
    int Nsub = N_atom + 1;                                                                                 // N_full minus the ground node which is cut from the graph

    // compute the index arrays to build the CSR representation of X (from 0 to Nsub):
    int *X_row_ptr;
    int *X_col_indices;
    int X_nnz = 0;
    Assemble_X_sparsity(N_atom, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                        gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                        gpubuf.lattice, pbc, nn_dist, tol, 
                        num_source_inj, num_ground_ext, num_layers_contact,
                        num_metals, &X_row_ptr, &X_col_indices, &X_nnz);

    // get the row indices for COO
    int *X_row_indices_h = new int[X_nnz];
    int *X_row_ptr_h = new int[N_atom + 2];

    gpuErrchk( hipMemcpy(X_row_ptr_h, X_row_ptr, (N_atom + 2) * sizeof(int), hipMemcpyDeviceToHost) );
    for(int i = 0; i < N_atom + 1; i++){
        for(int j = X_row_ptr_h[i]; j < X_row_ptr_h[i+1]; j++){
            X_row_indices_h[j] = i;
        }
    }
    int *X_row_indices;
    gpuErrchk( hipMalloc((void **)&X_row_indices, X_nnz * sizeof(int)) );
    gpuErrchk( hipMemcpy(X_row_indices, X_row_indices_h, X_nnz * sizeof(int), hipMemcpyHostToDevice) );
    free(X_row_indices_h);
    free(X_row_ptr_h);
    
    auto t2 = std::chrono::steady_clock::now();
    std::chrono::duration<double> dt1 = t2 - t1;
    std::cout << "time to assemble X sparsity: " << dt1.count() << "\n";

    // Assemble the nonzero value array of X in CSR (from 0 to Nsub):
    double *X_data;                                                                                             // [1] Transmission matrix
    // Assemble_X(N_atom, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
    //            gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
    //            gpubuf.lattice, pbc, nn_dist, tol, Vd, m_e, V0, high_G, low_G, loop_G,
    //            num_source_inj, num_ground_ext, num_layers_contact,
    //            num_metals, &X_data, &X_row_ptr, &X_col_indices, &X_nnz);

    // double *X_data2;                                                                                          // [1] Transmission matrix
    Assemble_X2(N_atom, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                gpubuf.lattice, pbc, nn_dist, tol, Vd, m_e, V0, high_G, low_G, loop_G,
                num_source_inj, num_ground_ext, num_layers_contact,
                num_metals, &X_data, &X_row_indices, &X_row_ptr, &X_col_indices, &X_nnz);

    // dump_csr_matrix_txt(Nsub, X_nnz, X_row_ptr, X_col_indices, X_data, 0); // figure out why the vector lengths are wrong according to the python script
    // std::cout << "dumped sparse matrix\n";
    // exit(1);
    
    // gpuErrchk( hipFree(X_row_indices) );
    // double *X_data_h = new double[X_nnz];
    // double *X_data2_h = new double[X_nnz];
    // gpuErrchk( hipMemcpy(X_data_h, X_data, X_nnz * sizeof(double), hipMemcpyDeviceToHost) );
    // gpuErrchk( hipMemcpy(X_data2_h, X_data2, X_nnz * sizeof(double), hipMemcpyDeviceToHost) );

    // for (int i = 0; i < X_nnz; i++)
    // {

    //     // if (X_data_h[i] == X_data2_h[i])
    //     // {
    //     //     std::cout << "X_data match at index " << i << " with value " << X_data_h[i] << "\n";
    //     // }
    //     if (X_data_h[i] != X_data2_h[i])
    //     {
    //         std::cout << "X_data mismatch at index " << i << " with values " << X_data_h[i] << " and " << X_data2_h[i] << "\n";
    //     }
    // }

    auto t3 = std::chrono::steady_clock::now();
    std::chrono::duration<double> dt2 = t3 - t2;
    std::cout << "time to assemble X data: " << dt2.count() << "\n";

    double *gpu_imacro, *gpu_m;
    gpuErrchk( hipMalloc((void **)&gpu_imacro, 1 * sizeof(double)) );                                       // [A] The macroscopic device current
    gpuErrchk( hipMalloc((void **)&gpu_m, (N_atom + 2) * sizeof(double)) );                                 // [V] Virtual potential vector    
    hipDeviceSynchronize();

    gpuErrchk( hipMemset(gpu_m, 0, (N_atom + 2) * sizeof(double)) );                                        // initialize the rhs for solving the system                                    
    thrust::device_ptr<double> m_ptr = thrust::device_pointer_cast(gpu_m);
    thrust::fill(m_ptr, m_ptr + 1, -loop_G * Vd);                                                            // max Current extraction (ground)                          
    thrust::fill(m_ptr + 1, m_ptr + 2, loop_G * Vd);                                                         // max Current injection (source)
    hipDeviceSynchronize();

    // ************************************************************
    // 2. Solve system of linear equations 
    
    // the initial guess for the solution is the current site-resolved potential inside the device
    double *gpu_virtual_potentials = gpubuf.atom_virtual_potentials;                                               // [V] Virtual potential vector  
    
    // making a copy so the original version won't be preconditioned inside the iterative solver
    double *X_data_copy;
    gpuErrchk( hipMalloc((void **)&X_data_copy, X_nnz * sizeof(double)) );
    gpuErrchk( hipMemcpyAsync(X_data_copy, X_data, X_nnz * sizeof(double), hipMemcpyDeviceToDevice) );
    gpuErrchk( hipDeviceSynchronize() );

    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);
    hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_DEVICE);

    // sparse solver with Jacobi preconditioning:
    // std::cout << "\n\n";
    solve_sparse_CG_Jacobi(handle, cusparseHandle, X_data_copy, X_row_ptr, X_col_indices, X_nnz, Nsub, gpu_m, gpu_virtual_potentials);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    // std::cout << "exiting after system solve\n";
    // exit(1);

    double check_element;
    gpuErrchk( hipMemcpy(&check_element, gpu_virtual_potentials + num_source_inj, sizeof(double), hipMemcpyDeviceToHost) );
    if (std::abs(check_element - Vd) > 0.1)
    {
        std::cout << "WARNING: non-negligible potential drop of " << std::abs(check_element - Vd) <<
                    " across the contact at VD = " << Vd << "\n";
    }

    auto t4 = std::chrono::steady_clock::now();
    std::chrono::duration<double> dt3 = t4 - t3;
    std::cout << "time to solve linear system: " << dt3.count() << "\n";


    // ****************************************************
    // 3. Calculate the net current flowing into the device

    // scale the virtual potentials by G0 (conductance quantum) instead of multiplying inside the X matrix
    thrust::device_ptr<double> gpu_virtual_potentials_ptr = thrust::device_pointer_cast(gpu_virtual_potentials);
    thrust::transform(gpu_virtual_potentials_ptr, gpu_virtual_potentials_ptr + N_atom + 2, gpu_virtual_potentials_ptr, thrust::placeholders::_1 * G0);

    // macroscopic device current
    gpuErrchk( hipMemset(gpu_imacro, 0, sizeof(double)) ); 
    hipDeviceSynchronize();

    // dot product of first row of X[i] times M[0] - M[i]
    int num_threads = 512;
    int num_blocks = (N_atom - 1) / num_threads + 1;
    get_imacro_sparse<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(X_data, X_row_ptr, X_col_indices, gpu_virtual_potentials, gpu_imacro);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    gpuErrchk( hipMemcpy(imacro, gpu_imacro, sizeof(double), hipMemcpyDeviceToHost) );

    auto t5 = std::chrono::steady_clock::now();
    std::chrono::duration<double> dt4 = t5 - t4;
    std::cout << "time to compute current: " << dt4.count() << "\n";

    // std::cout << "I_macro: " << *imacro * (1e6) << "\n";
    // std::cout << "exiting after I_macro\n"; exit(1);

    // **********************************************
    // 4. Calculate the dissipated power at each atom

if (solve_heating_local || solve_heating_global)
{    
    // Shift the virtual potential so that it is all positive, as we will take differences
    double min_index = thrust::min_element(thrust::device, gpu_virtual_potentials + 2, gpu_virtual_potentials + N_atom + 2) - gpu_virtual_potentials;
    num_threads = 512;
    num_blocks = (N_atom + 2 - 1) / num_threads + 1;
    update_m<<<num_blocks, num_threads>>>(gpu_virtual_potentials, min_index, N_atom + 2);
    gpuErrchk( hipPeekAtLastError() );

    // Collect the forward currents into I_neg, the diagonals are once again the sum of each row
    int *ineg_row_ptr;                                                                                          // [A] Current inflow matrix
    int *ineg_col_indices;
    double *ineg_data;
    gpuErrchk( hipMalloc((void**) &ineg_row_ptr, (N_atom + 1 + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void**) &ineg_col_indices, X_nnz * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)&ineg_data, X_nnz * sizeof(double)) );
    gpuErrchk( hipMemcpyAsync(ineg_row_ptr, X_row_ptr, (N_atom + 1 + 1) * sizeof(int), hipMemcpyDeviceToDevice) );
    gpuErrchk( hipMemcpyAsync(ineg_col_indices, X_col_indices, X_nnz * sizeof(int), hipMemcpyDeviceToDevice) );
    gpuErrchk( hipMemset(ineg_data, 0, X_nnz*sizeof(double)) ); 
    hipDeviceSynchronize();

    num_threads = 512;
    num_blocks = (Nsub - 1) / num_threads + 1;
    int N_atomsub = N_atom - 1;
    set_ineg_sparse<<<num_blocks, num_threads>>>(ineg_data, ineg_row_ptr, ineg_col_indices, X_data, X_row_ptr, X_col_indices, gpu_virtual_potentials, Vd, N_atomsub);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // sum off-diagonals into diagonal:
    num_threads = 512;
    num_blocks = (Nsub - 1) / num_threads + 1;
    reduce_rows_into_diag<<<num_blocks, num_threads>>>(ineg_col_indices, ineg_row_ptr, ineg_data, Nsub);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // Compute the dissipated power at each atom with [P]_Nx1 = [I]_NxN * [V]_Nx1 (gemv --> spmv)
    double *gpu_pdisp;
    gpuErrchk( hipMalloc((void **)&gpu_pdisp, N_atom * sizeof(double)) );                                   // [W] Dissipated power vector
    gpuErrchk( hipMemset(gpu_pdisp, 0, N_atom*sizeof(double)) ); 

    hipsparseStatus_t status;
    hipsparseSpMatDescr_t mat_ineg;
    status = hipsparseCreateCsr(&mat_ineg, Nsub, Nsub, X_nnz, ineg_row_ptr, ineg_col_indices, ineg_data, 
                               HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "ERROR: creation of sparse matrix descriptor in update_power_gpu_sparse() failed!\n";
    }
    hipsparseDnVecDescr_t vec_virtual_potentials, vec_pdisp;
    hipsparseCreateDnVec(&vec_virtual_potentials, Nsub, gpu_virtual_potentials, HIP_R_64F);
    hipsparseCreateDnVec(&vec_pdisp, Nsub, gpu_pdisp, HIP_R_64F);

    size_t MVBufferSize;
    void *MVBuffer = 0;
    double *one_d, *zero_d;
    double one = 1.0;
    double zero = 0.0;
    gpuErrchk( hipMalloc((void**)&one_d, sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&zero_d, sizeof(double)) );
    gpuErrchk( hipMemcpy(one_d, &one, sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(zero_d, &zero, sizeof(double), hipMemcpyHostToDevice) );

    status = hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, mat_ineg, 
                                     vec_virtual_potentials, zero_d, vec_pdisp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &MVBufferSize);  
    gpuErrchk( hipMalloc((void**)&MVBuffer, sizeof(double) * MVBufferSize) );
    status = hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, mat_ineg,                         
                          vec_virtual_potentials, zero_d, vec_pdisp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, MVBuffer);          
    
    // copy the dissipated power into the site attributes
    num_threads = 512;
    num_blocks = (N_atom - 1) / num_threads + 1;
    num_blocks = min(65535, num_blocks);
    copy_pdisp<<<num_blocks, num_threads>>>(gpubuf.site_power, gpubuf.site_element, gpubuf.metal_types, gpu_pdisp, atom_gpu_index, N_atom, num_metals, alpha_disp);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // !!! the dissipated power does not yet perfectly match the dense version !!!
    // !!! there is probably a small expected change due to removing the ground node, but this should be double checked !!!
    
    // double *host_pdisp = new double[N_atom];
    // hipMemcpy(host_pdisp, gpu_pdisp, N_atom * sizeof(double), hipMemcpyDeviceToHost);
    // double sum = 0.0;
    // for (int i = 0; i < N_atom; ++i) {
    //     sum += host_pdisp[i];
    // }
    // std::cout << "Sum of atom-resolved power: " << sum << std::endl;
    // exit(1);

    hipFree(ineg_row_ptr);
    hipFree(ineg_col_indices);
    hipFree(ineg_data);
    hipFree(gpu_pdisp);
    hipFree(MVBuffer); 
    hipFree(one_d);
    hipFree(zero_d);
}

    hipFree(X_data);
    hipFree(X_data_copy);
    hipFree(X_row_ptr);
    hipFree(X_row_indices);
    hipFree(X_col_indices);
    hipFree(gpu_imacro);
    hipFree(gpu_m);
    hipFree(gpu_index);
    hipFree(atom_gpu_index);
}

// *** DENSE MATRIX VERSION ***

__global__ void create_X(
    double *X,
    const double *posx, const double *posy, const double *posz,
    const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge,
    const double *lattice, bool pbc, double high_G, double low_G, double loop_G,
    double nn_dist, double m_e, double V0, int num_source_inj, int num_ground_ext, const int num_layers_contact,
    int N, int num_metals, const double Vd, const double tol)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    int N_full = N + 2;

    // TODO: Does it make sense to restructure for N_full * N_full threads?
    for (auto idx = tid_total; idx < N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        double dist_angstrom = site_dist_gpu(posx[i], posy[i], posz[i], 
                                             posx[j], posy[j], posz[j], 
                                             lattice[0], lattice[1], lattice[2], pbc);

        bool neighbor = (dist_angstrom < nn_dist) && (i != j);

        // tunneling terms occur between not-neighbors
        if (i != j && !neighbor)
        { 
            bool any_vacancy1 = element[i] == VACANCY;
            bool any_vacancy2 = element[j] == VACANCY;

            // contacts, excluding the last layer 
            bool metal1p = is_in_array_gpu(metals, element[i], num_metals) 
                                       && (i > ((num_layers_contact - 1)*num_source_inj))
                                       && (i < (N - (num_layers_contact - 1)*num_ground_ext)); 

            bool metal2p = is_in_array_gpu(metals, element[j], num_metals)
                                       && (j > ((num_layers_contact - 1)*num_source_inj))
                                       && (j < (N - (num_layers_contact - 1)*num_ground_ext));  

            // types of tunnelling conditions considered
            bool trap_to_trap = (any_vacancy1 && any_vacancy2);
            bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
            bool contact_to_contact = (metal1p && metal2p);

            double local_E_drop = atom_CB_edge[i] - atom_CB_edge[j];                // [eV] difference in energy between the two atoms

            // compute the WKB tunneling coefficients for all the tunnelling conditions
            if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
            {
                
                double prefac = -(sqrt( 2 * m_e ) / h_bar) * (2.0 / 3.0);           // [s/(kg^1/2 * m^2)] coefficient inside the exponential
                double dist = (1e-10)*dist_angstrom;                                // [m] 3D distance between atoms i and j

                if (contact_to_trap)
                {
                    double energy_window = fabs(local_E_drop);                      // [eV] energy window for tunneling from the contacts
                    double dV = 0.01;                                               // [V] energy spacing for numerical integration
                    double dE = eV_to_J * dV;                                       // [eV] energy spacing for numerical integration
                        
                    // integrate over all the occupied energy levels in the contact
                    double T = 0.0;
                    for (double iv = 0; iv < energy_window; iv += dE)
                    {
                        double E1 = eV_to_J * V0 + iv;                                  // [J] Energy distance to CB before tunnelling
                        double E2 = E1 - fabs(local_E_drop);                            // [J] Energy distance to CB after tunnelling

                        if (E2 > 0)                                                     // trapezoidal potential barrier (low field)                 
                        {                                                           
                            T += exp(prefac * (dist / fabs(local_E_drop)) * ( pow(E1, 1.5) - pow(E2, 1.5) ) );
                        }

                        if (E2 < 0)                                                      // triangular potential barrier (high field)                               
                        {
                            T += exp(prefac * (dist / fabs(local_E_drop)) * ( pow(E1, 1.5) )); 
                        } 
                    }
                    X[N_full * (i + 2) + (j + 2)] = -T;      
                } 
                else 
                {
                    double E1 = eV_to_J * V0;                                        // [J] Energy distance to CB before tunnelling
                    double E2 = E1 - fabs(local_E_drop);                             // [J] Energy distance to CB after tunnelling
                          
                    if (E2 > 0)                                                      // trapezoidal potential barrier (low field)
                    {                                                           
                        double T = exp(prefac * (dist / fabs(E1 - E2)) * ( pow(E1, 1.5) - pow(E2, 1.5) ) );
                        X[N_full * (i + 2) + (j + 2)] = -T; 
                    }

                    if (E2 < 0)                                                        // triangular potential barrier (high field)
                    {
                        double T = exp(prefac * (dist / fabs(E1 - E2)) * ( pow(E1, 1.5) ));
                        X[N_full * (i + 2) + (j + 2)] = -T; 
                    }
                }
            }
        }

        // direct terms occur between neighbors 
        if (i != j && neighbor)
        {
            // contacts
            bool metal1 = is_in_array_gpu(metals, element[i], num_metals);
            bool metal2 = is_in_array_gpu(metals, element[j], num_metals);

            // conductive vacancy sites
            bool cvacancy1 = (element[i] == VACANCY) && (atom_charge[i] == 0);
            bool cvacancy2 = (element[j] == VACANCY) && (atom_charge[j] == 0);

            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                X[N_full * (i + 2) + (j + 2)] = -high_G;
            }
            else
            {
                X[N_full * (i + 2) + (j + 2)] = -low_G;
            }
        }

        // NOTE: Is there a data race here?
        // connect the source/ground nodes to the first/last contact layers
        if (i < num_source_inj && j == 0)
        {
            X[1 * N_full + (i + 2)] = -high_G;
            X[(i + 2) * N_full + 1] = -high_G;
        }

        if (i > (N - num_ground_ext) && j == 0)
        {
            X[0 * N_full + (i + 2)] = -high_G;
            X[(i + 2) * N_full + 0] = -high_G;
        }

        if (i == 0 && j == 0)
        {
            X[0 * N_full + 1] = -loop_G;
            X[1 * N_full + 0] = -loop_G;
        }
    }
}


template <int NTHREADS>
__global__ void get_imacro(const double *x, const double *m, double *imacro, int N)
{
    int num_threads = blockDim.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int total_tid = bid * num_threads + tid;

    __shared__ double buf[NTHREADS];

    buf[tid] = 0.0;

    if ((total_tid >= 0 && total_tid < N) && (total_tid >= 2)) 
    {
        buf[tid] = x[(N + 2) * 0 + (total_tid + 2)] * (m[0] - m[total_tid + 2]);            // extracted (M[0] = 0)
    }

    int width = num_threads / 2;
    while (width != 0)
    {
        __syncthreads();
        if (tid < width)
        {
            buf[tid] += buf[tid + width];
        }
        width /= 2;
    }

    if (tid == 0)
    {
        atomicAdd(imacro, buf[0]);
    }
}

__global__ void set_ineg(double *ineg, const double *x, const double *m, double Vd, int N)
{
    // ineg is matrix N x N
    // x is matrix (N+2) x (N+2)
    // m is vector (N + 2)

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto idx = tid_total; idx < N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        ineg[i * N + j] = 0.0;
        double ical = x[(N + 2) * (i + 2) + (j + 2)] * (m[i + 2] - m[j + 2]);
        
        if (ical < 0 && Vd > 0)
        {
            ineg[i * N + j] = -ical;
        }
        else if (ical > 0 && Vd < 0)
        { 
            ineg[i * N + j] = -ical;
        }
    }
}


void update_power_gpu(hipblasHandle_t handle, hipsolverHandle_t handle_cusolver, GPUBuffers &gpubuf, 
                      const int num_source_inj, const int num_ground_ext, const int num_layers_contact,
                      const double Vd, const int pbc, const double high_G, const double low_G, const double loop_G, const double G0, const double tol,
                      const double nn_dist, const double m_e, const double V0, int num_metals, double *imacro,
                      const bool solve_heating_local, const bool solve_heating_global, const double alpha_disp)
{

    // ***************************************************************************************
    // 1. Update the atoms array from the sites array using copy_if with is_defect as a filter
    int *gpu_index;
    int *atom_gpu_index;
    gpuErrchk( hipMalloc((void **)&gpu_index, gpubuf.N_ * sizeof(int)) );                                           // indices of the site array
    gpuErrchk( hipMalloc((void **)&atom_gpu_index, gpubuf.N_ * sizeof(int)) );                                      // indices of the atom array

    thrust::device_ptr<int> gpu_index_ptr = thrust::device_pointer_cast(gpu_index);
    thrust::sequence(gpu_index_ptr, gpu_index_ptr + gpubuf.N_, 0);

    double *last_atom = thrust::copy_if(thrust::device, gpubuf.site_x, gpubuf.site_x + gpubuf.N_, gpubuf.site_element, gpubuf.atom_x, is_defect());
    int N_atom = last_atom - gpubuf.atom_x;
    thrust::copy_if(thrust::device, gpubuf.site_y, gpubuf.site_y + gpubuf.N_, gpubuf.site_element, gpubuf.atom_y, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_z, gpubuf.site_z + gpubuf.N_, gpubuf.site_element, gpubuf.atom_z, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_charge, gpubuf.site_charge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_charge, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_element, gpubuf.site_element + gpubuf.N_, gpubuf.site_element, gpubuf.atom_element, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_CB_edge, gpubuf.site_CB_edge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_CB_edge, is_defect());
    thrust::copy_if(thrust::device, gpu_index, gpu_index + gpubuf.N_, gpubuf.site_element, atom_gpu_index, is_defect());

    // ***************************************************************************************
    // 2. Assemble the transmission matrix (X) with both direct and tunnel connections and the
    // solution vector (M) which represents the current inflow/outflow

    // USE SIZE_T FOR ALLOCATIONS
    double *gpu_imacro, *gpu_m, *gpu_x, *gpu_ineg, *gpu_diag, *gpu_pdisp, *gpu_A;
    gpuErrchk( hipMalloc((void **)&gpu_imacro, 1 * sizeof(double)) );                                       // [A] The macroscopic device current
    gpuErrchk( hipMalloc((void **)&gpu_m, (N_atom + 2) * sizeof(double)) );                                 // [V] Virtual potential vector    
    gpuErrchk( hipMalloc((void **)&gpu_x, (N_atom + 2) * (N_atom + 2) * sizeof(double)) );                  // [1] Transmission matrix
    gpuErrchk( hipMalloc((void **)&gpu_ineg, N_atom * N_atom * sizeof(double)) );                           // [A] Current inflow matrix
    gpuErrchk( hipMalloc((void **)&gpu_diag, (N_atom + 2) * sizeof(double)) );                              // diagonal elements of the transmission matrix
    gpuErrchk( hipMalloc((void **)&gpu_pdisp, N_atom * sizeof(double)) );                                   // [W] Dissipated power vector
    gpuErrchk( hipMalloc((void **)&gpu_A, (N_atom + 1) * (N_atom + 1) * sizeof(double)) );                  // A - copy buffer for the transmission matrix
    hipDeviceSynchronize();

    gpuErrchk( hipMemset(gpu_x, 0, (N_atom + 2) * (N_atom + 2) * sizeof(double)) );                         // initialize the transmission matrix to zeros
    gpuErrchk( hipMemset(gpu_m, 0, (N_atom + 2) * sizeof(double)) );                                        // initialize the rhs for solving the system                                    
    thrust::device_ptr<double> m_ptr = thrust::device_pointer_cast(gpu_m);
    thrust::fill(m_ptr, m_ptr + 1, -loop_G * Vd);                                               // max Current extraction (ground)                          
    thrust::fill(m_ptr + 1, m_ptr + 2, loop_G * Vd);                                            // max Current injection (source)
    hipDeviceSynchronize();

    int num_threads = 128;
    int blocks_per_row = (N_atom - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * gpubuf.N_;

    // fill off diagonals of X
    create_X<<<num_blocks, num_threads>>>(
        gpu_x, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
        gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
        gpubuf.lattice, pbc, high_G, low_G, loop_G,
        nn_dist, m_e, V0, num_source_inj, num_ground_ext, num_layers_contact,
        N_atom, num_metals, Vd, tol);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // fill diagonal of X (all rows sum to zero)
    gpuErrchk( hipMemset(gpu_diag, 0, (N_atom + 2) * sizeof(double)) );
    num_threads = 512;
    blocks_per_row = (N_atom + 2 - 1) / num_threads + 1;
    num_blocks = blocks_per_row * (gpubuf.N_ + 2);
    row_reduce<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_x, gpu_diag, N_atom + 2);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    write_to_diag<<<blocks_per_row, num_threads>>>(gpu_x, gpu_diag, N_atom + 2);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // ************************************************************
    // 2. Solve system of linear equations using LU (direct solver)

    int lwork = 0;              /* size of workspace */
    double *gpu_work = nullptr; /* device workspace for getrf */
    int *gpu_info = nullptr;    /* error info */
    int *gpu_ipiv;

    gpuErrchk( hipMalloc((void **)&gpu_ipiv, (N_atom + 1) * sizeof(int)) );
    gpuErrchk( hipMalloc((void **)(&gpu_info), sizeof(int)) );
    gpuErrchk( hipMemcpy2D(gpu_A, (N_atom + 1) * sizeof(double), gpu_x, (N_atom + 2) * sizeof(double), (N_atom + 1) * sizeof(double), (N_atom + 1), hipMemcpyDeviceToDevice) );
    hipDeviceSynchronize();

    // Solve Ax=B through LU factorization
    CheckCusolverDnError(hipsolverDnDgetrf_bufferSize(handle_cusolver, N_atom + 1, N_atom + 1, gpu_A, N_atom + 1, &lwork));
    gpuErrchk( hipMalloc((void **)(&gpu_work), sizeof(double) * lwork) );
    hipDeviceSynchronize();
    CheckCusolverDnError(hipsolverDnDgetrf(handle_cusolver, N_atom + 1, N_atom + 1, gpu_A, N_atom + 1, gpu_work, gpu_ipiv, gpu_info));
    hipDeviceSynchronize();
    CheckCusolverDnError(hipsolverDnDgetrs(handle_cusolver, HIPBLAS_OP_T, N_atom + 1, 1, gpu_A, N_atom + 1, gpu_ipiv, gpu_m, N_atom + 1, gpu_info));
    hipDeviceSynchronize();

    int host_info;
    gpuErrchk( hipMemcpy(&host_info, gpu_info, sizeof(int), hipMemcpyDeviceToHost) );
    if (host_info)
    {
        std::cout << "WARNING: Info for gesv in update_power is " << host_info << "\n";
    }

    double check_element;
    gpuErrchk( hipMemcpy(&check_element, gpu_m + num_source_inj, sizeof(double), hipMemcpyDeviceToHost) );
    if (std::abs(check_element - Vd) > 0.1)
    {
        std::cout << "WARNING: non-negligible potential drop of " << std::abs(check_element - Vd) <<
                    " across the contact at VD = " << Vd << "\n";
    }

    // scale the virtual potentials by G0 (conductance quantum) instead of multiplying inside the X matrix
    thrust::device_ptr<double> gpu_m_ptr = thrust::device_pointer_cast(gpu_m);
    thrust::transform(gpu_m_ptr, gpu_m_ptr + N_atom + 1, gpu_m_ptr, thrust::placeholders::_1 * G0);

    // ****************************************************
    // 3. Calculate the net current flowing into the device

    num_threads = 512;
    num_blocks = (N_atom - 1) / num_threads + 1;
    gpuErrchk( hipMemset(gpu_imacro, 0, sizeof(double)) ); 
    get_imacro<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_x, gpu_m, gpu_imacro, N_atom);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    gpuErrchk( hipMemcpy(imacro, gpu_imacro, sizeof(double), hipMemcpyDeviceToHost) );
    std::cout << "I_macro: " << *imacro * (1e6) << "\n";

    // **********************************************
    // 4. Calculate the dissipated power at each atom

if (solve_heating_local || solve_heating_global)
{   
        // Shift the virtual potential so that it is all positive, as we will take differences
        double min_index = thrust::min_element(thrust::device, gpu_m + 2, gpu_m + N_atom + 2) - gpu_m;
        num_threads = 512;
        blocks_per_row = (N_atom + 2 - 1) / num_threads + 1;
        num_blocks = blocks_per_row;
        update_m<<<num_blocks, num_threads>>>(gpu_m, min_index, N_atom + 2);
        gpuErrchk( hipPeekAtLastError() );

        // Collect the forward currents into I_neg, the diagonals are once again the sum of each row
        num_threads = 512;
        blocks_per_row = (N_atom - 1) / num_threads + 1;
        num_blocks = blocks_per_row * gpubuf.N_;
        set_ineg<<<num_blocks, num_threads>>>(gpu_ineg, gpu_x, gpu_m, Vd, N_atom);
        gpuErrchk( hipPeekAtLastError() );
        hipDeviceSynchronize();
        gpuErrchk( hipMemset(gpu_diag, 0, (N_atom + 2) * sizeof(double)) );
        hipDeviceSynchronize();
        row_reduce<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_ineg, gpu_diag, N_atom);
        gpuErrchk( hipPeekAtLastError() );
        hipDeviceSynchronize();
        write_to_diag<<<blocks_per_row, num_threads>>>(gpu_ineg, gpu_diag, N_atom);
        gpuErrchk( hipPeekAtLastError() );
        hipDeviceSynchronize();

        // Compute the dissipated power at each atom with [P]_Nx1 = [I]_NxN * [V]_Nx1 (gemv)
        double alpha = 1.0, beta = 0.0;
        CheckCublasError( hipblasDgemv(handle, HIPBLAS_OP_T, N_atom, N_atom, &alpha, gpu_ineg, N_atom, gpu_m + 2, 1, &beta, gpu_pdisp, 1) );
        hipDeviceSynchronize();

        // Extract the power dissipated between the contacts
        num_threads = 512;
        num_blocks = (N_atom - 1) / num_threads + 1;
        num_blocks = min(65535, num_blocks);
        copy_pdisp<<<num_blocks, num_threads>>>(gpubuf.site_power, gpubuf.site_element, gpubuf.metal_types, gpu_pdisp, atom_gpu_index, N_atom, num_metals, alpha_disp);
        gpuErrchk( hipPeekAtLastError() );
        hipDeviceSynchronize();

        double *host_pdisp = new double[N_atom];
        hipMemcpy(host_pdisp, gpu_pdisp, N_atom * sizeof(double), hipMemcpyDeviceToHost);
        double sum = 0.0;
        for (int i = 0; i < N_atom; ++i) {
            sum += host_pdisp[i];
        }
        std::cout << "Sum of atom-resolved power * 1e9: " << sum*(1e9) << std::endl;
        // exit(1);
} // if (solve_heating_local || solve_heating_global)

    hipFree(gpu_ipiv);
    hipFree(gpu_work);
    hipFree(gpu_imacro);
    hipFree(gpu_m);
    hipFree(gpu_x);
    hipFree(gpu_ineg);
    hipFree(gpu_diag);
    hipFree(gpu_pdisp);
    hipFree(gpu_A);
    hipFree(gpu_info);
    hipFree(gpu_index);
    hipFree(atom_gpu_index);
}