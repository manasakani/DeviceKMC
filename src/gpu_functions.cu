#include "hip/hip_runtime.h"
#include "cuda_wrapper.h"
#include <stdio.h>
#include <vector>
#include <cassert>
#include <hip/hip_runtime.h>
#include <cmath>
#include <math.h>

#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>

// #include <hipsparse.h>
#include <hipsparse.h>

const double eV_to_J = 1.6e-19;
const double h_bar_sq = 4.3957e-67;
constexpr double kB = 8.617333262e-5;     
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include "gpu_buffers.h"
#define NUM_THREADS 512
#define MAX_NUM_LAYERS 5

// in GPU cache
__constant__ double E_gen_const[MAX_NUM_LAYERS];
__constant__ double E_rec_const[MAX_NUM_LAYERS];
__constant__ double E_Vdiff_const[MAX_NUM_LAYERS];
__constant__ double E_Odiff_const[MAX_NUM_LAYERS];

void get_gpu_info(char *gpu_string, int dev){
 struct hipDeviceProp_t dprop;
 hipGetDeviceProperties(&dprop, dev);
 strcpy(gpu_string,dprop.name);
}

void set_gpu(int dev){
 hipSetDevice(dev);
}

// ********************************************************
// *************** GPU HELPER FUNCTIONS *******************
// ********************************************************

struct is_defect
{
    __host__ __device__ bool operator()(const ELEMENT element)
    {
        return (element != DEFECT);
    }
};

// Define a structure to hold COO elements
struct COOElement {
    int row;
    int col;
    double value;
};

// returns true if thing is present in the array of things
template <typename T>
__device__ int is_in_array_gpu(const T *array, const T element, const int size) {

    for (int i = 0; i < size; ++i) {
        if (array[i] == element) {
        return 1;
        }
    }
    return 0;
}

__device__ double site_dist_gpu(double pos1x, double pos1y, double pos1z,
                                double pos2x, double pos2y, double pos2z,
                                double lattx, double latty, double lattz, bool pbc)
{

    double dist = 0;

    if (pbc == 1)
    {
        double dist_x = pos1x - pos2x;
        double distance_frac[3];

        distance_frac[1] = (pos1y - pos2y) / latty;
        distance_frac[1] -= round(distance_frac[1]);
        distance_frac[2] = (pos1z - pos2z) / lattz;
        distance_frac[2] -= round(distance_frac[2]);

        double dist_xyz[3];
        dist_xyz[0] = dist_x;

        dist_xyz[1] = distance_frac[1] * latty;
        dist_xyz[2] = distance_frac[2] * lattz;

        dist = sqrt(dist_xyz[0] * dist_xyz[0] + dist_xyz[1] * dist_xyz[1] + dist_xyz[2] * dist_xyz[2]);
        
    }
    else
    {
        dist = sqrt(pow(pos2x - pos1x, 2) + pow(pos2y - pos1y, 2) + pow(pos2z - pos1z, 2));
    }

    return dist;
}

__device__ double v_solve_gpu(double r_dist, int charge, const double *sigma, const double *k) { 

    double q = 1.60217663e-19;              // [C]
    double vterm = static_cast<double>(charge) * erfc(r_dist / ((*sigma) * sqrt(2.0))) * (*k) * q / r_dist; 

    return vterm;
}


// ********************************************************
// ******************** KERNELS ***************************
// ********************************************************

__global__ void remake_atom_list(int N, double *site_x, double *site_y, double *site_z, ELEMENT *metals, const ELEMENT *element, double *site_potential, double *site_power,
                                 double *atom_x, double *atom_y, double *atom_z, ELEMENT *atom_element, double *atom_potential,
                                 double *atom_power, int *Natoms)
{
    // Removes defects sites from the site list
    // One thread kernel at the moment
    int atom_index = 0;

    for (auto i = 0; i < N; i += 1)
    {
        if (element[i] == DEFECT || element[i] == OXYGEN_DEFECT)
        {
            // do nothing
        }
        else
        {
            atom_x[atom_index] = site_x[i];
            atom_y[atom_index] = site_y[i];
            atom_z[atom_index] = site_z[i];
            atom_element[atom_index] = element[i];
            atom_potential[atom_index] = site_potential[i];
            atom_power[atom_index] = site_power[i];
            ++atom_index;
        }
    }
    Natoms[0] = atom_index;
}

__global__ void set_potential(double *A, double *B, int N)
{
    int didx = blockIdx.x * blockDim.x + threadIdx.x;
    for (auto i = didx; i < N; i += gridDim.x * blockDim.x)
    {
        A[i] = -B[i];
    }
}

__global__ void set_diag_K(double *A, double *diag, int N)
{
    int didx = blockIdx.x * blockDim.x + threadIdx.x;
    for (auto i = didx; i < N; i += gridDim.x * blockDim.x)
    {
        double tmp = A[i];
        A[i] = tmp + diag[i];
    }
}

template <int NTHREADS>
__global__ void diagonal_sum_K(
    double *A,
    double *diag,
    double *V,
    int N, int NI, int NJ)
{

    int num_threads = blockDim.x;
    int blocks_per_row = (NJ - 1) / num_threads + 1;
    int block_id = blockIdx.x;

    int row = block_id / blocks_per_row;
    int scol = (block_id % blocks_per_row) * num_threads;
    int lcol = min(NJ, scol + num_threads);

    int tid = threadIdx.x;

    __shared__ double buf[NTHREADS];

    for (auto ridx = row; ridx < NI; ridx += gridDim.x)
    {

        buf[tid] = 0.0;
        if (tid + scol < lcol)
        {
            buf[tid] = A[ridx * N + scol + tid] * V[tid + scol];
            // if (ridx == 7039) {
            //     printf("Thread %d (%d, %d) A=%E, V=%E, buf=%E\n", tid, ridx, tid + scol, A[ridx * N + tid + scol], V[tid + scol], buf[tid]);
            // }
        }

        int width = num_threads / 2;
        while (width != 0)
        {
            __syncthreads();
            if (tid < width)
            {
                buf[tid] += buf[tid + width];
            }
            width /= 2;
        }

        if (tid == 0)
        {
            atomicAdd(diag + ridx, buf[0]);
        }
    }
}

__global__ void set_diag(double *A, double *diag, int N)
{
    int didx = blockIdx.x * blockDim.x + threadIdx.x;
    if (didx < N)
    {
        double tmp = A[didx * N + didx];
        A[didx * N + didx] = 2 * tmp - diag[didx];
    }
}

__global__ void copy_pdisp(double *site_power, ELEMENT *element, const ELEMENT *metals, double *pdisp, int *atom_gpu_index, int N_atom,
                           const int num_metals)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int idx = tid; idx < N_atom; idx += total_threads)
    {
        bool metal = is_in_array_gpu(metals, element[atom_gpu_index[idx]], num_metals);
        if (!metal)
            site_power[atom_gpu_index[idx]] = -0.1 * pdisp[idx];
    }
}

template <int NTHREADS>
__global__ void get_imacro(const double *x, const double *m, double *imacro, double Vd, int N)
{

    int num_threads = blockDim.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int total_tid = bid * num_threads + tid;

    __shared__ double buf[NTHREADS];

    // total_tid >= 2 && total_tid < N

    buf[tid] = 0.0;
    double tmp = x[(N + 2) * (3) + (total_tid + 2)] * (m[total_tid + 2] - m[3]);
    if ((total_tid > 1 && total_tid < N) && ((tmp > 0 && Vd > 0) || (tmp < 0 && Vd < 0)))
    {
        buf[tid] = tmp;
    }

    int width = num_threads / 2;
    while (width != 0)
    {
        __syncthreads();
        if (tid < width)
        {
            buf[tid] += buf[tid + width];
        }
        width /= 2;
    }

    if (tid == 0)
    {
        atomicAdd(imacro, buf[0]);
    }
}

template <int NTHREADS>
__global__ void diagonal_sum(double *A, double *diag, int N)
{

    int num_threads = blockDim.x;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int block_id = blockIdx.x;

    int tid = threadIdx.x;

    __shared__ double buf[NTHREADS];

    for (auto idx = block_id; idx < N * blocks_per_row; idx += gridDim.x)
    {

        int ridx = idx / blocks_per_row;
        int scol = (idx % blocks_per_row) * num_threads;
        int lcol = min(N, scol + num_threads);

        buf[tid] = 0.0;
        if (tid + scol < lcol)
        {
            buf[tid] = A[ridx * N + scol + tid];
        }

        int width = num_threads / 2;
        while (width != 0)
        {
            __syncthreads();
            if (tid < width)
            {
                buf[tid] += buf[tid + width];
            }
            width /= 2;
        }

        if (tid == 0)
        {
            atomicAdd(diag + ridx, buf[0]);
        }
    }
}

// __global__ void create_K_sparse(double *X, int *gpu_X_ind,
//     const double *posx, const double *posy, const double *posz,
//     const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
//     const double *lattice, const bool pbc, const double high_G, const double low_G_1,
//     const double nn_dist, const int N, const int num_metals)
// {
//     int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
//     int num_threads_total = blockDim.x * gridDim.x;

//     // each thread gets a site
//     for (auto i = tid_total; i < (size_t) N; i += num_threads_total)
//     {
//         int neighbor_count = 0; // tracks the neighbors
//         for (auto j = 0; j < N; j++)
//         {
//             // this is now the connection between sites i and j
//             bool metal1 = is_in_array_gpu(metals, element[i], num_metals);
//             bool metal2 = is_in_array_gpu(metals, element[j], num_metals);
//             bool ischarged1 = site_charge[i] != 0;
//             bool ischarged2 = site_charge[j] != 0;
//             bool isVacancy1 = element[i] == VACANCY;
//             bool isVacancy2 = element[j] == VACANCY;
//             bool cvacancy1 = isVacancy1 && !ischarged1;
//             bool cvacancy2 = isVacancy2 && !ischarged2;
//             double dist = site_dist_gpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);

//             bool neighbor = false;
//             if (dist < nn_dist && i != j)
//                 neighbor = true;

//             if (i != j && neighbor)
//             {
//                 // log the coordinates of this pair
//                 gpu_k_xind[i*N + neighbor_count] = i;
//                 gpu_k_yind[i*N + neighbor_count] = j;

//                 if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
//                 {
//                     X[i*N + neighbor_count] = -high_G;
//                 }
//                 else
//                 {
//                     X[i*N + neighbor_count] = -low_G_1;
//                 }
                
//                 neighbor_count++;
//             }
//         }
//     }
// }

__global__ void create_K(
    double *X,
    const double *posx, const double *posy, const double *posz,
    const ELEMENT *metals, const ELEMENT *element, const int *site_charge,
    const double *lattice, const bool pbc, const double high_G, const double low_G_1,
    const double nn_dist, const int N, const int num_metals)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto idx = tid_total; idx < (size_t) N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        bool metal1 = is_in_array_gpu(metals, element[i], num_metals);
        bool metal2 = is_in_array_gpu(metals, element[j], num_metals);
        bool ischarged1 = site_charge[i] != 0;
        bool ischarged2 = site_charge[j] != 0;
        bool isVacancy1 = element[i] == VACANCY;
        bool isVacancy2 = element[j] == VACANCY;
        bool cvacancy1 = isVacancy1 && !ischarged1;
        bool cvacancy2 = isVacancy2 && !ischarged2;
        double dist = site_dist_gpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);

        bool neighbor = false;
        if (dist < nn_dist && i != j)
            neighbor = true;

        // direct terms:
        if (i != j && neighbor)
        {
            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                X[N * (i) + (j)] = -high_G;
            }
            else
            {
                X[N * (i) + (j)] = -low_G_1;
            }
        }
    }
}

__global__ void set_ineg(double *ineg, const double *x,
                         const double *m, double Vd, int N, const double *posx, const double *posy, const double *posz,
                         const int pbc, const double *lattice, const double nn_dist,
                         const double V0, const double *atom_potential, const double t_ox)
{
    // ineg is matrix N x N
    // x is matrix (N+2) x (N+2)
    // m is vector (N + 2)

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    for (auto idx = tid_total; idx < N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;

        double xdiff = (1e-10) * (posx[j] - posx[i]);
        double dist = site_dist_gpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
        bool neighbor = false;
        if (dist < nn_dist && i != j)
            neighbor = true;

        ineg[i * N + j] = 0.0;
        double ical = x[(N + 2) * (i + 2) + (j + 2)] * (m[i + 2] - m[j + 2]);
        if (neighbor && i != j && ical < 0 && Vd > 0)
        {
            ineg[i * N + j] = -ical;
        }
        else if (ical < 0 && Vd > 0 && xdiff < t_ox * V0 && xdiff > nn_dist && !neighbor)
        { // excluding Fowler Nordheim tunneling
            ineg[i * N + j] = -ical;
        }
    }
}

// iterates over every pair of sites, and does an operation based on the distance
// NOTE: There is an error in the case of block overflow in the input matrix! But this will never
// be triggered because the number of blocks can reach 2^31.
template <int NTHREADS>
__global__ void calculate_pairwise_interaction(const double* posx, const double* posy, const double*posz, 
                                               const double *lattice, const int pbc, 
                                               const int N, const double *sigma, const double *k, 
                                               const int *charge, double* potential){

    // Version with reduction, where every thread evaluates site-site interaction term
    int num_threads = blockDim.x;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int block_id = blockIdx.x;

    int row = block_id / blocks_per_row;
    int scol = (block_id % blocks_per_row) * num_threads;
    int lcol = min(N, scol + num_threads);

    int tid = threadIdx.x;

    __shared__ double buf[NTHREADS];
    double dist;
    int i, j;

    for (int ridx = row; ridx < N; ridx += gridDim.x) {

        buf[tid] = 0.0;
        if (tid + scol < lcol) {

            i = ridx;
            j = scol+tid;
            if (i != j && charge[j] != 0){
                dist = 1e-10 * site_dist_gpu(posx[i], posy[i], posz[i], 
                                             posx[j], posy[j], posz[j], 
                                             lattice[0], lattice[1], lattice[2], pbc);
                buf[tid] = v_solve_gpu(dist, charge[j], sigma, k);

            }
        }

        int width = num_threads / 2;
        while (width != 0) {
            __syncthreads();
            if (tid < width) {
                buf[tid] += buf[tid + width];
            }
            width /= 2;
        }

        if (tid == 0) {
            atomicAdd(potential + ridx, buf[0]);
        }
    
    }

    // // Version without reduction, where every thread evaluates a site:
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // int total_threads = blockDim.x * gridDim.x;
    // double V_temp = 0;
    // double dist;

    // for (int i = tid; i < N; i += total_threads) {
        
    //     // iterate over the neighbors
    //     for (int j = 0; j < N; j++){
    //         if (i != j && charge[j] != 0){

    //             dist = 1e-10 * site_dist_gpu(posx[i], posy[i], posz[i], 
    //                                          posx[j], posy[j], posz[j], 
    //                                          lattice[0], lattice[1], lattice[2], pbc);
    //             V_temp += v_solve_gpu(dist, charge[j], sigma, k);
                
    //         }
    //     }
    //     potential[tid] += V_temp;
    // }
}

__global__ void update_m(double *m, long minidx, int np2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int bid = blockIdx.x;

    if (idx < np2)
    {
        double minm = m[minidx];
        m[idx] += abs(minm);
    }
}

__global__ void create_X(
    double *X,
    const double *posx, const double *posy, const double *posz,
    const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_potential,
    const double *lattice, bool pbc, double high_G, double low_G_1,
    double nn_dist, double m_e, double V0, int num_source_inj, int num_ground_ext, int N, int num_metals, const double t_ox, const double Vd)
{

    int tid_total = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;

    int N_full = N + 2;

    // TODO: Does it make sense to restructure for N_full * N_full threads?
    for (auto idx = tid_total; idx < N * N; idx += num_threads_total)
    {
        int i = idx / N;
        int j = idx % N;
        // if (i < N && j < N) {

        bool metal1 = is_in_array_gpu(metals, element[i], num_metals);
        bool metal2 = is_in_array_gpu(metals, element[j], num_metals);

        // bool cvacancy1 = is_vacant[i] && is_charged[i];
        // bool cvacancy2 = is_vacant[j] && is_charged[i];
        // bool vacancy1 = is_vacant[i] && !is_charged[i];
        // bool vacancy2 = is_vacant[j] && !is_charged[i];

        bool ischarged1 = atom_charge[i] != 0;
        bool ischarged2 = atom_charge[j] != 0;

        bool isVacancy1 = element[i] == VACANCY;
        bool isVacancy2 = element[j] == VACANCY;

        bool cvacancy1 = isVacancy1 && !ischarged1;
        bool cvacancy2 = isVacancy2 && !ischarged2;

        bool vacancy1 = isVacancy1 && ischarged1;
        bool vacancy2 = isVacancy2 && ischarged2;
        double dist = site_dist_gpu(posx[i], posy[i], posz[i], posx[j], posy[j], posz[j], lattice[0], lattice[1], lattice[2], pbc);
        
        // dist = abs(dist);
        bool neighbor = false;
        if (dist < nn_dist && i != j)
            neighbor = true;

        // direct terms:
        if (i != j && neighbor)
        {
            if ((metal1 && metal2) || (cvacancy1 && cvacancy2))
            {
                X[N_full * (i + 2) + (j + 2)] = -high_G;
                // X[N_full * (j + 2) + (i + 2)] = -high_G;
            }
            else
            {
                X[N_full * (i + 2) + (j + 2)] = -low_G_1;
                // X[N_full * (j + 2) + (i + 2)] = -low_G_1;
            }
        }

        // tunneling terms
        if (i != j && !neighbor && j > i)
        { 
            bool V_V = (vacancy1 && vacancy2) || (vacancy2 && cvacancy1) || (vacancy1 && cvacancy2) || (cvacancy1 && cvacancy2);

            if (V_V)
            {
                double Vdiff = Vd;
                double xdiff = (1e-10) * (posx[j] - posx[i]); // potential accross the x-direction => if x_j < x_i then Vdiff < 0
                double b = Vdiff / t_ox;
                double a = 1e18; // zero prob
                if (abs(V0 / b - xdiff) < 1e-18 && xdiff > 0)
                {
                    a = 2.0 / 3.0 * sqrt(V0) * xdiff;
                }
                else if (xdiff < V0 / b && xdiff > nn_dist)
                {                                                                     // if Vdiff < 0 then lower prob
                    a = -2.0 / 3.0 * (1 / b) * (pow(V0 - b * xdiff, 1.5) - pow(V0, 1.5)); // always +
                }
                else if (xdiff > V0 / b < 0 && xdiff > 0)
                {
                    a = -2.0 / 3.0 * (1 / b) * (-1) * pow(V0, 3 / 2); // always +
                }
                double T = exp(-2 * sqrt((2 * m_e * eV_to_J) / (h_bar_sq)) * a);
                double G = 2 * 3.8612e-5 * T;
                X[N_full * (i + 2) + (j + 2)] = -G;
                X[N_full * (j + 2) + (i + 2)] = -G;
            }
        }

        // NOTE: Is there a data race here?
        // connect the source/ground nodes to the first/last contact layers
        __syncthreads();
        if (i < num_source_inj && j == 0)
        {
            X[1 * N_full + (i + 2)] = -high_G;
            X[(i + 2) * N_full + 1] = -high_G;
        }

        if (i > (N - num_ground_ext) && j == 0)
        {
            X[0 * N_full + (i + 2)] = -high_G;
            X[(i + 2) * N_full + 0] = -high_G;
        }

        __syncthreads();
        if (i == 0 && j == 0)
        {
            X[0 * N_full + 1] = -high_G;
            X[1 * N_full + 0] = -high_G;
        }
    }
}

__global__ void update_charge(const ELEMENT *element, 
                              int *charge, 
                              const int *neigh_idx, 
                              const int N, const int nn, 
                              const ELEMENT* metals, const int num_metals){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
    int Vnn = 0;

    // each thread gets a different site to evaluate
    for (int idx = tid; idx < N; idx += total_threads) {
        
        if (tid < N && element[tid] == VACANCY){
            charge[tid] = 2;

            // iterate over the neighbors
            for (int j = tid * nn; j < (tid + 1) * nn; ++j){
                if (element[neigh_idx[j]] == VACANCY){
                    Vnn++;
                }
                if (is_in_array_gpu(metals, element[neigh_idx[j]], num_metals)){
                    charge[tid] = 0;
                }
                if (Vnn >= 3){
                    charge[tid] = 0;
                }
            }
        }

        if (tid < N && element[tid] == OXYGEN_DEFECT){
            charge[tid] = -2;

            // iterate over the neighbors
            for (int j = tid * nn; j < (tid + 1) * nn; ++j){
                
                if (is_in_array_gpu(metals, element[neigh_idx[j]], num_metals)){
                    charge[tid] = 0;
                }
            }
        }
    }
}

//reduces the array into the value 
template <typename T, int NTHREADS>
__global__ void reduce(const T* array_to_reduce, T* value, const int N){

    __shared__ T buf[NTHREADS];
    
    int num_threads = blockDim.x;                           // number of threads in this block
    int blocks_per_row = (N-1)/num_threads + 1;             // number of blocks to fit in this array
    int block_id = blockIdx.x;                              // id of the block
    int tid = threadIdx.x;                                  // local thread id to this block
    int row = block_id / blocks_per_row;                    // which 'row' of the array to work on, rows are the overflow

    buf[tid] = 0;

    for (int ridx = row; ridx < N/(blocks_per_row*num_threads) + 1; ridx++){
    
        if (ridx*blocks_per_row*num_threads + block_id * num_threads + tid < N){
            buf[tid] = array_to_reduce[ridx*blocks_per_row*num_threads + block_id * num_threads + tid];
        }
       
        int width = num_threads / 2;

        while (width != 0){
            __syncthreads();
            if (tid < width){
                buf[tid] += buf[tid+width];
            }
            width /= 2;
        }

        if (tid == 0){
           atomicAdd(value, buf[0]);
        }
    }
}

// Kernel to extract COO struct data from a dense matrix
__global__ void extractCOOData(double* matrix, int N, COOElement* d_cooData, int* numNonZero) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < N; i += stride) {
        for (int j = 0; j < N; j++) {
            double value = matrix[i * N + j];
            if (value != 0.0) {
                int index = atomicAdd(numNonZero, 1);
                d_cooData[index].row = i;
                d_cooData[index].col = j;
                d_cooData[index].value = value;
            }
        }
    }
}

// Kernel to extract COO data to CSR format
__global__ void extractCOOToCSR(const COOElement* d_cooData, int numNonZero, int N, int* d_csrRowPtr, int* d_csrColIndices, double* d_csrValues) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < numNonZero; i += stride) {
        int row = d_cooData[i].row;
        d_csrValues[i] = d_cooData[i].value;
        d_csrColIndices[i] = d_cooData[i].col;

        // inclusive scan to get the row pointer
        if (i == 0 || row != d_cooData[i - 1].row) {
            d_csrRowPtr[row] = i;
        }
    }

    if (tid == 0) {
        // Set the last element to numNonZero
        d_csrRowPtr[N] = numNonZero;
    }
}

//called by a single gpu-thread
__global__ void update_temp_global(double *P_tot, double* T_bg, const double a_coeff, const double b_coeff, const double number_steps, const double C_thermal, const double small_step)
{
    double c_coeff = b_coeff + *P_tot/C_thermal * small_step;
    double T_intermediate = *T_bg;
    int step = number_steps;
    *T_bg = c_coeff*(1.0-pow(a_coeff, (double) step)) / (1.0-a_coeff) + pow(a_coeff, (double) step)* T_intermediate;
}

__global__ void build_event_list(const int N, const int nn, const int *neigh_idx, 
                                 const int *layer, const double *lattice, const int pbc, 
                                 const double *T_bg, const double *freq, const double *sigma, const double *k, 
                                 const double *posx, const double *posy, const double *posz,
                                 const double *potential, const double *temperature,
                                 const ELEMENT *element, const int *charge, EVENTTYPE *event_type, double *event_prob)
{
    int total_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int idx = total_tid; idx < N * nn; idx += total_threads) {

        EVENTTYPE event_type_ = NULL_EVENT;
        double P = 0.0;

        int i = static_cast<int>(floorf(idx / nn));
        int j = neigh_idx[idx];

        // condition for neighbor existing
        if (j >= 0 && j < N) {
            double dist = 1e-10 * site_dist_gpu(posx[i], posy[i], posz[i], 
                                                posx[j], posy[j], posz[j], 
                                                lattice[0], lattice[1], lattice[2], pbc);

            // Generation
            if (element[i] == DEFECT && element[j] == O_EL)
            {

                double E = 2 * (potential[i] - potential[j]);
                double zero_field_energy = E_gen_const[layer[j]]; 
                event_type_ = VACANCY_GENERATION;
                double Ekin = kB * (temperature[j] - temperature[i]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Recombination
            else if (element[i] == OXYGEN_DEFECT && element[j] == VACANCY) 
            {
                int charge_abs = 2;
                double self_int_V = v_solve_gpu(dist, charge_abs, sigma, k);

                int charge_state = charge[i] - charge[j];
                double E = charge_state * (potential[i] - potential[j] + (charge_state / 2) * self_int_V);
                double zero_field_energy = E_rec_const[layer[j]];

                event_type_ = VACANCY_RECOMBINATION;
                double Ekin = kB * (temperature[i] - temperature[j]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Vacancy diffusion
            if (element[i] == VACANCY && element[j] == O_EL)
            {

                double self_int_V = 0.0;
                if (charge[i] != 0)
                {
                    double self_int_V = v_solve_gpu(dist, charge[i], sigma, k);
                }

                event_type_ = VACANCY_DIFFUSION;
                double E = (charge[i] - charge[j]) * (potential[i] - potential[j] + self_int_V);
                double zero_field_energy = E_Vdiff_const[layer[j]];  
                double Ekin = kB * (temperature[j] - temperature[i]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Ion diffusion
            else if (element[i] == OXYGEN_DEFECT && element[j] == DEFECT)
            {
                int charge_abs = 2;
                double self_int_V = 0.0;
                if (charge[i] != 0)
                {                    
                    double self_int_V = v_solve_gpu(dist, charge_abs, sigma, k);
                }

                double E = (charge[i] - charge[j]) * (potential[i] - potential[j] - self_int_V);
                double zero_field_energy = E_Odiff_const[layer[j]];

                event_type_ = ION_DIFFUSION;
                double Ekin = kB * (temperature[i] - temperature[j]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }
        }
        event_type[idx] = event_type_;
        event_prob[idx] = P;
    }
}

// ********************************************************
// ****************** KERNEL UNIT TESTS *******************
// ********************************************************

// unit test for reduce kernel, checks correctness for large arrays
void test_reduce()
{
    int N = 70000;

    int num_threads = 512;
    int num_blocks = (N - 1) / num_threads + 1;
    // num_blocks = min(65535, num_blocks);

    double *gpu_test_array;
    double *gpu_test_sum;
    double t_test = 0.0;
    std::vector<double> test_array(N, 1.0);

    gpuErrchk( hipMalloc((void**)&gpu_test_array, N * sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&gpu_test_sum, 1 * sizeof(double)) );
    gpuErrchk( hipMemcpy(gpu_test_array, test_array.data(), N * sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(gpu_test_sum, &t_test, 1 * sizeof(double), hipMemcpyHostToDevice) );

    reduce<double, NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS*sizeof(double)>>>(gpu_test_array, gpu_test_sum, N);
    gpuErrchk( hipGetLastError() );

    gpuErrchk( hipMemcpy(&t_test, gpu_test_sum, 1 * sizeof(double), hipMemcpyDeviceToHost));
    assert(t_test == 70000.0);
    std::cout << "--> Ran test for kernel reduce()\n";
}                

// ********************************************************
// *************** WRAPPER FUNCTIONS **********************
// ********************************************************

void update_charge_gpu(ELEMENT *site_element, 
                       int *site_charge,
                       int *neigh_idx, int N, int nn, 
                       const ELEMENT *metals, const int num_metals){

    int num_threads = 512;
    int num_blocks = (N * nn - 1) / num_threads + 1;

    update_charge<<<num_blocks, num_threads>>>(site_element, site_charge, neigh_idx, N, nn, metals, num_metals);
}

void update_temperatureglobal_gpu(const double *site_power, double *T_bg, const int N, const double a_coeff, const double b_coeff, const double number_steps, const double C_thermal, const double small_step){

    int num_threads = 512;
    int num_blocks = (N - 1) / num_threads + 1;

    double *P_tot;
    gpuErrchk( hipMalloc((void**)&P_tot, 1 * sizeof(double)) );
    gpuErrchk( hipMemset(P_tot, 0, 1 * sizeof(double)) );

    //collect site_power
    reduce<double, NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS*sizeof(double)>>>(site_power, P_tot, N);

    //update the temperature
    update_temp_global<<<1, 1>>>(P_tot, T_bg, a_coeff, b_coeff, number_steps, C_thermal, small_step);

    // double p_test = 0.0;
    // gpuErrchk( hipMemcpy(&p_test, P_tot, 1 * sizeof(double), hipMemcpyDeviceToHost));
    // std::cout << "power: " << p_test << "\n";
    // double t_test = 0.0;
    // gpuErrchk( hipMemcpy(&t_test, T_bg, 1 * sizeof(double), hipMemcpyDeviceToHost));
    // std::cout << "temperature: " << t_test << "\n";
    hipFree(P_tot);
}

// check that sparse and dense versions are the same
void check_sparse_dense_match(int m, int nnz, double *dense_matrix, int* d_csrRowPtr, int* d_csrColInd, double* d_csrVal){
    
    double *h_D = (double *)calloc(m*m, sizeof(double));
    double *h_D_csr = (double *)calloc(nnz, sizeof(double));
    int *h_pointers = (int *)calloc((m + 1), sizeof(int));
    int *h_inds = (int *)calloc(nnz, sizeof(int));

    gpuErrchk( hipMemcpy(h_D, dense_matrix, m*m * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_D_csr, d_csrVal, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_pointers, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_inds, d_csrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int nnz_count = 0;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < m; col++) {
            int i = row * m + col;  // Linear index in dense matrix
            // Check if the element in the dense matrix is non-zero
            if (h_D[i] != 0) {
                // Compare the row and column indices
                if (h_D[i] != h_D_csr[nnz_count] || col != h_inds[nnz_count]) {
                    std::cout << "Mismatch found at (row, col) = (" << row << ", " << col << ")\n";
                }
                nnz_count++;
            }
        }
    }
}

// dump sparse matrix into a file
void dump_csr_matrix_txt(int m, int nnz, int* d_csrRowPtr, int* d_csrColIndices, double* d_csrValues, int kmc_step_count){

    // Copy matrix back to host memory
    double *h_csrValues = (double *)calloc(nnz, sizeof(double));
    int *h_csrRowPtr = (int *)calloc((m + 1), sizeof(int));
    int *h_csrColIndices = (int *)calloc(nnz, sizeof(int));
    gpuErrchk( hipMemcpy(h_csrValues, d_csrValues, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_csrRowPtr, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_csrColIndices, d_csrColIndices, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    // print to file, tagged with the kmc step number
    std::ofstream fout_val("/scratch/sem23f28/manasa_kmc/csrValues_step#" + std::to_string(kmc_step_count) + ".txt");
    for(int i = 0; i < nnz; i++){
        fout_val << h_csrValues[i] << " "; 
    }
    std::ofstream fout_row("/scratch/sem23f28/manasa_kmc/csrRowPtr_step#" + std::to_string(kmc_step_count) + ".txt");
    for(int i = 0; i < (m + 1); i++){
        fout_row << h_csrRowPtr[i] << " "; 
    }
    std::ofstream fout_col("/scratch/sem23f28/manasa_kmc/csrColIndices_step#" + std::to_string(kmc_step_count) + ".txt");
    for(int i = 0; i < nnz; i++){
        fout_col << h_csrColIndices[i] << " "; 
    }

    free(h_csrValues);
    free(h_csrRowPtr);
    free(h_csrColIndices);
}

// Solution of A*x = y using cusolver in host pointer mode
void sparse_system_solve(hipsolverSpHandle_t handle, int* d_csrRowPtr, int* d_csrColInd, double* d_csrVal,
                         int nnz, int m, double *d_x, double *d_y){

    // Ref: https://stackoverflow.com/questions/31840341/solving-general-sparse-linear-systems-in-cuda

    // cusolverSpDcsrlsvlu only supports the host path
    int *h_A_RowIndices = (int *)malloc((m + 1) * sizeof(int));
    int *h_A_ColIndices = (int *)malloc(nnz * sizeof(int));
    double *h_A_Val = (double *)malloc(nnz * sizeof(double));
    double *h_x = (double *)malloc(m * sizeof(double));
    double *h_y = (double *)malloc(m * sizeof(double));
    gpuErrchk( hipMemcpy(h_A_RowIndices, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_A_ColIndices, d_csrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_A_Val, d_csrVal, nnz * sizeof(double), hipMemcpyDeviceToHost) );   
    gpuErrchk( hipMemcpy(h_x, d_x, m * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_y, d_y, m * sizeof(double), hipMemcpyDeviceToHost) );

    hipsparseMatDescr_t matDescrA;
    hipsparseCreateMatDescr(&matDescrA);
    hipsparseSetMatType(matDescrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matDescrA, HIPSPARSE_INDEX_BASE_ZERO);

    int singularity;
    double tol = 0.00000001;

    // Solve with LU
    // CheckCusolverDnError( cusolverSpDcsrlsvluHost(handle, m, nnz, matDescrA, h_A_Val, h_A_RowIndices, 
    //                       h_A_ColIndices, h_y, tol, 0, h_x, &singularity) );
    
    // Solve with QR
    // CheckCusolverDnError( cusolverSpDcsrlsvqrHost(handle, m, nnz, matDescrA, h_A_Val, h_A_RowIndices, 
    //                       h_A_ColIndices, h_y, tol, 1, h_x, &singularity) );

    // Solve with Cholesky
    CheckCusolverDnError( hipsolverSpDcsrlsvcholHost(handle, m, nnz, matDescrA, h_A_Val, h_A_RowIndices,
                          h_A_ColIndices, h_y, tol, 1, h_x, &singularity) );

    gpuErrchk( hipDeviceSynchronize() );
    if (singularity != -1){
        std::cout << "In sparse_system_solve: Matrix has a singularity at : " << singularity << "\n";
    }

    // copy back the solution vector:
    gpuErrchk( hipMemcpy(d_x, h_x, m * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverSpDestroy(handle);
    hipsparseDestroyMatDescr(matDescrA);
    free(h_A_RowIndices);
    free(h_A_ColIndices);
    free(h_A_Val);
    free(h_x);
    free(h_y);
}

// Iterative sparse linear solver using CG steps
void sparse_system_solve_iterative(hipblasHandle_t handle_cublas, hipsparseHandle_t handle, 
               int* d_csrRowPtr, int* d_csrColInd, double* d_csrVal,
               int nnz, int m, double *d_x, double *d_y){

    // follows the manual version: https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-generic-apis
    // A is m x m sparse matrix represented by CSR format,
    // - (d_csrRowPtr, d_csrColInd, d_csrVal) is CSR of A in gpu memory,
    // - d_x is right hand side vector in gpu memory,
    // - d_y is solution vector in gpu memory.
    // - d_z is intermediate result on gpu memory.

    // Decides whether to find the initial guess through Incomplete LU factorization
    // + triangular solve of the matrix A !!CURRENTLY LEADS TO DIVERGENCE OF THE CG STEPS!!
    bool preconditioned_guess = 0; 

    // Sets the initial guess for the solution vector to zero
    bool zero_guess = 1;

    // Error tolerance for the norm of the residual in the CG steps
    double tol = 1e-1;

    // constants:
    const double one = 1.0;
    const double n_one = -1.0;
    const double zero = 0.0;
    double *one_d, *n_one_d, *zero_d;
    gpuErrchk( hipMalloc((void**)&one_d, sizeof(double)) );
    gpuErrchk( hipMemcpy(one_d, &one, sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMalloc((void**)&n_one_d, sizeof(double)) );
    gpuErrchk( hipMemcpy(n_one_d, &n_one, sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMalloc((void**)&zero_d, sizeof(double)) );
    gpuErrchk( hipMemcpy(zero_d, &zero, sizeof(double), hipMemcpyHostToDevice) );
    hipsparseStatus_t status;

    // ************************************
    // ** Precondioner and Initial Guess **

    // Use solution of Incomplete LU as initial guess
    if (preconditioned_guess) 
    {
        // d_z is the intermediate result on device memory
        double *d_z;
        gpuErrchk( hipMalloc((void **)&d_z, m * sizeof(double)) ); 

        hipsparseMatDescr_t descr_M = 0;
        hipsparseMatDescr_t descr_L = 0;
        hipsparseMatDescr_t descr_U = 0;
        csrilu02Info_t info_M  = 0;
        csrsv2Info_t  info_L  = 0;
        csrsv2Info_t  info_U  = 0;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer = 0;
        int structural_zero;
        int numerical_zero;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL; 
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        // Create descriptors
        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);      // - matrix L is lower triangular
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);       // - matrix L has unit diagonal
        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);      // - matrix U is upper triangular
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);   // - matrix U has non-unit diagonal

        // Create Info structures for ilu and sv2
        hipsparseCreateCsrilu02Info(&info_M);
        hipsparseCreateCsrsv2Info(&info_L);
        hipsparseCreateCsrsv2Info(&info_U);
        gpuErrchk( hipDeviceSynchronize() );

        // Query how much memory used in csrilu02 and csrsv2, and allocate the buffer
        status = hipsparseDcsrilu02_bufferSize(handle, m, nnz,
            descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, &pBufferSize_M);
        status = hipsparseDcsrsv2_bufferSize(handle, trans_L, m, nnz,
            descr_L, d_csrVal, d_csrRowPtr, d_csrColInd, info_L, &pBufferSize_L);
        status = hipsparseDcsrsv2_bufferSize(handle, trans_U, m, nnz,
            descr_U, d_csrVal, d_csrRowPtr, d_csrColInd, info_U, &pBufferSize_U);
        gpuErrchk( hipDeviceSynchronize() );
        pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
        gpuErrchk( hipMalloc((void**)&pBuffer, sizeof(double) * pBufferSize) );

        //         perform analysis of incomplete LU on M
        //         perform analysis of triangular solve on L
        //         perform analysis of triangular solve on U
        // The lower(upper) triangular part of M has the same sparsity pattern as L(U) for ILU
        // can do analysis of csrilu0 and csrsv2 simultaneously.
        status = hipsparseDcsrilu02_analysis(handle, m, nnz, descr_M,
                                            d_csrVal, d_csrRowPtr, d_csrColInd, info_M,
                                            policy_M, pBuffer);
        gpuErrchk( hipDeviceSynchronize() );

        status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
            printf("M(%d,%d) is zero\n", structural_zero, structural_zero);
        }

        hipsparseDcsrsv2_analysis(handle, trans_L, m, nnz, descr_L,
            d_csrVal, d_csrRowPtr, d_csrColInd,
            info_L, policy_L, pBuffer);
        gpuErrchk( hipDeviceSynchronize() );

        hipsparseDcsrsv2_analysis(handle, trans_U, m, nnz, descr_U,
            d_csrVal, d_csrRowPtr, d_csrColInd,
            info_U, policy_U, pBuffer);
        gpuErrchk( hipDeviceSynchronize() );

        // Incomplete LU Factorization of M = L * U
        status = hipsparseDcsrilu02(handle, m, nnz, descr_M,
            d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer);
        gpuErrchk( hipDeviceSynchronize() );

        status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &numerical_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
            printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
        }

        // Triangular Solve steps ( A*y = x --> L*U*y = x --> U*y = z )
        
        // solve L*z = x
        status = hipsparseDcsrsv2_solve(handle, trans_L, m, nnz, one_d, descr_L, 
                                    d_csrVal, d_csrRowPtr, d_csrColInd, info_L,
                                    d_x, d_z, policy_L, pBuffer); 
        gpuErrchk( hipDeviceSynchronize() );

        // solve U*y = z
        status = hipsparseDcsrsv2_solve(handle, trans_U, m, nnz, one_d, descr_U, 
                                       d_csrVal, d_csrRowPtr, d_csrColInd, info_U,
                                       d_z, d_y, policy_U, pBuffer);
        gpuErrchk( hipDeviceSynchronize() );

        hipFree(d_z);
        hipFree(pBuffer);
        hipsparseDestroyMatDescr(descr_M);
        hipsparseDestroyMatDescr(descr_L);
        hipsparseDestroyMatDescr(descr_U);
        hipsparseDestroyCsrilu02Info(info_M);
        hipsparseDestroyCsrsv2Info(info_L);
        hipsparseDestroyCsrsv2Info(info_U);
    }
    if (zero_guess)
    {
        // Set the initial guess for the solution vector to zero
        gpuErrchk( hipMemset(d_y, 0, m * sizeof(double)) ); 
        gpuErrchk( hipDeviceSynchronize() );
    }

    // *******************************
    // ** Iterative refinement loop **

    // initialize variables for the residual calculation
    double h_norm;
    double *d_r, *d_p, *d_temp;
    gpuErrchk( hipMalloc((void**)&d_r, m * sizeof(double)) ); 
    gpuErrchk( hipMalloc((void**)&d_p, m * sizeof(double)) ); 
    gpuErrchk( hipMalloc((void**)&d_temp, m * sizeof(double)) ); 
    gpuErrchk( hipDeviceSynchronize() );

    // for SpMV:
    // - d_x is right hand side vector
    // - d_y is solution vector
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY, vecR, vecP, vectemp;
    hipsparseCreateCsr(&matA, m, m, nnz, d_csrRowPtr, d_csrColInd, d_csrVal, 
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnVec(&vecX, m, d_x, HIP_R_64F);
    hipsparseCreateDnVec(&vecY, m, d_y, HIP_R_64F);
    hipsparseCreateDnVec(&vecR, m, d_r, HIP_R_64F);
    hipsparseCreateDnVec(&vecP, m, d_p, HIP_R_64F);
    hipsparseCreateDnVec(&vectemp, m, d_temp, HIP_R_64F);

    // // Buffer for SpMV:
    size_t MVBufferSize;
    void *MVBuffer = 0;
    status = hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, matA, 
                          vecY, zero_d, vecR, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &MVBufferSize);
    gpuErrchk( hipMalloc((void**)&MVBuffer, sizeof(double) * MVBufferSize) );
    
    // Initialize the residual and conjugate vectors
    // r = A*y - x & p = -r
    status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, matA, 
                          vecY, zero_d, vecR, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, MVBuffer);         // r = A*y
    gpuErrchk( hipDeviceSynchronize() );
    CheckCublasError( hipblasDaxpy(handle_cublas, m, &n_one, d_x, 1, d_r, 1) );                          // r = -x + r
    gpuErrchk( hipDeviceSynchronize() );
    CheckCublasError(hipblasDcopy(handle_cublas, m, d_r, 1, d_p, 1));                                    // p = r
    gpuErrchk( hipDeviceSynchronize() );
    CheckCublasError(hipblasDscal(handle_cublas, m, &n_one, d_p, 1));                                    // p = -p
    gpuErrchk( hipDeviceSynchronize() );

    // calculate the error (norm of the residual)
    CheckCublasError( hipblasDnrm2(handle_cublas, m, d_r, 1, &h_norm) );
    gpuErrchk( hipDeviceSynchronize() );
    
    // Conjugate Gradient steps
    int counter = 0;
    double t, tnew, alpha, beta, alpha_temp;
    while (h_norm > tol){

        // alpha = rT * r / (pT * A * p)
        CheckCublasError( hipblasDdot (handle_cublas, m, d_r, 1, d_r, 1, &t) );                         // t = rT * r
        gpuErrchk( hipDeviceSynchronize() );

        status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, matA, 
                              vecP, zero_d, vectemp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, MVBuffer); // temp = A*p
        gpuErrchk( hipDeviceSynchronize() );

        CheckCublasError( hipblasDdot (handle_cublas, m, d_p, 1, d_temp, 1, &alpha_temp) );             // alpha = pT*temp = pT*A*p
        gpuErrchk( hipDeviceSynchronize() );

        alpha = t / alpha_temp; 

        // y = y + alpha * p
        CheckCublasError(hipblasDaxpy(handle_cublas, m, &alpha, d_p, 1, d_y, 1));                       // y = y + alpha * p
        gpuErrchk( hipDeviceSynchronize() );

        // r = r + alpha * A * p 
        CheckCublasError(hipblasDaxpy(handle_cublas, m, &alpha, d_temp, 1, d_r, 1));                    // r = r + alpha * temp
        gpuErrchk( hipDeviceSynchronize() );

        // beta = (rT * r) / t
        CheckCublasError( hipblasDdot (handle_cublas, m, d_r, 1, d_r, 1, &tnew) );                       // tnew = rT * r
        gpuErrchk( hipDeviceSynchronize() );
        beta = tnew / t;

        // p = -r + beta * p
        CheckCublasError(hipblasDscal(handle_cublas, m, &beta, d_p, 1));                                  // p = p * beta
        gpuErrchk( hipDeviceSynchronize() );
        CheckCublasError(hipblasDaxpy(handle_cublas, m, &n_one, d_r, 1, d_p, 1));                         // p = p - r
        gpuErrchk( hipDeviceSynchronize() );

        // calculate the error (norm of the residual)
        CheckCublasError( hipblasDnrm2(handle_cublas, m, d_r, 1, &h_norm) );
        gpuErrchk( hipDeviceSynchronize() );
        std::cout << h_norm << "\n";

        counter++;
        if (counter > 1000){
            std::cout << "WARNING: might be stuck in diverging CG iterations, check the residual!\n";
        }
    }

    // // check solution vector
    // double *copy_back = (double *)calloc(m, sizeof(double));
    // gpuErrchk( hipMemcpy(copy_back, d_y, m * sizeof(double), hipMemcpyDeviceToHost) );
    // for (int i = 0; i < m; i++){
    //     std::cout << copy_back[i] << " ";
    // }
    
}


template <typename T>
void writeArrayToTxtFile(const T* array, int numElements, const std::string& filename) {
    std::ofstream file(filename);
    if (file.is_open()) {
        for(int i = 0; i < numElements; i++){
            file << array[i] << " "; 
        }
        file.close();
        std::cout << "Array data written to file: " << filename << std::endl;
    } else {
        std::cerr << "Unable to open the file for writing." << std::endl;
    }
}

template <typename T>
void writeArrayToBinFile(T* array, int numElements, const std::string& filename) {
    std::ofstream file(filename, std::ios::binary);
    if (file.is_open()) {
        file.write(reinterpret_cast<char*>(array), numElements*sizeof(T));
        file.close();
        std::cout << "Array data written to file: " << filename << std::endl;
    } else {
        std::cerr << "Unable to open the file for writing." << std::endl;
    }
}


// TODO maybe seperate cuda/MPI such that 
// every .cpp is compiled with mpic++
// and .cu with nvcc
void background_potential_gpu_sparse(hipblasHandle_t handle_cublas, hipsolverHandle_t handle_cusolver, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double nn_dist,
                              const int num_metals, int kmc_step_count)
{
    // prepare contact potentials and input size
    int N_interface = N - (N_left_tot + N_right_tot);

    double *VL, *VR;
    gpuErrchk( hipMalloc((void **)&VL, N_left_tot * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&VR, N_right_tot * sizeof(double)) );
    thrust::device_ptr<double> VL_ptr = thrust::device_pointer_cast(VL);
    thrust::fill(VL_ptr, VL_ptr + N_left_tot, -Vd/2);
    thrust::device_ptr<double> VR_ptr = thrust::device_pointer_cast(VR);
    thrust::fill(VR_ptr, VR_ptr + N_right_tot, Vd/2);

    // ************************************************************
    // 0. Create the dense matrix K (Conductance)

    int num_threads = 512;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int num_blocks = blocks_per_row;

    // off-diagonal elements
    double *gpu_k;
    gpuErrchk( hipMalloc((void **)&gpu_k, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipMemset(gpu_k, 0, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

   create_K<<<num_blocks, num_threads>>>(
        gpu_k, gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        gpubuf.lattice, pbc, d_high_G, d_low_G,
        nn_dist, N, num_metals);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // diagonal elements
    double *gpu_diag;
    gpuErrchk( hipMalloc((void **)&gpu_diag, N * sizeof(double)) );
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );
    diagonal_sum<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    num_blocks = (N - 1) / num_threads + 1;
    set_diag<<<num_blocks, num_threads>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    blocks_per_row = (N_left_tot - 1) / num_threads + 1;
    num_blocks = blocks_per_row * N_interface;
    diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(&gpu_k[N_left_tot * N], gpu_diag, VL, N, N_interface, N_left_tot);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    blocks_per_row = (N_right_tot - 1) / num_threads + 1;
    num_blocks = blocks_per_row * N_interface;
    diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(&gpu_k[N_left_tot * N + N - N_right_tot], gpu_diag, VR, N, N_interface, N_right_tot);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Create the RHS vector gpu_k_sub
    double *gpu_k_sub;
    gpuErrchk( hipMalloc((void **)&gpu_k_sub, N_interface * sizeof(double)) ); 
    gpuErrchk( hipMemset(gpu_k_sub, 0, N_interface * sizeof(double)) );
    num_blocks = (N_interface - 1) / num_threads + 1;
    set_diag_K<<<blocks_per_row, num_threads>>>(gpu_k_sub, gpu_diag, N_interface);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipFree(gpu_diag);

    // Create D matrix for the solver
    double* gpu_D = gpu_k + (N_left_tot * N) + N_left_tot;

    // ************************************************************
    // 1. Convert dense D to CSR:
    // TODO: this could include some errors

    hipsolverSpHandle_t handle;
    hipsolverSpCreate(&handle);
    hipsparseStatus_t status;
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);
    hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_DEVICE);
    hipsparseDirection_t direction = HIPSPARSE_DIRECTION_ROW;
    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    
    int nnz = 0;
    int *d_nnzPerRow;
    int *d_numNonZero;
    gpuErrchk( hipMalloc((void**)&d_numNonZero, sizeof(int)) );
    gpuErrchk( hipMemcpy(d_numNonZero, &nnz, sizeof(int), hipMemcpyHostToDevice) );
    gpuErrchk( hipMalloc((void **)&d_nnzPerRow, N_interface * sizeof(int)) ); 

    // get number of non zeros per row
    status = hipsparseDnnz(cusparseHandle, direction, N_interface,
                          N_interface, descr, gpu_D, N, d_nnzPerRow, d_numNonZero);
    gpuErrchk( hipDeviceSynchronize() );
    hipMemcpy(&nnz, d_numNonZero, sizeof(int), hipMemcpyDeviceToHost);

    // prepare sparse representation arrays
    int* d_csrRowPtr;
    int* d_csrColIndices;
    double* d_csrValues;
    hipMalloc((void**)&d_csrRowPtr, (N_interface + 1) * sizeof(int));
    thrust::device_ptr<int> thrust_nnzPerRow(d_nnzPerRow);
    thrust::inclusive_scan(thrust_nnzPerRow, thrust_nnzPerRow + N_interface, thrust::device_pointer_cast(d_csrRowPtr + 1));
    hipMalloc((void**)&d_csrColIndices, nnz * sizeof(int));
    hipMalloc((void**)&d_csrValues, nnz * sizeof(double));
    hipMemset(d_csrValues, 0, nnz * sizeof(double));
    gpuErrchk( hipDeviceSynchronize() );


    // std::string filename = "/scratch/sem23f28/manasa_kmc/KD2S_" + std::to_string(kmc_step_count) + ".txt";
    // double* cpu_k = (double*)malloc(N * N * sizeof(double));
    // hipMemcpy(cpu_k, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost);
    // writeArrayToTxtFile<double>(cpu_k, N * N, filename);

    std::string filename = "/usr/scratch/mont-fort17/almaeder/manasa_kmc_matrices/K_" + std::to_string(kmc_step_count) + ".bin";
    double* cpu_k = (double*)malloc(N * N * sizeof(double));
    hipMemcpy(cpu_k, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost);
    writeArrayToBinFile<double>(cpu_k, N * N, filename);



    // fill in sparse representation
    status = hipsparseDdense2csr(cusparseHandle, N_interface, N_interface,
                                descr, gpu_D, N, d_nnzPerRow,
                                d_csrValues, d_csrRowPtr, d_csrColIndices);
    gpuErrchk( hipDeviceSynchronize() );
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("CUSPARSE dense-to-sparse conversion failed!\n");
    }

    /// Printing matrices @Alex Maeder ///
    dump_csr_matrix_txt(N_interface, nnz, d_csrRowPtr, d_csrColIndices, d_csrValues, kmc_step_count);
    /// Printing matrices @Alex Maeder ///

    // ************************************************************
    // 2. Solve system of linear equations 

    double *v_soln;
    gpuErrchk( hipMalloc((void **)&v_soln, N_interface * sizeof(double)) ); 

    // option 1 - Iterative manual, using device pointers ( need to debug solution )
    // set_potential<<<num_blocks, num_threads>>>(v_soln, gpubuf.site_potential + N_left_tot, N_interface); // initial guess is the current potential
    // sparse_system_solve_iterative(handle_cublas, cusparseHandle, d_csrRowPtr, d_csrColIndices, d_csrValues,
    //                               nnz, N_interface, gpu_k_sub, v_soln);

    // option 2 - Using CuSolver with host pointers
    sparse_system_solve(handle, d_csrRowPtr, d_csrColIndices, d_csrValues,
                        nnz, N_interface, v_soln, gpu_k_sub);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // ************************************************************
    // 3. Copy solution back to site_potential

    num_blocks = (N_interface - 1) / num_threads + 1;
    set_potential<<<num_blocks, num_threads>>>(gpubuf.site_potential + N_left_tot, v_soln, N_interface);
    gpuErrchk( hipPeekAtLastError() ); 
    gpuErrchk( hipDeviceSynchronize() ); 

    gpuErrchk( hipMemcpy(gpubuf.site_potential, VL, N_left_tot * sizeof(double), hipMemcpyDeviceToDevice) );
    gpuErrchk( hipMemcpy(gpubuf.site_potential + N_left_tot + N_interface, VR, N_right_tot * sizeof(double), hipMemcpyDeviceToDevice) );

    hipsparseDestroy(cusparseHandle);
    hipFree(gpu_k);
    hipFree(gpu_k_sub);
    hipFree(VL);
    hipFree(VR);
    hipFree(v_soln);
    
}




void background_potential_gpu(hipsolverHandle_t handle, const GPUBuffers &gpubuf, const int N, const int N_left_tot, const int N_right_tot,
                              const double Vd, const int pbc, const double d_high_G, const double d_low_G, const double nn_dist,
                              const int num_metals, int kmc_step_count)
{

    int N_interface = N - (N_left_tot + N_right_tot);

    double *VL, *VR;
    gpuErrchk( hipMalloc((void **)&VL, N_left_tot * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&VR, N_right_tot * sizeof(double)) );

    double *gpu_k;
    double *gpu_diag;
    gpuErrchk( hipMalloc((void **)&gpu_k, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&gpu_diag, N * sizeof(double)) );
    gpuErrchk( hipMemset(gpu_k, 0, (size_t) N * N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    // prepare contact potentials
    thrust::device_ptr<double> VL_ptr = thrust::device_pointer_cast(VL);
    thrust::fill(VL_ptr, VL_ptr + N_left_tot, -Vd/2);
    thrust::device_ptr<double> VR_ptr = thrust::device_pointer_cast(VR);
    thrust::fill(VR_ptr, VR_ptr + N_right_tot, Vd/2);

    //  BUILDING THE CONDUCTIVITY MATRIX
    int num_threads = 512;
    int blocks_per_row = (N - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * N;

    // compute the off-diagonal elements of K
    create_K<<<num_blocks, num_threads>>>(
        gpu_k, gpubuf.site_x, gpubuf.site_y, gpubuf.site_z,
        gpubuf.metal_types, gpubuf.site_element, gpubuf.site_charge,
        gpubuf.lattice, pbc, d_high_G, d_low_G,
        nn_dist, N, num_metals);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Update the diagonal of K
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );
    diagonal_sum<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    num_blocks = (N - 1) / num_threads + 1;
    set_diag<<<num_blocks, num_threads>>>(gpu_k, gpu_diag, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemset(gpu_diag, 0, N * sizeof(double)) );
    gpuErrchk( hipDeviceSynchronize() );

    blocks_per_row = (N_left_tot - 1) / num_threads + 1;
    num_blocks = blocks_per_row * N_interface;
    diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(&gpu_k[N_left_tot * N], gpu_diag, VL, N, N_interface, N_left_tot);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    blocks_per_row = (N_right_tot - 1) / num_threads + 1;
    num_blocks = blocks_per_row * N_interface;
    diagonal_sum_K<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(&gpu_k[N_left_tot * N + N - N_right_tot], gpu_diag, VR, N, N_interface, N_right_tot);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //  SOLVING FOR THE NEGATIVE INTERNAL POTENTIALS (KSUB)
    double *gpu_k_sub;
    gpuErrchk( hipMalloc((void **)&gpu_k_sub, N_interface * sizeof(double)) ); 
    gpuErrchk( hipMemset(gpu_k_sub, 0, N_interface * sizeof(double)) );
    num_blocks = (N_interface - 1) / num_threads + 1;
    set_diag_K<<<blocks_per_row, num_threads>>>(gpu_k_sub, gpu_diag, N_interface);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipFree(gpu_diag);

    // ** Solve Ax=B through LU factorization **

    int lwork = 0;              /* size of workspace */
    double *gpu_work = nullptr; /* device workspace for getrf */
    int *gpu_info = nullptr;    /* error info */
    int *gpu_ipiv; // int info;
    gpuErrchk( hipMalloc((void **)&gpu_ipiv, N_interface * sizeof(int)) ); 
    gpuErrchk( hipMalloc((void **)(&gpu_info), sizeof(int)) );

    printf("N_interface: %i \n", N_interface);
    printf("N_left_tot: %i \n", N_left_tot);
    printf("N_right_tot: %i \n", N_right_tot);
    printf("N: %i \n", N);

    // std::string filename_A = "/scratch/sem23f28/manasa_kmc/K_" + std::to_string(kmc_step_count) + ".txt";
    // double* cpu_k = (double*)malloc(N * N * sizeof(double));
    // hipMemcpy(cpu_k, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost);
    // writeArrayToTxtFile<double>(cpu_k, N * N, filename_A);

    std::string filename_k = "/usr/scratch/mont-fort17/almaeder/manasa_kmc_matrices/K_" + std::to_string(kmc_step_count) + ".bin";
    double* cpu_k = (double*)malloc(N * N * sizeof(double));
    hipMemcpy(cpu_k, gpu_k, N * N * sizeof(double), hipMemcpyDeviceToHost);
    writeArrayToBinFile<double>(cpu_k, N * N, filename_k);


    std::string filename_rhs = "/usr/scratch/mont-fort17/almaeder/manasa_kmc_matrices/rhs_" + std::to_string(kmc_step_count) + ".bin";
    double* cpu_rhs = (double*) malloc(N_interface * sizeof(double));
    hipMemcpy(cpu_rhs, gpu_k_sub, N_interface * sizeof(double), hipMemcpyDeviceToHost);
    writeArrayToBinFile<double>(cpu_k, N_interface, filename_rhs);

    // points to the start of Koxide inside K:
    double* gpu_D = gpu_k + (N_left_tot * N) + N_left_tot;

    CheckCusolverDnError(hipsolverDnDgetrf_bufferSize(handle, N_interface, N_interface, gpu_D, N, &lwork));
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMalloc((void **)(&gpu_work), sizeof(double) * lwork) );

    CheckCusolverDnError(hipsolverDnDgetrf(handle, N_interface, N_interface, gpu_D, N, gpu_work, gpu_ipiv, gpu_info));
    // hipMemcpy(&info, gpu_info, sizeof(int), hipMemcpyDeviceToHost); // printf("info for hipsolverDnDgetrf: %i \n", info);
    gpuErrchk( hipDeviceSynchronize() );

    CheckCusolverDnError(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, N_interface, 1, gpu_D, N, gpu_ipiv, gpu_k_sub, N_interface, gpu_info));
    // hipMemcpy(&info, gpu_info, sizeof(int), hipMemcpyDeviceToHost); // printf("info for hipsolverDnDgetrs: %i \n", info);
    gpuErrchk( hipDeviceSynchronize() );

    hipFree(gpu_k);

    num_blocks = (N_interface - 1) / num_threads + 1;
    set_potential<<<num_blocks, num_threads>>>(gpubuf.site_potential + N_left_tot, gpu_k_sub, N_interface);
    gpuErrchk( hipPeekAtLastError() ); 
    gpuErrchk( hipDeviceSynchronize() ); 
    hipFree(gpu_k_sub);

    gpuErrchk( hipMemcpy(gpubuf.site_potential, VL, N_left_tot * sizeof(double), hipMemcpyDeviceToDevice) );
    gpuErrchk( hipMemcpy(gpubuf.site_potential + N_left_tot + N_interface, VR, N_right_tot * sizeof(double), hipMemcpyDeviceToDevice) );

    hipFree(gpu_ipiv);
    hipFree(gpu_work);
    hipFree(gpu_info);
    hipFree(VL);
    hipFree(VR);

}

void poisson_gridless_gpu(const int num_atoms_contact, const int pbc, const int N, const double *lattice, 
                          const double *sigma, const double *k,
                          const double *posx, const double *posy, const double *posz, 
                          const int *site_charge, double *site_potential){

    int num_threads = 1024;
    int blocks_per_row = (N - 1) / num_threads + 1; 
    int num_blocks = blocks_per_row * N; // NOTE: fix the kernel for block overflow!

    calculate_pairwise_interaction<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(posx, posy, posz, lattice, pbc, N, sigma, k, site_charge, site_potential);
}

void update_power_gpu(hipblasHandle_t handle, hipsolverHandle_t handle_cusolver, GPUBuffers &gpubuf, const int N, const int num_source_inj, const int num_ground_ext,
                      const double Vd, const int pbc, const double high_G, const double low_G,
                      const double nn_dist, const double m_e, const double V0, int num_metals, const double t_ox, double *imacro)
{
    int *gpu_index;
    hipMalloc((void **)&gpu_index, N * sizeof(int)); // indices of the site array
    int *atom_gpu_index;
    hipMalloc((void **)&atom_gpu_index, N * sizeof(int)); // indices of the atom array

    thrust::device_ptr<int> gpu_index_ptr = thrust::device_pointer_cast(gpu_index);
    thrust::sequence(gpu_index_ptr, gpu_index_ptr + N, 0);

    double *last_atom = thrust::copy_if(thrust::device, gpubuf.site_x, gpubuf.site_x + N, gpubuf.site_element, gpubuf.atom_x, is_defect());
    int N_atom = last_atom - gpubuf.atom_x;
    thrust::copy_if(thrust::device, gpubuf.site_y, gpubuf.site_y + N, gpubuf.site_element, gpubuf.atom_y, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_z, gpubuf.site_z + N, gpubuf.site_element, gpubuf.atom_z, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_charge, gpubuf.site_charge + N, gpubuf.site_element, gpubuf.atom_charge, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_potential, gpubuf.site_potential + N, gpubuf.site_element, gpubuf.atom_potential, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_element, gpubuf.site_element + N, gpubuf.site_element, gpubuf.atom_element, is_defect());
    thrust::copy_if(thrust::device, gpu_index, gpu_index + N, gpubuf.site_element, atom_gpu_index, is_defect());

    double *gpu_imacro, *gpu_m, *gpu_x, *gpu_ineg, *gpu_diag, *gpu_pdisp, *gpu_A;
    hipMalloc((void **)&gpu_imacro, 1 * sizeof(double));                      // IMACRO
    hipMalloc((void **)&gpu_m, (N_atom + 2) * sizeof(double));                // M
    hipMalloc((void **)&gpu_x, (size_t) (N_atom + 2) * (N_atom + 2) * sizeof(double)); // X
    hipMalloc((void **)&gpu_ineg, (size_t) N_atom * N_atom * sizeof(double));          // INEG
    hipMalloc((void **)&gpu_diag, (N_atom + 2) * sizeof(double));             // DIAG
    hipMalloc((void **)&gpu_pdisp, N_atom * sizeof(double));                  // PDISP
    hipMalloc((void **)&gpu_A, (size_t) (N_atom + 1) * (N_atom + 1) * sizeof(double)); // A

    hipMemset(gpu_x, 0, (size_t) (N_atom + 2) * (N_atom + 2) * sizeof(double));
    hipDeviceSynchronize();

    hipMemset(gpu_m, 0, (N_atom + 2) * sizeof(double));
    hipDeviceSynchronize();

    // Make M vector
    hipMemset(gpu_m, 0, (N_atom + 2) * sizeof(double));
    hipDeviceSynchronize();

    thrust::device_ptr<double> m_ptr = thrust::device_pointer_cast(gpu_m);
    thrust::fill(m_ptr, m_ptr + 1, -high_G * Vd);
    thrust::fill(m_ptr + 1, m_ptr + 2, high_G * Vd);

    // Create X
    int num_threads = 128;
    int blocks_per_row = (N_atom - 1) / num_threads + 1;
    int num_blocks = blocks_per_row * N;
    create_X<<<num_blocks, num_threads>>>(
        gpu_x, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
        gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_potential,
        gpubuf.lattice, pbc, high_G, low_G,
        nn_dist, m_e, V0, num_source_inj, num_ground_ext, N_atom, num_metals, t_ox, Vd);
    hipDeviceSynchronize();

    // Diag X
    hipMemset(gpu_diag, 0, (N_atom + 2) * sizeof(double));
    hipDeviceSynchronize();
    num_threads = 512;
    blocks_per_row = (N_atom + 2 - 1) / num_threads + 1;
    num_blocks = blocks_per_row * (N + 2);
    diagonal_sum<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_x, gpu_diag, N_atom + 2);
    hipDeviceSynchronize();

    set_diag<<<blocks_per_row, num_threads>>>(gpu_x, gpu_diag, N_atom + 2);
    hipDeviceSynchronize();

    // GESV
    int lwork = 0;              /* size of workspace */
    double *gpu_work = nullptr; /* device workspace for getrf */
    int *gpu_info = nullptr;    /* error info */
    int *gpu_ipiv;

    // double *gpu_A = gpu_x + (N_atom + 2) + 1
    // double* gpu_D = gpu_k + (N_left_tot * N) + N_left_tot;

    hipMalloc((void **)&gpu_ipiv, (N_atom + 1) * sizeof(int));
    hipMalloc((void **)(&gpu_info), sizeof(int));
    hipMemcpy2D(gpu_A, (N_atom + 1) * sizeof(double), gpu_x, (N_atom + 2) * sizeof(double), (N_atom + 1) * sizeof(double), (N_atom + 1), hipMemcpyDeviceToDevice);

    CheckCusolverDnError(hipsolverDnDgetrf_bufferSize(handle_cusolver, N_atom + 1, N_atom + 1, gpu_A, N_atom + 1, &lwork));
    hipDeviceSynchronize();

    hipMalloc((void **)(&gpu_work), sizeof(double) * lwork);

    // Solve Ax=B through LU factorization
    CheckCusolverDnError(hipsolverDnDgetrf(handle_cusolver, N_atom + 1, N_atom + 1, gpu_A, N_atom + 1, gpu_work, gpu_ipiv, gpu_info));
    hipDeviceSynchronize();

    CheckCusolverDnError(hipsolverDnDgetrs(handle_cusolver, HIPBLAS_OP_T, N_atom + 1, 1, gpu_A, N_atom + 1, gpu_ipiv, gpu_m, N_atom + 1, gpu_info));
    hipDeviceSynchronize();

    // NOTE: M is different from the cpu code in the 4th decimal place!!!
    // Compute I_macro
    hipMemset(gpu_imacro, 0, sizeof(double));
    hipDeviceSynchronize();

    num_threads = 512;
    num_blocks = (N_atom - 1) / num_threads + 1;
    // num_blocks = min(65535, num_blocks);
    get_imacro<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_x, gpu_m, gpu_imacro, Vd, N_atom);
    hipDeviceSynchronize();

    // Copy back I_macro to the GPU
    double i_macro = 0;
    hipMemcpy(&i_macro, gpu_imacro, sizeof(double), hipMemcpyDeviceToHost);
    *imacro = i_macro;

    // Find index of minimum element in m[2:N+2]
    auto min_index = thrust::min_element(thrust::device, gpu_m + 2, gpu_m + N_atom + 2) - gpu_m;

    num_threads = 512;
    blocks_per_row = (N_atom + 2 - 1) / num_threads + 1;
    num_blocks = blocks_per_row;
    update_m<<<num_blocks, num_threads>>>(gpu_m, min_index, N_atom + 2);
    hipDeviceSynchronize();

    // Compute I_neg
    num_threads = 128;
    blocks_per_row = (N_atom - 1) / num_threads + 1;
    num_blocks = blocks_per_row * N;

    set_ineg<<<num_blocks, num_threads>>>(gpu_ineg, gpu_x, gpu_m, Vd, N_atom,
                                          gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                                          pbc, gpubuf.lattice, nn_dist,
                                          V0, gpubuf.atom_potential, t_ox);
    hipDeviceSynchronize();


    // Update I_neg diagonal
    hipMemset(gpu_diag, 0, (N_atom + 2) * sizeof(double));
    hipDeviceSynchronize();

    diagonal_sum<NUM_THREADS><<<num_blocks, num_threads, NUM_THREADS * sizeof(double)>>>(gpu_ineg, gpu_diag, N_atom);
    hipDeviceSynchronize();

    set_diag<<<blocks_per_row, num_threads>>>(gpu_ineg, gpu_diag, N_atom);
    hipDeviceSynchronize();

    double alpha = 1.0, beta = 0.0;
    double *gpu_alpha, *gpu_beta;
    hipMalloc((void **)&gpu_alpha, sizeof(double));
    hipMalloc((void **)&gpu_beta, sizeof(double));
    hipMemcpy(gpu_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_beta, &beta, sizeof(double), hipMemcpyHostToDevice);

    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    CheckCublasError( hipblasDgemv(handle, HIPBLAS_OP_T, N_atom, N_atom, gpu_alpha, gpu_ineg, N_atom, gpu_m + 2, 1, gpu_beta, gpu_pdisp, 1) );
    hipDeviceSynchronize();

    num_threads = 512;
    num_blocks = (N_atom - 1) / num_threads + 1;
    num_blocks = min(65535, num_blocks);
    copy_pdisp<<<num_blocks, num_threads>>>(gpubuf.site_power, gpubuf.site_element, gpubuf.metal_types, gpu_pdisp, atom_gpu_index, N_atom, num_metals);
    hipDeviceSynchronize();

    hipFree(gpu_ipiv);
    hipFree(gpu_work);
    hipFree(gpu_imacro);
    hipFree(gpu_m);
    hipFree(gpu_x);
    hipFree(gpu_ineg);
    hipFree(gpu_diag);
    hipFree(gpu_pdisp);
    hipFree(gpu_A);
    hipFree(gpu_info);
    hipFree(gpu_alpha);
    hipFree(gpu_beta);
    hipFree(gpu_index);
    hipFree(atom_gpu_index);
}

double execute_kmc_step_gpu(const int N, const int nn, const int *neigh_idx, const int *site_layer,
                            const double *lattice, const int pbc, const double *T_bg, 
                            const double *freq, const double *sigma, const double *k,
                            const double *posx, const double *posy, const double *posz, 
                            const double *site_potential, const double *site_temperature,
                            ELEMENT *site_element, int *site_charge, RandomNumberGenerator &rng, const int *neigh_idx_host){

    // **************************
    // **** Build Event List ****
    // **************************

    // the KMC event list arrays only exist in gpu memory
    EVENTTYPE *event_type; 
    double    *event_prob; 
    gpuErrchk( hipMalloc((void**)&event_type, N * nn * sizeof(EVENTTYPE)) );
    gpuErrchk( hipMalloc((void**)&event_prob, N * nn * sizeof(double)) );
                
    int num_threads = 512;
    int num_blocks = (N * nn - 1) / num_threads + 1;

    // populate the event_type and event_prob arrays:
    build_event_list<<<num_blocks, num_threads>>>(N, nn, neigh_idx, 
                                                 site_layer, lattice, pbc,
                                                 T_bg, freq, sigma, k,
                                                 posx, posy, posz, 
                                                 site_potential, site_temperature, 
                                                 site_element, site_charge, event_type, event_prob);

    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

    // **************************
    // ** Event Execution Loop **
    // **************************

    // helper variables:
    // NOTE: INITIALIZE THESE ON GPU AND USE MEMCPY DEVICETODEVICE INSTEAD
    int two_host = 2;
    int two_neg_host = -2;
    int zero_host = 0;
    ELEMENT defect_element_host = DEFECT;
    ELEMENT O_defect_element_host = OXYGEN_DEFECT;
    ELEMENT vacancy_element_host = VACANCY;
    ELEMENT O_element_host = O_EL;

    double *event_prob_cum;
    gpuErrchk( hipMalloc((void**)&event_prob_cum, N * nn * sizeof(double)) );
 
    double freq_host;
    gpuErrchk( hipMemcpy(&freq_host, freq, 1 * sizeof(double), hipMemcpyDeviceToHost) );

    double event_time = 0.0;
    while (event_time < 1 / freq_host) {

        // get the cumulative sum of the probabilities
        thrust::inclusive_scan(thrust::device, event_prob, event_prob + N * nn, event_prob_cum);

        // select an event
        double Psum_host;
        gpuErrchk( hipMemcpy(&Psum_host, event_prob_cum + N * nn - 1, sizeof(double), hipMemcpyDeviceToHost) );
        double number = rng.getRandomNumber() * Psum_host;
        int event_idx = thrust::upper_bound(thrust::device, event_prob_cum, event_prob_cum + N * nn, number) - event_prob_cum;
        // std::cout << "selected event: " << event_idx << "\n";

        EVENTTYPE sel_event_type = NULL_EVENT;
        gpuErrchk( hipMemcpy(&sel_event_type, event_type + event_idx, sizeof(EVENTTYPE), hipMemcpyDeviceToHost) );

        // test output:
        // double sel_event_prob;
        // gpuErrchk( hipMemcpy(&sel_event_prob, event_prob + event_idx, sizeof(double), hipMemcpyDeviceToHost) );
        // std::cout << "Selected event index: " << event_idx << " with type "
        //           << sel_event_type << " and probability " << sel_event_prob << std::endl;

        // get attributes of the sites involved:
        int i_host = static_cast<int>(floorf(event_idx / nn));
        int j_host;
        ELEMENT element_i_host, element_j_host;
        int charge_i_host, charge_j_host;

        gpuErrchk( hipMemcpy(&j_host, neigh_idx + event_idx, sizeof(int), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&element_i_host, site_element + i_host, sizeof(ELEMENT), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&element_j_host, site_element + j_host, sizeof(ELEMENT), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&charge_i_host, site_charge + i_host, sizeof(int), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&charge_j_host, site_charge + j_host, sizeof(int), hipMemcpyDeviceToHost) );

        // Event execution loop
        switch (sel_event_type)
        {
        case VACANCY_GENERATION:
        {
            if (element_i_host != DEFECT || element_j_host != O_EL)
            {
                print("Wrong event type - VACANCY_GENERATION!");
                print(return_element(element_i_host) << " and " << return_element(element_j_host));
            }

            // turn the defect (i) into an oxygen ion
            // turn the oxygen (j) into a charged vacancy
            gpuErrchk( hipMemcpy(site_element + i_host, &O_defect_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, &vacancy_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, &two_neg_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, &two_host, 1 * sizeof(int), hipMemcpyHostToDevice) );

            break;
        }
        case VACANCY_RECOMBINATION:
        {
            if (element_i_host != OXYGEN_DEFECT || element_j_host != VACANCY)
            {
                print("Wrong event type - VACANCY_RECOMBINATION!");
                print(return_element(element_i_host) << " and " << return_element(element_j_host));
            }

            // turn the oxygen (i) into a defect
            // turn the vacancy (j) into an oxygen atom
            gpuErrchk( hipMemcpy(site_element + i_host, &defect_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, &O_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, &zero_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, &zero_host, 1 * sizeof(int), hipMemcpyHostToDevice) );

            break;
        }
        case VACANCY_DIFFUSION:
        {
            if (element_i_host != VACANCY || element_j_host != O_EL)
            {
                print("Wrong event type - VACANCY_DIFFUSION!");
                print(return_element(element_i_host) << " and " << return_element(element_j_host));
            }

            // swap element and charge
            gpuErrchk( hipMemcpy(site_element + i_host, &element_j_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, &element_i_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, &charge_j_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, &charge_i_host, 1 * sizeof(int), hipMemcpyHostToDevice) );

            break;
        }
        case ION_DIFFUSION:
        {
            if (element_i_host != OXYGEN_DEFECT || element_j_host != DEFECT)
            {
                print("Wrong event type - ION_DIFFUSION!");
                print(return_element(element_i_host) << " and " << return_element(element_j_host));
            }

            // swap element and charge
            gpuErrchk( hipMemcpy(site_element + i_host, &element_j_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, &element_i_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, &charge_j_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, &charge_i_host, 1 * sizeof(int), hipMemcpyHostToDevice) );

            break;
        }
        default:
            print("error: unidentified event key found: ");
            print(sel_event_type);
        }

        // Deactivate conflicting events

        EVENTTYPE null_event_host = NULL_EVENT;
        double zero_double_host = 0.0;

        // other site's events with i or j
         int i_, j_;
        for (auto idx = 0; idx < N * nn; ++idx){
            i_ = std::floor(idx / nn);
            j_ = neigh_idx_host[idx];

            if (i_host == i_ || j_host == j_ || i_host == j_ || j_host == i_){

                gpuErrchk( hipMemcpy(event_type + idx, &null_event_host, 1 * sizeof(EVENTTYPE), hipMemcpyHostToDevice) );
                gpuErrchk( hipMemcpy(event_prob + idx, &zero_double_host, 1 * sizeof(double), hipMemcpyHostToDevice) );
            }
        }

        // REPLACE WITH THRUST::FILL AND CUDAMEMSET
        for (int fill_ind = 0; fill_ind < (nn-1); fill_ind++){

            // i's events with its neighbors    
            gpuErrchk( hipMemcpy(event_type + i_host * nn + fill_ind, &null_event_host, 1 * sizeof(EVENTTYPE), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(event_type + i_host * nn + fill_ind, &zero_double_host, 1 * sizeof(double), hipMemcpyHostToDevice) );

            // j's events with its neighbors
            gpuErrchk( hipMemcpy(event_type + j_host * nn + fill_ind, &null_event_host, 1 * sizeof(EVENTTYPE), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(event_type + j_host * nn + fill_ind, &zero_double_host, 1 * sizeof(double), hipMemcpyHostToDevice) );
        }

        event_time = -log(rng.getRandomNumber()) / Psum_host;
        // std::cout << "event time: " << event_time << "\n";
    }

    gpuErrchk( hipFree(event_prob_cum) );
    gpuErrchk( hipFree(event_type) );
    gpuErrchk( hipFree(event_prob) );

    return event_time;    
}

void copytoConstMemory(std::vector<double> E_gen, std::vector<double> E_rec, std::vector<double> E_Vdiff, std::vector<double> E_Odiff)
{   
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_gen_const), E_gen.data(), E_gen.size() * sizeof(double)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_rec_const), E_rec.data(), E_rec.size() * sizeof(double)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_Vdiff_const), E_Vdiff.data(), E_Vdiff.size() * sizeof(double)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_Odiff_const), E_Odiff.data(), E_Odiff.size() * sizeof(double)) );
}

    // # if __CUDA_ARCH__>=200
    // printf("%i \n", tid);
    // #endif  


        // std::cout << "created K"
    //           << "\n";

    // // debug - use floor() for i
    // std::vector<double> new_k(N * N);
    // gpuErrchk( hipMemcpy(new_k.data(), gpu_k , N * N * sizeof(double), hipMemcpyDeviceToHost) );
    // std::cout << "copied";
    // std::ofstream fout("new_k.txt");
    // for(int i = 0; i< N*N; i++){
    //     if (new_k[i] != 0){
    //         fout << new_k[i]; 
    //         fout << ' ';
    //     }
    // }
    // exit(1);


    // // debug
    // double *M = (double *)calloc(N, sizeof(double));
    // gpuErrchk( hipMemcpy(M, gpubuf.site_potential, N * sizeof(double), hipMemcpyDeviceToHost) );
    // std::cout << "copied\n";
    // std::ofstream fout2("gpu_M.txt");
    // for(int i = 0; i< N; i++){
    //     if (M[i] != 0){
    //         fout2 << M[i]; 
    //         fout2 << ' ';
    //     }
    // }


    // // ************************************************************
    // // 1. Convert D matrix to sparse COO format using the COOElement struct
    // int numNonZero = 0;

    // COOElement* d_cooData;
    // hipMalloc((void**)&d_cooData, N_interface * N_interface * sizeof(COOElement));

    // int* d_numNonZero;
    // hipMalloc((void**)&d_numNonZero, sizeof(int));
    // hipMemcpy(d_numNonZero, &numNonZero, sizeof(int), hipMemcpyHostToDevice);

    // int numThreadsPerBlock = 256;
    // int numBlocks = (N_interface + numThreadsPerBlock - 1) / numThreadsPerBlock;
    // extractCOOData<<<numBlocks, numThreadsPerBlock>>>(gpu_D, N_interface, d_cooData, d_numNonZero);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // hipMemcpy(&numNonZero, d_numNonZero, sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << "number of nonzeros in COO rep: " << numNonZero << "\n";

    // // ************************************************************
    // // 2. Sort and filter the coordinates
    // thrust::device_vector<COOElement> d_cooDataVec(d_cooData, d_cooData + numNonZero);
    
    // // Sort the COO data based on row-major order 
    // thrust::sort(thrust::device, d_cooDataVec.begin(), d_cooDataVec.end(), [] __device__ (const COOElement& a, const COOElement& b) {
    //     return a.row < b.row || (a.row == b.row && a.col < b.col);
    // });

    // // Remove duplicate entries 
    // auto newEnd = thrust::unique(thrust::device, d_cooDataVec.begin(), d_cooDataVec.end(), [] __device__ (const COOElement& a, const COOElement& b) {
    //     return a.row == b.row && a.col == b.col;
    // });
    // numNonZero = newEnd - d_cooDataVec.begin();

    // // ************************************************************
    // // 3. Extract CSR data from COO representation and nnz
    // int* d_csrRowPtr;
    // int* d_csrColIndices;
    // double* d_csrValues;

    // hipMalloc((void**)&d_csrRowPtr, (N_interface + 1) * sizeof(int));   // +1 for the row pointers
    // hipMalloc((void**)&d_csrColIndices, numNonZero * sizeof(int));      // Allocate space for the maximum number of non-zero elements
    // hipMalloc((void**)&d_csrValues, numNonZero * sizeof(double));       // Allocate space for the maximum number of non-zero elements
    // hipMemset(d_csrRowPtr, 0, (N_interface + 1) * sizeof(int));
    // gpuErrchk( hipDeviceSynchronize() );

    // extractCOOToCSR<<<numBlocks, numThreadsPerBlock>>>(thrust::raw_pointer_cast(d_cooDataVec.data()), numNonZero, N_interface, d_csrRowPtr, d_csrColIndices, d_csrValues);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // std::cout << "number of nonzeros in CSR rep: " << numNonZero << "\n";
