#include "cuda_wrapper.h"

extern "C"
void get_gpu_info(char *gpu_string, int dev){
 struct hipDeviceProp_t dprop;
 hipGetDeviceProperties(&dprop, dev);
 strcpy(gpu_string,dprop.name);
}

extern "C"
void set_gpu(int dev){
 hipSetDevice(dev);
}

