#include "hip/hip_runtime.h"
#include "cuda_wrapper.h"

// check that sparse and dense versions are the same
void check_sparse_dense_match(int m, int nnz, double *dense_matrix, int* d_csrRowPtr, int* d_csrColInd, double* d_csrVal){
    
    double *h_D = (double *)calloc(m*m, sizeof(double));
    double *h_D_csr = (double *)calloc(nnz, sizeof(double));
    int *h_pointers = (int *)calloc((m + 1), sizeof(int));
    int *h_inds = (int *)calloc(nnz, sizeof(int));

    gpuErrchk( hipMemcpy(h_D, dense_matrix, m*m * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_D_csr, d_csrVal, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_pointers, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_inds, d_csrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int nnz_count = 0;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < m; col++) {
            int i = row * m + col;  // Linear index in dense matrix
            // Check if the element in the dense matrix is non-zero
            if (h_D[i] != 0) {
                // Compare the row and column indices
                if (h_D[i] != h_D_csr[nnz_count] || col != h_inds[nnz_count]) {
                    std::cout << "Mismatch found at (row, col) = (" << row << ", " << col << ")\n";
                }
                nnz_count++;
            }
        }
    }
}

// dump sparse matrix into a file
void dump_csr_matrix_txt(int m, int nnz, int* d_csrRowPtr, int* d_csrColIndices, double* d_csrValues, int kmc_step_count){

    // Copy matrix back to host memory
    double *h_csrValues = (double *)calloc(nnz, sizeof(double));
    int *h_csrRowPtr = (int *)calloc((m + 1), sizeof(int));
    int *h_csrColIndices = (int *)calloc(nnz, sizeof(int));
    gpuErrchk( hipMemcpy(h_csrValues, d_csrValues, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_csrRowPtr, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_csrColIndices, d_csrColIndices, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    // print to file, tagged with the kmc step number
    std::ofstream fout_val("csrValues_step#" + std::to_string(kmc_step_count) + ".txt");
    for(int i = 0; i < nnz; i++){
        fout_val << h_csrValues[i] << " "; 
    }
    std::ofstream fout_row("csrRowPtr_step#" + std::to_string(kmc_step_count) + ".txt");
    for(int i = 0; i < (m + 1); i++){
        fout_row << h_csrRowPtr[i] << " "; 
    }
    std::ofstream fout_col("csrColIndices_step#" + std::to_string(kmc_step_count) + ".txt");
    for(int i = 0; i < nnz; i++){
        fout_col << h_csrColIndices[i] << " "; 
    }

    free(h_csrValues);
    free(h_csrRowPtr);
    free(h_csrColIndices);
}

// Solution of A*x = y using cusolver in host pointer mode
void sparse_system_solve(hipsolverSpHandle_t handle, int* d_csrRowPtr, int* d_csrColInd, double* d_csrVal,
                         int nnz, int m, double *d_x, double *d_y){

    // Ref: https://stackoverflow.com/questions/31840341/solving-general-sparse-linear-systems-in-cuda

    // cusolverSpDcsrlsvlu only supports the host path
    int *h_A_RowIndices = (int *)malloc((m + 1) * sizeof(int));
    int *h_A_ColIndices = (int *)malloc(nnz * sizeof(int));
    double *h_A_Val = (double *)malloc(nnz * sizeof(double));
    double *h_x = (double *)malloc(m * sizeof(double));
    double *h_y = (double *)malloc(m * sizeof(double));
    gpuErrchk( hipMemcpy(h_A_RowIndices, d_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_A_ColIndices, d_csrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_A_Val, d_csrVal, nnz * sizeof(double), hipMemcpyDeviceToHost) );   
    gpuErrchk( hipMemcpy(h_x, d_x, m * sizeof(double), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(h_y, d_y, m * sizeof(double), hipMemcpyDeviceToHost) );

    hipsparseMatDescr_t matDescrA;
    hipsparseCreateMatDescr(&matDescrA);
    hipsparseSetMatType(matDescrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matDescrA, HIPSPARSE_INDEX_BASE_ZERO);

    int singularity;
    double tol = 0.00000001;

    // Solve with LU
    // CheckCusolverDnError( cusolverSpDcsrlsvluHost(handle, m, nnz, matDescrA, h_A_Val, h_A_RowIndices, 
    //                       h_A_ColIndices, h_y, tol, 0, h_x, &singularity) );
    
    // Solve with QR
    // CheckCusolverDnError( cusolverSpDcsrlsvqrHost(handle, m, nnz, matDescrA, h_A_Val, h_A_RowIndices, 
    //                       h_A_ColIndices, h_y, tol, 1, h_x, &singularity) );

    // Solve with Cholesky
    CheckCusolverDnError( hipsolverSpDcsrlsvcholHost(handle, m, nnz, matDescrA, h_A_Val, h_A_RowIndices,
                          h_A_ColIndices, h_y, tol, 1, h_x, &singularity) );

    gpuErrchk( hipDeviceSynchronize() );
    if (singularity != -1){
        std::cout << "In sparse_system_solve: Matrix has a singularity at : " << singularity << "\n";
    }

    // copy back the solution vector:
    gpuErrchk( hipMemcpy(d_x, h_x, m * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverSpDestroy(handle);
    hipsparseDestroyMatDescr(matDescrA);
    free(h_A_RowIndices);
    free(h_A_ColIndices);
    free(h_A_Val);
    free(h_x);
    free(h_y);
}

// Iterative sparse linear solver using CG steps
void sparse_system_solve_iterative(hipblasHandle_t handle_cublas, hipsparseHandle_t handle, 
								   hipsparseSpMatDescr_t matA, int m, double *d_x, double *d_y){

    // A is an m x m sparse matrix represented by CSR format
    // - d_x is right hand side vector in gpu memory,
    // - d_y is solution vector in gpu memory.
    // - d_z is intermediate result on gpu memory.

    // Sets the initial guess for the solution vector to zero
    bool zero_guess = 0;

    // Error tolerance for the norm of the residual in the CG steps
    double tol = 1e-12;

    double one = 1.0;
    double n_one = -1.0;
    double zero = 0.0;
    double *one_d, *n_one_d, *zero_d;
    gpuErrchk( hipMalloc((void**)&one_d, sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&n_one_d, sizeof(double)) );
    gpuErrchk( hipMalloc((void**)&zero_d, sizeof(double)) );
    gpuErrchk( hipMemcpy(one_d, &one, sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(n_one_d, &n_one, sizeof(double), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(zero_d, &zero, sizeof(double), hipMemcpyHostToDevice) );
    hipsparseStatus_t status;

    // ************************************
    // ** Precondioner and Initial Guess **

    if (zero_guess)
    {
        // Set the initial guess for the solution vector to zero
        gpuErrchk( hipMemset(d_y, 0, m * sizeof(double)) ); 
        gpuErrchk( hipDeviceSynchronize() );
    }

    // *******************************
    // ** Iterative refinement loop **

    // initialize variables for the residual calculation
    double h_norm;
    double *d_r, *d_p, *d_temp;
    gpuErrchk( hipMalloc((void**)&d_r, m * sizeof(double)) ); 
    gpuErrchk( hipMalloc((void**)&d_p, m * sizeof(double)) ); 
    gpuErrchk( hipMalloc((void**)&d_temp, m * sizeof(double)) ); 

    // for SpMV:
    // - d_x is right hand side vector
    // - d_y is solution vector
    hipsparseDnVecDescr_t vecY, vecR, vecP, vectemp; 
    hipsparseCreateDnVec(&vecY, m, d_y, HIP_R_64F);
    hipsparseCreateDnVec(&vecR, m, d_r, HIP_R_64F);
    hipsparseCreateDnVec(&vecP, m, d_p, HIP_R_64F);
    hipsparseCreateDnVec(&vectemp, m, d_temp, HIP_R_64F);

    size_t MVBufferSize;
    void *MVBuffer = 0;
    status = hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, matA, 
                          vecY, zero_d, vectemp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &MVBufferSize);
    gpuErrchk( hipMalloc((void**)&MVBuffer, sizeof(double) * MVBufferSize) );
    
    // Initialize the residual and conjugate vectors
    // r = A*y - x & p = -r
    status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, matA, 
                          vecY, zero_d, vecR, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, MVBuffer);         // r = A*y
    //gpuErrchk( hipDeviceSynchronize() );
    CheckCublasError( hipblasDaxpy(handle_cublas, m, &n_one, d_x, 1, d_r, 1) );                          // r = -x + r
    //gpuErrchk( hipDeviceSynchronize() );
    CheckCublasError(hipblasDcopy(handle_cublas, m, d_r, 1, d_p, 1));                                    // p = r
    //gpuErrchk( hipDeviceSynchronize() );
    CheckCublasError(hipblasDscal(handle_cublas, m, &n_one, d_p, 1));                                    // p = -p
    //gpuErrchk( hipDeviceSynchronize() );

    // calculate the error (norm of the residual)
    CheckCublasError( hipblasDnrm2(handle_cublas, m, d_r, 1, &h_norm) );
    gpuErrchk( hipDeviceSynchronize() );
    
    // Conjugate Gradient steps
    int counter = 0;
    double t, tnew, alpha, beta, alpha_temp;
    while (h_norm > tol){

        // alpha = rT * r / (pT * A * p)
        CheckCublasError( hipblasDdot (handle_cublas, m, d_r, 1, d_r, 1, &t) );                         // t = rT * r
        //gpuErrchk( hipDeviceSynchronize() );
        status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, one_d, matA, 
                              vecP, zero_d, vectemp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, MVBuffer); // temp = A*p
        //gpuErrchk( hipDeviceSynchronize() );
        CheckCublasError( hipblasDdot (handle_cublas, m, d_p, 1, d_temp, 1, &alpha_temp) );             // alpha = pT*temp = pT*A*p
        //gpuErrchk( hipDeviceSynchronize() );
        alpha = t / alpha_temp; 

        // y = y + alpha * p
        CheckCublasError(hipblasDaxpy(handle_cublas, m, &alpha, d_p, 1, d_y, 1));                       // y = y + alpha * p
        //gpuErrchk( hipDeviceSynchronize() );

        // r = r + alpha * A * p 
        CheckCublasError(hipblasDaxpy(handle_cublas, m, &alpha, d_temp, 1, d_r, 1));                    // r = r + alpha * temp
        //gpuErrchk( hipDeviceSynchronize() );

        // beta = (rT * r) / t
        CheckCublasError( hipblasDdot (handle_cublas, m, d_r, 1, d_r, 1, &tnew) );                       // tnew = rT * r
        //gpuErrchk( hipDeviceSynchronize() );
        beta = tnew / t;

        // p = -r + beta * p
        CheckCublasError(hipblasDscal(handle_cublas, m, &beta, d_p, 1));                                  // p = p * beta
        //gpuErrchk( hipDeviceSynchronize() );
        CheckCublasError(hipblasDaxpy(handle_cublas, m, &n_one, d_r, 1, d_p, 1));                         // p = p - r
        //gpuErrchk( hipDeviceSynchronize() );

        // calculate the error (norm of the residual)
        CheckCublasError( hipblasDnrm2(handle_cublas, m, d_r, 1, &h_norm) );
        //gpuErrchk( hipDeviceSynchronize() );
        //std::cout << h_norm << "\n";

        counter++;
        if (counter > 10000){
            std::cout << "WARNING: probably stuck in diverging CG iterations, check the residual!\n";
        }
    }
    std::cout << "# CG steps: " << counter << "\n";

    // // check solution vector
    // double *copy_back = (double *)calloc(m, sizeof(double));
    // gpuErrchk( hipMemcpy(copy_back, d_y, m * sizeof(double), hipMemcpyDeviceToHost) );
    // for (int i = 0; i < m; i++){
    //     std::cout << copy_back[i] << " ";
    // }
    
}