#include "hip/hip_runtime.h"
#include "gpu_solvers.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

const double eV_to_J = 1.60217663e-19;          // [C]
const double h_bar = 1.054571817e-34;           // [Js]

// Compute the number of nonzeros per row of the matrix including the injection, extraction, and device nodes (excluding the ground). 
// Has dimensions of Nsub by Nsub (by the cpu code)
__global__ void calc_nnz_per_row_T( const double *posx_d, const double *posy_d, const double *posz_d,
                                    const ELEMENT *metals, const ELEMENT *element, const double *atom_CB_edge, const double *lattice, bool pbc,
                                    double nn_dist, const double tol,
                                    int num_source_inj, int num_ground_ext, const int num_layers_contact,
                                    int num_metals, int N_sub,
                                    int block_size_i, int block_size_j, int block_start_i, int block_start_j, int *nnz_per_row_d){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int Natom = N_sub - 1; 

    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){  // Nsub x Nsub matrix divided into blocks by rank
        int nnz_row = 0;
        for(int col = 0; col < block_size_j; col++){

            int i = block_start_i + row; // i indexes the matrix and i-2 indexes the atoms
            int j = block_start_j + col; 

            if ( i == j ) // all diagonal terms
            {
                nnz_row++;
            }

            if ( (i == 0 && j == 1)  || (i == 1 && j == 0) ) // loop connection
            {
                nnz_row++;
            }

            if ( i == 0 && ( j > ((N_sub+1) - num_ground_ext) )) // extraction terms minus ground node
            {
                nnz_row++;
            }

            if ( i == 1 && (j > 1) && (j < num_source_inj+2) ) // injection terms minus ground node
            {
                nnz_row++;
            }

            if (i > 1 && i != j)
            {
                // inj/ext terms
                if ( (j == 0) && ( i > ((N_sub+1) - num_ground_ext) ) )
                {
                    nnz_row++;
                }

                if ( (j == 1) && (i > 1) && (i < num_source_inj + 2) ) 
                {
                    nnz_row++;
                }

                double dist = site_dist_gpu(posx_d[i-2], posy_d[i-2], posz_d[i-2],
                                            posx_d[j-2], posy_d[j-2], posz_d[j-2],
                                            lattice[0], lattice[1], lattice[2], pbc);

                // direct terms 
                if ( j > 1 && i != j && dist < nn_dist )
                {
                    nnz_row++;
                }

                // // tunneling terms 
                // if ( i != j && dist > nn_dist )
                // { 
                //     bool any_vacancy1 = element[i-2] == VACANCY;
                //     bool any_vacancy2 = element[j-2] == VACANCY;

                //     // contacts, excluding the last layer 
                //     bool metal1p = is_in_array_gpu(metals, element[i-2], num_metals) 
                //                                     && (i-2 > ((num_layers_contact - 1)*num_source_inj))
                //                                     && (i-2 < (Natom - (num_layers_contact - 1)*num_ground_ext)); 

                //     bool metal2p = is_in_array_gpu(metals, element[j-2], num_metals)
                //                                     && (j-2 > ((num_layers_contact - 1)*num_source_inj))
                //                                     && (j-2 < (Natom - (num_layers_contact - 1)*num_ground_ext));  

                //     // types of tunnelling conditions considered
                //     bool trap_to_trap = (any_vacancy1 && any_vacancy2);
                //     bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
                //     bool contact_to_contact = (metal1p && metal2p);
                //     double local_E_drop = atom_CB_edge[i-2] - atom_CB_edge[j-2];                

                //     if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
                //     {
                //         nnz_row++;
                //     }
                // }
            }
        }
        __syncthreads();
        atomicAdd(&nnz_per_row_d[row], nnz_row); 
    }

}


__global__ void assemble_T_col_indices(const double *posx_d, const double *posy_d, const double *posz_d,
                                        const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge,
                                        const double *lattice, bool pbc, double nn_dist, const double tol,
                                        int num_source_inj, int num_ground_ext, const int num_layers_contact,
                                        int num_metals, int N_sub, int *row_ptr_d, int *col_indices_d,
                                        int block_size_i, int block_size_j, int block_start_i, int block_start_j)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int Natom = N_sub - 1; 

    for(int row = idx; row < block_size_i; row += blockDim.x * gridDim.x){  // Nsub x Nsub matrix divided into blocks by rank
        int nnz_row = 0;
        for(int col = 0; col < block_size_j; col++){

            int i = block_start_i + row; // this is where this rank's rows start, i indexes the matrix and i-2 indexes the atoms
            int j = block_start_j + col; 

            if ( i == j ) // all diagonal terms
            {
                col_indices_d[row_ptr_d[row] + nnz_row] = col;
                nnz_row++;
            }

            if ( (i == 0 && j == 1)  || (i == 1 && j == 0) ) // loop connection
            {
                col_indices_d[row_ptr_d[row] + nnz_row] = col;
                nnz_row++;
            }

            if ( i == 0 && ( j > ((N_sub+1) - num_ground_ext) )) // extraction terms minus ground node
            {
                col_indices_d[row_ptr_d[row] + nnz_row] = col;
                nnz_row++;
            }

            if ( i == 1 && (j > 1) && (j < num_source_inj+2) ) // injection terms minus ground node
            {
                col_indices_d[row_ptr_d[row] + nnz_row] = col;
                nnz_row++;
            }

            if (i > 1 && i != j)
            {
                // source/ground terms
                if ( (j == 1) && (i > 1) && (i < num_source_inj + 2) ) 
                {
                    col_indices_d[row_ptr_d[row] + nnz_row] = col;
                    nnz_row++;
                }
                if ( (j == 0) && ( i > ((N_sub+1) - num_ground_ext) ) )
                {
                    col_indices_d[row_ptr_d[row] + nnz_row] = col;
                    nnz_row++;
                }

                double dist = site_dist_gpu(posx_d[i-2], posy_d[i-2], posz_d[i-2],
                                            posx_d[j-2], posy_d[j-2], posz_d[j-2]);

                // direct terms 
                if ( i != j && j > 1 && dist < nn_dist )
                {
                    col_indices_d[row_ptr_d[row] + nnz_row] = col;
                    nnz_row++;
                }

                // tunneling terms 
                // if ( i != j && dist > nn_dist )
                // { 
                //     bool any_vacancy1 = element[i-2] == VACANCY;
                //     bool any_vacancy2 = element[j-2] == VACANCY;

                //     // contacts, excluding the last layer 
                //     bool metal1p = is_in_array_gpu(metals, element[i-2], num_metals) 
                //                                     && (i-2 > ((num_layers_contact - 1)*num_source_inj))
                //                                     && (i-2 < (Natom - (num_layers_contact - 1)*num_ground_ext)); 

                //     bool metal2p = is_in_array_gpu(metals, element[j-2], num_metals)
                //                                     && (j-2 > ((num_layers_contact - 1)*num_source_inj))
                //                                     && (j-2 < (Natom - (num_layers_contact - 1)*num_ground_ext));  

                //     // types of tunnelling conditions considered
                //     bool trap_to_trap = (any_vacancy1 && any_vacancy2);
                //     bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
                //     bool contact_to_contact = (metal1p && metal2p);
                //     double local_E_drop = atom_CB_edge[i-2] - atom_CB_edge[j-2];                

                //     if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
                //     {
                //         col_indices_d[row_ptr_d[row] + nnz_row] = col;
                //         nnz_row++;
                //     }
                // }
            }
        }
    }
}


__global__ void calc_nnz_per_row_tunnel(const double *posx, const double *posy, const double *posz,
                                        const double *atom_CB_edge, const ELEMENT *element, const int *atom_charge,
                                        const double nn_dist, const double tol, int *tunnel_indices_global, 
                                        int num_layers_contact, int num_source_inj, int num_ground_ext,
                                        const ELEMENT *metals, int num_metals, int num_tunnel_points, int Natom,
                                        int block_size, int block_start, int *nnz_per_row_d)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;
    int Nsub = Natom + 1;

    for(int row_id = idx; row_id < block_size; row_id += blockDim.x * gridDim.x){  // Nsub x Nsub matrix divided into blocks by rank
        int nnz_row = 0;
        for(int col_id = 0; col_id < num_tunnel_points; col_id++){

                int i = block_start + row_id; // this is where this rank's rows start, i indexes the matrix 
                int j = col_id; 

                int ind_i = tunnel_indices_global[i];
                int ind_j = tunnel_indices_global[j];

                //  ACCCESS INDIRECTLY THOUGH TUNNEL INDICES
                double posx_i = posx[ind_i];
                double posx_j = posx[ind_j];

                double posy_i = posy[ind_i];
                double posy_j = posy[ind_j];

                double posz_i = posz[ind_i];
                double posz_j = posz[ind_j];

                double atom_CB_edge_i = atom_CB_edge[ind_i];
                double atom_CB_edge_j = atom_CB_edge[ind_j];

                ELEMENT element_i = element[ind_i];
                ELEMENT element_j = element[ind_j];



                double dist = site_dist_gpu(posx_i, posy_i, posz_i,
                                            posx_j, posy_j, posz_j);

                // all diagonal terms
                if ( i == j )
                {
                    nnz_row++;
                }

                // tunneling terms 
                if ( i != j && dist > nn_dist )
                { 
                    bool any_vacancy1 = element_i == VACANCY;
                    bool any_vacancy2 = element_j == VACANCY;

                    // contacts, excluding the last layer 
                    bool metal1p = is_in_array_gpu(metals, element_i, num_metals)
                                                && (ind_i > ((num_layers_contact - 1)*num_source_inj))
                                                && (ind_i < (Natom - (num_layers_contact - 1)*num_ground_ext)); 

                    bool metal2p = is_in_array_gpu(metals, element_j, num_metals)
                                                && (ind_j > ((num_layers_contact - 1)*num_source_inj))
                                                && (ind_j < (Natom - (num_layers_contact - 1)*num_ground_ext));  

                    // types of tunnelling conditions considered
                    bool trap_to_trap = (any_vacancy1 && any_vacancy2);
                    bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
                    bool contact_to_contact = (metal1p && metal2p);
                    double local_E_drop = atom_CB_edge_i - atom_CB_edge_j;                

                    if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
                    {
                        nnz_row++;
                    }
                }
        }
        nnz_per_row_d[row_id] = nnz_row; 
    }

}

__global__ void assemble_tunnel_col_indices(const double *posx, const double *posy, const double *posz,
                                            const double *atom_CB_edge, const ELEMENT *element, const int *atom_charge,
                                            const double nn_dist, const double tol, int *tunnel_indices_global, 
                                            int num_layers_contact, int num_source_inj, int num_ground_ext,
                                            const ELEMENT *metals, int num_metals, int num_tunnel_points, int Natom,
                                            int block_size, int block_start, int *row_ptr_d, int *col_indices_d)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;
    int Nsub = Natom + 1;

    for(int row_id = idx; row_id < block_size; row_id += blockDim.x * gridDim.x){  // Nsub x Nsub matrix divided into blocks by rank
        int nnz_row = 0;
        for(int col_id = 0; col_id < num_tunnel_points; col_id++){

                int i = block_start + row_id; // this is where this rank's rows start, i indexes the matrix 
                int j = col_id; 

                int ind_i = tunnel_indices_global[i];
                int ind_j = tunnel_indices_global[j];

                //  ACCCESS INDIRECTLY THOUGH TUNNEL INDICES
                double posx_i = posx[ind_i];
                double posx_j = posx[ind_j];

                double posy_i = posy[ind_i];
                double posy_j = posy[ind_j];

                double posz_i = posz[ind_i];
                double posz_j = posz[ind_j];

                double atom_CB_edge_i = atom_CB_edge[ind_i];
                double atom_CB_edge_j = atom_CB_edge[ind_j];

                ELEMENT element_i = element[ind_i];
                ELEMENT element_j = element[ind_j];



                double dist = site_dist_gpu(posx_i, posy_i, posz_i,
                                            posx_j, posy_j, posz_j);

                // all diagonal terms
                if ( i == j )
                {
                    col_indices_d[row_ptr_d[row_id] + nnz_row] = col_id;
                    nnz_row++;
                }

                // tunneling terms 
                if ( i != j && dist > nn_dist )
                { 
                    bool any_vacancy1 = element_i == VACANCY;
                    bool any_vacancy2 = element_j == VACANCY;

                    // contacts, excluding the last layer 
                    bool metal1p = is_in_array_gpu(metals, element_i, num_metals)
                                                && (ind_i > ((num_layers_contact - 1)*num_source_inj))
                                                && (ind_i < (Natom - (num_layers_contact - 1)*num_ground_ext)); 

                    bool metal2p = is_in_array_gpu(metals, element_j, num_metals)
                                                && (ind_j > ((num_layers_contact - 1)*num_source_inj))
                                                && (ind_j < (Natom - (num_layers_contact - 1)*num_ground_ext));  

                    // types of tunnelling conditions considered
                    bool trap_to_trap = (any_vacancy1 && any_vacancy2);
                    bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
                    bool contact_to_contact = (metal1p && metal2p);
                    double local_E_drop = atom_CB_edge_i - atom_CB_edge_j;                

                    if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
                    {
                        col_indices_d[row_ptr_d[row_id] + nnz_row] = col_id;
                        nnz_row++;
                    }
                }
        }
    }

}


__global__ void populate_T_tunnel_dist(const double *posx, const double *posy, const double *posz,
                                       const ELEMENT *metals, const ELEMENT *element, const int *atom_charge, const double *atom_CB_edge, 
                                       const double nn_dist, const double tol, const double high_G, const double low_G, const double loop_G, 
                                       const double Vd, const double m_e, const double V0,
                                       int num_layers_contact, int num_source_inj, int num_ground_ext,
                                       int num_metals, const int num_tunnel_points, int *tunnel_indices, 
                                       int *row_ptr_d, int *col_indices_d, double *data_d, int Natom,
                                       int block_size, int block_start)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_total = blockDim.x * gridDim.x;
    int Nsub = Natom + 1;

    for(int row_id = idx; row_id < block_size; row_id += blockDim.x * gridDim.x){  // Nsub x Nsub matrix divided into blocks by rank
        int nnz_row = 0;
        for(int col_id = row_ptr_d[row_id]; col_id < row_ptr_d[row_id+1]; col_id++){ 
        

            int i = block_start + row_id; // this is where this rank's rows start, i indexes the matrix 
            int j = col_indices_d[col_id]; 

            //  ACCCESS INDIRECTLY THOUGH TUNNEL INDICES
            double posx_i = posx[tunnel_indices[i]];
            double posx_j = posx[tunnel_indices[j]];

            double posy_i = posy[tunnel_indices[i]];
            double posy_j = posy[tunnel_indices[j]];

            double posz_i = posz[tunnel_indices[i]];
            double posz_j = posz[tunnel_indices[j]];

            double atom_CB_edge_i = atom_CB_edge[tunnel_indices[i]];
            double atom_CB_edge_j = atom_CB_edge[tunnel_indices[j]];

            ELEMENT element_i = element[tunnel_indices[i]];
            ELEMENT element_j = element[tunnel_indices[j]];

            int ind_i = tunnel_indices[i];
            int ind_j = tunnel_indices[j];

            double dist_angstrom = site_dist_gpu(posx_i, posy_i, posz_i, 
                                                 posx_j, posy_j, posz_j);

                bool neighbor = (dist_angstrom < nn_dist) && (i != j);

                // tunneling terms occur between not-neighbors
                if (i != j && !neighbor)
                {  
                bool any_vacancy1 = element_i == VACANCY;
                bool any_vacancy2 = element_j == VACANCY;

                // contacts, excluding the last layer 
                bool metal1p = is_in_array_gpu(metals, element_i, num_metals)
                                            && (ind_i > ((num_layers_contact - 1)*num_source_inj))
                                            && (ind_i < (Natom - (num_layers_contact - 1)*num_ground_ext)); 

                bool metal2p = is_in_array_gpu(metals, element_j, num_metals)
                                            && (ind_j > ((num_layers_contact - 1)*num_source_inj))
                                            && (ind_j < (Natom - (num_layers_contact - 1)*num_ground_ext));  

                // types of tunnelling conditions considered
                bool trap_to_trap = (any_vacancy1 && any_vacancy2);
                bool contact_to_trap = (any_vacancy1 && metal2p) || (any_vacancy2 && metal1p);
                bool contact_to_contact = (metal1p && metal2p);
                double local_E_drop = atom_CB_edge_i - atom_CB_edge_j;                

                if ((trap_to_trap || contact_to_trap || contact_to_contact)  && (fabs(local_E_drop) > tol))
                {
                    double prefac = -(sqrt( 2 * m_e ) / h_bar) * (2.0 / 3.0);           // [s/(kg^1/2 * m^2)] coefficient inside the exponential
                    double dist = (1e-10)*dist_angstrom;                                // [m] 3D distance between atoms i and j

                    if (contact_to_trap)
                    {
                        double energy_window = fabs(local_E_drop);                      // [eV] energy window for tunneling from the contacts
                        double dV = 0.01;                                               // [V] energy spacing for numerical integration
                        double dE = eV_to_J * dV;                                       // [eV] energy spacing for numerical integration
                        // double dE = eV_to_J * dV * 10; // NOTE: @Manasa this is a temporary fix to avoid MPI issues!


                        // integrate over all the occupied energy levels in the contact
                        double T = 0.0;
                        for (double iv = 0; iv < energy_window; iv += dE)
                        {
                            double E1 = eV_to_J * V0 + iv;                                  // [J] Energy distance to CB before tunnelling
                            double E2 = E1 - fabs(local_E_drop);                            // [J] Energy distance to CB after tunnelling

                            if (E2 > 0)                                                     // trapezoidal potential barrier (low field)                 
                            {                                                           
                                T += exp(prefac * (dist / fabs(local_E_drop)) * ( pow(E1, 1.5) - pow(E2, 1.5) ) );
                            }

                            if (E2 < 0)                                                      // triangular potential barrier (high field)                               
                            {
                                T += exp(prefac * (dist / fabs(local_E_drop)) * ( pow(E1, 1.5) )); 
                            } 
                        }
                        data_d[col_id] = -T;
                    } 
                    else 
                    {
                        double E1 = eV_to_J * V0;                                        // [J] Energy distance to CB before tunnelling
                        double E2 = E1 - fabs(local_E_drop);                             // [J] Energy distance to CB after tunnelling
                                        
                        if (E2 > 0)                                                      // trapezoidal potential barrier (low field)
                        {                                                           
                            double T = exp(prefac * (dist / fabs(E1 - E2)) * ( pow(E1, 1.5) - pow(E2, 1.5) ) );
                            data_d[col_id] = -T;
                        }

                        if (E2 < 0)                                                        // triangular potential barrier (high field)
                        {
                            double T = exp(prefac * (dist / fabs(E1 - E2)) * ( pow(E1, 1.5) ));
                            data_d[col_id] = -T;
                        }
                    }
                }
            }
        }
    }
}


// Collect the indices of the contacts and the vacancies
__global__ void get_is_tunnel_mpi(int *is_tunnel, int *tunnel_indices, const ELEMENT *element, 
                                  int Nsub, int num_layers_contact, int num_source_inj, int num_ground_ext, int counts, int displacements)
{
    int total_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
    int N_atom = Nsub - 1;

    for (int idx_local = total_tid; idx_local < counts; idx_local += total_threads)
    {
        int yes = 0; 
        int idx = idx_local + displacements - 2;

        if (idx >= 0 && idx < Nsub-1)
        {
            // vacancies and contacts have states within the bandgap which are included in the tunneling model
            // include the first layer of the contacts, as the rest are directly connected to it
            // METALS ARE HARDCODED

            if ( element[idx] == VACANCY || 
            ( (element[idx] == Ti_EL || element[idx] == N_EL) &&  (idx > (num_layers_contact - 1)*num_source_inj) && (idx < (N_atom - (num_layers_contact - 1)*num_ground_ext)) )) 
            {
                yes = 1;
            }

            is_tunnel[idx_local] = yes;
            tunnel_indices[idx_local] = yes * idx;
        }

    }
}

// template <typename T>
__global__ void shift_vector_by_constant(int *vec, int constant, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += total_threads)
    {
        vec[i] += constant;
    }
}


__global__ void calc_diagonal_T_tunnel( int *col_indices, int *row_ptr, double *data, int matrix_size, int displacement, double *diagonal)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < matrix_size; i += blockDim.x * gridDim.x){ 
        //reduce the elements in the row
        double tmp = 0.0;
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i + displacement != col_indices[j]){
                tmp += data[j];
            }
        }
        //write the sum of the off-diagonals onto the existing diagonal element
        for(int j = row_ptr[i]; j < row_ptr[i+1]; j++){
            if(i + displacement == col_indices[j]){
                data[j] = -tmp;
                diagonal[i] = data[j];
            }
        }
    }
}


int assemble_sparse_T_submatrix(GPUBuffers &gpubuf, const int N_atom, const double nn_dist, int num_source_inj, int num_ground_ext, int num_layers_contact, 
                                 const double high_G, const double low_G, const double loop_G, const double Vd, const double m_e, const double V0,
                                 Distributed_subblock_sparse &T_tunnel, Distributed_matrix *T_neighbor, double *&diag_tunnel_local,
                                 int *&tunnel_indices_local_d)
{
    // The tunnel submatrix has size num_tunnel_points x num_tunnel_points
    // it is distributed over rows, NOT over blocks

    // The tunnel indices have the size of Nsub
    int Nsub = N_atom + 1;
    int rank = T_neighbor->rank;
    int size = T_neighbor->size;
    int counts_this_rank = T_neighbor->counts[rank];
    int disp_this_rank = T_neighbor->displacements[rank];

    // Collect the indices of the contacts and the vacancies    
    int num_threads = 1024;
    int num_blocks = (counts_this_rank - 1) / num_threads + 1;

    // indices of the tunneling connections (contacts and vacancies) in the Natom array
    int *is_tunnel; // [0, 1, 0, 0, 1...] where 1 indicates a tunnel connection
    int *is_tunnel_indices; // [0, 1, 0, 0, 4...] storing the indices of the tunnel connections
    
    gpuErrchk( hipMalloc((void **)&is_tunnel, counts_this_rank * sizeof(int)) );    
    gpuErrchk( hipMalloc((void **)&is_tunnel_indices, counts_this_rank * sizeof(int)) );  
    hipMemset(is_tunnel, 0, counts_this_rank * sizeof(int));
    hipMemset(is_tunnel_indices, 0, counts_this_rank * sizeof(int));

    hipLaunchKernelGGL(get_is_tunnel_mpi, num_blocks, num_threads, 0, 0, is_tunnel, is_tunnel_indices, gpubuf.atom_element, Nsub, num_layers_contact, num_source_inj, num_ground_ext,
                       counts_this_rank, disp_this_rank);
    gpuErrchk( hipPeekAtLastError() );

    int num_tunnel_points_local;
    num_tunnel_points_local = thrust::reduce(thrust::device, is_tunnel, is_tunnel + counts_this_rank, 0); // sum([0, 1, 0, 0, 1...])

    // allreduce num_tunnel_points_local
    int num_tunnel_points_global;
    MPI_Allreduce(&num_tunnel_points_local, &num_tunnel_points_global, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
    std::cout << "size of tunneling submatrix: " << num_tunnel_points_global << "\n";

    // allgather the num_tunnel_points_local for every rank
    int *counts_subblock = new int[size];
    int *displ_subblock = new int[size];
    MPI_Allgather(&num_tunnel_points_local, 1, MPI_INT, counts_subblock, 1, MPI_INT, MPI_COMM_WORLD);
    displ_subblock[0] = 0;
    for(int i = 1; i < size; i++){
        displ_subblock[i] = displ_subblock[i-1] + counts_subblock[i-1];
    }

    // assemble the local indices into the atoms array for the peice owned by each rank
    // int *tunnel_indices_local_d; // [1, 4...]
    gpuErrchk( hipMalloc((void **)&tunnel_indices_local_d, num_tunnel_points_local * sizeof(int)) );    
    thrust::copy_if(thrust::device, is_tunnel_indices, is_tunnel_indices + counts_this_rank, tunnel_indices_local_d, is_not_zero());
    
    // for the row-wise kernel, every rank needs to iterate over all columns, so we allgather the tunnel indices
    // make global tunnel indices device poinwe
    int *tunnel_indices_global_d;
    gpuErrchk( hipMalloc((void **)&tunnel_indices_global_d, num_tunnel_points_global * sizeof(int)) );
    hipDeviceSynchronize();
    MPI_Allgatherv(tunnel_indices_local_d, num_tunnel_points_local, MPI_INT, tunnel_indices_global_d, counts_subblock, displ_subblock, MPI_INT, MPI_COMM_WORLD);

    int *tunnel_indices_global_h = new int[num_tunnel_points_global];
    // sum them up
    gpuErrchk( hipMemcpy(tunnel_indices_global_h, tunnel_indices_global_d, num_tunnel_points_global * sizeof(int), hipMemcpyDeviceToHost) );
    int sum_indices = 0;
    for(int i = 0; i < num_tunnel_points_global; i++){
        sum_indices += tunnel_indices_global_h[i];
    }
    std::cout << "rank: " << rank << " sum_indices: " << sum_indices << std::endl;

    // make the nnz vector for each rank:    
    // loop over the size to determine neighbours
    int *dist_nnz_per_row_d;

    //sync device
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMalloc((void **)&dist_nnz_per_row_d, counts_subblock[rank] * sizeof(int)) );
    gpuErrchk(hipMemset(dist_nnz_per_row_d, 0, counts_subblock[rank] * sizeof(int)));

    int threads = 1024;
    //start with self
    int blocks = (counts_subblock[rank] - 1) / threads + 1;
    double tol = eV_to_J * 0.01;                                                                // [eV] tolerance after which the barrier slope is considered
    int num_metals = 2;
    MPI_Barrier(MPI_COMM_WORLD);
    hipLaunchKernelGGL(calc_nnz_per_row_tunnel, blocks, threads, 0, 0, 
                        gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z, gpubuf.atom_CB_edge,
                        gpubuf.atom_element, gpubuf.atom_charge,
                        nn_dist, tol,
                        tunnel_indices_global_d,
                        num_layers_contact, num_source_inj, num_ground_ext,
                        gpubuf.metal_types, num_metals,
                        num_tunnel_points_global,
                        N_atom,
                        counts_subblock[rank],
                        displ_subblock[rank],
                        dist_nnz_per_row_d);
    gpuErrchk( hipPeekAtLastError() );

    // int *dist_nnz_per_row_global_d;
    // gpuErrchk( hipMalloc((void **)&dist_nnz_per_row_global_d, num_tunnel_points_global * sizeof(int)) );
    // std::cout << counts_subblock[rank] << std::endl;
    // std::cout << displ_subblock[rank] << std::endl;

    // hipDeviceSynchronize();
    // MPI_Allgatherv(dist_nnz_per_row_d, counts_subblock[rank], MPI_INT, dist_nnz_per_row_global_d, counts_subblock, displ_subblock, MPI_INT, MPI_COMM_WORLD);

    // if(rank == 0){
    //     int *dist_nnz_per_row_global_h = new int[num_tunnel_points_global];
    //     gpuErrchk( hipMemcpy(dist_nnz_per_row_global_h, dist_nnz_per_row_global_d, num_tunnel_points_global * sizeof(int), hipMemcpyDeviceToHost) );
    //     std::string name3 = "dist_nnz_per_row_global_h_" + std::to_string(size) + ".txt";
    //     std::ofstream file3(name3);
    //     for (int i = 0; i < num_tunnel_points_global; i++){
    //         file3 << dist_nnz_per_row_global_d[i] << " ";
    //     }
    //     file3.close();
    // }


    MPI_Barrier(MPI_COMM_WORLD);
    // ** Verified that the nnz_per_row is correct for 1 rank and 3 ranks

    int *row_ptr_subblock_d;
    gpuErrchk( hipMalloc((void **)&row_ptr_subblock_d, (counts_subblock[rank] + 1) * sizeof(int)) );    

    // // create row ptr    
    gpuErrchk(hipMemset(row_ptr_subblock_d, 0, (counts_subblock[rank] + 1) * sizeof(int)));
    void     *temp_storage_d = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes,
        dist_nnz_per_row_d, row_ptr_subblock_d+1, counts_subblock[rank]);
    gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
    hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes,
        dist_nnz_per_row_d, row_ptr_subblock_d+1, counts_subblock[rank]);
    gpuErrchk(hipFree(temp_storage_d));

    // ** Verified that the row ptrs are correct for 1 rank and 3 ranks

    // copy the last element of row_ptr_subblock_d back to host
    int nnz_subblock_local;
    gpuErrchk( hipMemcpy(&nnz_subblock_local, row_ptr_subblock_d + counts_subblock[rank], sizeof(int), hipMemcpyDeviceToHost) );

    int nnz_subblock[1];
    hipDeviceSynchronize();
    MPI_Allreduce(&nnz_subblock_local, nnz_subblock, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
    std::cout << "rank: " << rank << " nnz_subblock: " << nnz_subblock[0] << std::endl;

    MPI_Barrier(MPI_COMM_WORLD);        // remove

    // make col indices
    int *col_indices_subblock_d;
    gpuErrchk( hipMalloc((void **)&col_indices_subblock_d, nnz_subblock_local * sizeof(int)) );
    int *tmp = new int[nnz_subblock_local];
    for(int i = 0; i < nnz_subblock_local; i++){
        tmp[i] = i+8;
    }
    hipMemcpy(col_indices_subblock_d, tmp, nnz_subblock_local * sizeof(int), hipMemcpyHostToDevice);

    hipLaunchKernelGGL(assemble_tunnel_col_indices, blocks, threads, 0, 0, 
                        gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z, gpubuf.atom_CB_edge,
                        gpubuf.atom_element, gpubuf.atom_charge,
                        nn_dist, tol,
                        tunnel_indices_global_d,
                        num_layers_contact, num_source_inj, num_ground_ext,
                        gpubuf.metal_types, num_metals,
                        num_tunnel_points_global,
                        N_atom,
                        counts_subblock[rank],
                        displ_subblock[rank],
                        row_ptr_subblock_d, col_indices_subblock_d);

    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    // make the values (remmember the x10 for numerical stability)
    double *data_d;
    gpuErrchk( hipMalloc((void **)&data_d, nnz_subblock_local * sizeof(double)) );
    hipLaunchKernelGGL(populate_T_tunnel_dist, blocks, threads, 0, 0, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                        gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                        nn_dist, tol, high_G, low_G, loop_G, Vd, m_e, V0,
                        num_layers_contact, num_source_inj, num_ground_ext,
                        num_metals, num_tunnel_points_local,
                        tunnel_indices_global_d,
                        row_ptr_subblock_d, col_indices_subblock_d, data_d, N_atom,
                        counts_subblock[rank], displ_subblock[rank]);

    // add + 2 to tunnel inds before calling constructor, so now tunnel-indices will index the full matrix of Nsub x Nsub
    // int two = 2;
    hipLaunchKernelGGL(shift_vector_by_constant, blocks, threads, 0, 0, tunnel_indices_local_d,
        2-disp_this_rank, counts_subblock[rank]);

    // // dump the tunnel_indices_local_d to file 
    // int *tunnel_indices_local_h = (int *)calloc(counts_subblock[rank], sizeof(int));
    // gpuErrchk( hipMemcpy(tunnel_indices_local_h, tunnel_indices_local_d, counts_subblock[rank] * sizeof(int), hipMemcpyDeviceToHost) );
    // std::string name2 = "tunnel_indices_local_d_" + std::to_string(gpubuf.rank) + ".txt";
    // std::ofstream file2(name2);
    // for (int i = 0; i < counts_subblock[rank]; i++){
    //     file2 << tunnel_indices_local_h[i] << " ";
    // }
    // file2.close();

    // MPI_Barrier(MPI_COMM_WORLD);
    // exit(1);


    std::cout << "shifted tunnel indices" << std::endl;

    // row reduce the diagonals - diag_tunnel_local is passed in
    gpuErrchk( hipMalloc((void **)&diag_tunnel_local, counts_subblock[rank] * sizeof(double)) );
    hipLaunchKernelGGL(calc_diagonal_T_tunnel,
        blocks, threads, 0, 0, col_indices_subblock_d, row_ptr_subblock_d, data_d,
        counts_subblock[rank], displ_subblock[rank], diag_tunnel_local);


    // load data into subblock struct:
    rocsparse_spmat_descr subblock_descriptor;
    rocsparse_dnvec_descr subblock_vector_descriptor_in;
    rocsparse_dnvec_descr subblock_vector_descriptor_out;
    rocsparse_spmv_alg algo = rocsparse_spmv_alg_csr_adaptive;
    size_t subblock_buffersize;

    std::cout << "going to descr" << std::endl;

    rocsparse_create_csr_descr(&subblock_descriptor,
                                counts_subblock[rank],
                                num_tunnel_points_local,
                                nnz_subblock_local,
                                row_ptr_subblock_d,
                                col_indices_subblock_d,
                                data_d,
                                rocsparse_indextype_i32,
                                rocsparse_indextype_i32,
                                rocsparse_index_base_zero,
                                rocsparse_datatype_f64_r);


    rocsparse_handle rocsparse_handle;
    rocsparse_create_handle(&rocsparse_handle);

    std::cout << "going to spmv" << std::endl;

    double alpha = 1.0;
    double beta = 0.0;
    rocsparse_spmv(rocsparse_handle,
                    rocsparse_operation_none,
                    &alpha,
                    subblock_descriptor,
                    subblock_vector_descriptor_in,
                    &beta,
                    subblock_vector_descriptor_out,
                    rocsparse_datatype_f64_r,
                    algo,
                    &subblock_buffersize,
                    nullptr);
    double *subblock_buffer_d;
    hipMalloc(&subblock_buffer_d, subblock_buffersize);

    std::cout << "going to subblock" << std::endl;

    // Distributed_subblock_sparse A_subblock;
    T_tunnel.subblock_indices_local_d = tunnel_indices_local_d;
    T_tunnel.descriptor = &subblock_descriptor;
    T_tunnel.algo = algo;
    T_tunnel.buffersize = &subblock_buffersize;
    T_tunnel.buffer_d = subblock_buffer_d;
    T_tunnel.subblock_size = num_tunnel_points_local;
    T_tunnel.count_subblock_h = counts_subblock;
    T_tunnel.displ_subblock_h = displ_subblock;
    T_tunnel.send_subblock_requests = new MPI_Request[size-1];
    T_tunnel.recv_subblock_requests = new MPI_Request[size-1];
    T_tunnel.streams_recv_subblock = new hipStream_t[size-1];
    for(int i = 0; i < size-1; i++){
        hipStreamCreate(&T_tunnel.streams_recv_subblock[i]);
    }

    // dump_csr_matrix_txt(num_tunnel_points_local, nnz_subblock_local, row_ptr_subblock_d, col_indices_subblock_d, data_d, size+rank);
    // MPI_Barrier(MPI_COMM_WORLD);
    // exit(1); 



    // delete stuff
    // TODO delete outside
    // delete[] counts_subblock;
    // delete[] displ_subblock;
    // gpuErrchk( hipFree(tunnel_indices_local_d) );
    // gpuErrchk( hipFree(row_ptr_subblock_d) );
    // gpuErrchk( hipFree(col_indices_subblock_d) );
    // gpuErrchk( hipFree(data_d) );

    gpuErrchk( hipFree(is_tunnel) );
    gpuErrchk( hipFree(is_tunnel_indices) );
    gpuErrchk( hipFree(tunnel_indices_global_d) );
    gpuErrchk( hipFree(dist_nnz_per_row_d) );

    rocsparse_destroy_handle(rocsparse_handle);

    std::cout << "freed memory inside sparsity T" << std::endl;

    return num_tunnel_points_local;
}

void initialize_sparsity_T(GPUBuffers &gpubuf,
    int pbc, const double nn_dist, int num_source_inj, int num_ground_ext, int num_layers_contact, KMC_comm &kmc_comm)
{
    // copy atom arrays:
    int *gpu_index;
    int *atom_gpu_index;
    gpuErrchk( hipMalloc((void **)&gpu_index, gpubuf.N_ * sizeof(int)) );                                           // indices of the site array
    gpuErrchk( hipMalloc((void **)&atom_gpu_index, gpubuf.N_ * sizeof(int)) );                                      // indices of the atom array
    thrust::device_ptr<int> gpu_index_ptr = thrust::device_pointer_cast(gpu_index);
    thrust::sequence(gpu_index_ptr, gpu_index_ptr + gpubuf.N_, 0);
    double *last_atom = thrust::copy_if(thrust::device, gpubuf.site_x, gpubuf.site_x + gpubuf.N_, gpubuf.site_element, gpubuf.atom_x, is_defect());
    int N_atom = last_atom - gpubuf.atom_x;
    thrust::copy_if(thrust::device, gpubuf.site_y, gpubuf.site_y + gpubuf.N_, gpubuf.site_element, gpubuf.atom_y, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_z, gpubuf.site_z + gpubuf.N_, gpubuf.site_element, gpubuf.atom_z, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_charge, gpubuf.site_charge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_charge, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_element, gpubuf.site_element + gpubuf.N_, gpubuf.site_element, gpubuf.atom_element, is_defect());
    thrust::copy_if(thrust::device, gpubuf.site_CB_edge, gpubuf.site_CB_edge + gpubuf.N_, gpubuf.site_element, gpubuf.atom_CB_edge, is_defect());
    thrust::copy_if(thrust::device, gpu_index, gpu_index + gpubuf.N_, gpubuf.site_element, atom_gpu_index, is_defect());

    int N_sub = N_atom + 1;
    int rank = kmc_comm.rank_T;
    int size = kmc_comm.size_T;
    int rows_this_rank = kmc_comm.counts_T[rank];
    int disp_this_rank = kmc_comm.displs_T[rank];
    
    int *dist_nnz_h = new int[size];
    int *dist_nnz_d;
    int *dist_nnz_per_row_d;

    gpuErrchk( hipMalloc((void **)&dist_nnz_d, size * sizeof(int)) );
    gpuErrchk(hipMemset(dist_nnz_d, 0, size * sizeof(int)));
    gpuErrchk( hipMalloc((void **)&dist_nnz_per_row_d, size * rows_this_rank * sizeof(int)) );
    gpuErrchk(hipMemset(dist_nnz_per_row_d, 0, size * rows_this_rank * sizeof(int)));

    // Assemble the sparsity pattern

    // loop over the size to determine neighbours
    for(int i = 0; i < size; i++){
        int rows_other = kmc_comm.counts_T[i];
        int displ_other = kmc_comm.displs_T[i];

        int threads = 1024;
        //start with self
        int blocks = (rows_this_rank - 1) / threads + 1;

        double tol = eV_to_J * 0.01;                                                                // [eV] tolerance after which the barrier slope is considered
        int num_metals = 2;
        hipLaunchKernelGGL(calc_nnz_per_row_T, blocks, threads, 0, 0, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                           gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_CB_edge, gpubuf.lattice, pbc,
                           nn_dist, tol, num_source_inj, num_ground_ext, num_layers_contact,
                           num_metals, N_sub, rows_this_rank, rows_other, disp_this_rank, displ_other, dist_nnz_per_row_d + i * rows_this_rank);

        // reduce nnz per row
        void     *temp_storage_d = NULL;
        size_t   temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(
        temp_storage_d, temp_storage_bytes, 
            dist_nnz_per_row_d + i * rows_this_rank,
            dist_nnz_d + i, rows_this_rank);

        // Allocate temporary storage
        hipMalloc(&temp_storage_d, temp_storage_bytes);

        // Run sum-reduction
        hipcub::DeviceReduce::Sum(temp_storage_d, temp_storage_bytes,
            dist_nnz_per_row_d + i * rows_this_rank,
            dist_nnz_d + i, rows_this_rank);

    }

    gpuErrchk( hipMemcpy(dist_nnz_h, dist_nnz_d, size * sizeof(int), hipMemcpyDeviceToHost) );
    // counting neighbours
    int neighbor_count = 0;
    for(int i = 0; i < size; i++){
        if(dist_nnz_h[i] > 0){
            neighbor_count++;
        }
    }

    std::cout << "got neighbor nnz " << std::endl;

    // print dist_nnz_h:
    for (int i = 0; i < size; i++)
    {
        std::cout << "rank " << rank << "T dist_nnz_h[" << i << "] = " << dist_nnz_h[i] << std::endl;
    }
    std::cout << "rank " << rank <<  "T neighbor_count = " << neighbor_count << std::endl;

    // get the indices of the neighbours
    int *neighbor_idx = new int[neighbor_count];
    int *neighbor_nnz_h = new int[neighbor_count];
    int *neighbor_nnz_per_row_d;
    gpuErrchk( hipMalloc((void **)&neighbor_nnz_per_row_d, neighbor_count * rows_this_rank * sizeof(int)) );

    // determine neighbours
    neighbor_count = 0;
    for(int i = 0; i < size; i++){
        int neighbor = (i+rank) % size;
        if(dist_nnz_h[neighbor] > 0){
            neighbor_idx[neighbor_count] = neighbor;
            neighbor_count++;
        }
    }    

    std::cout << "rank " << rank <<  "T neighbor_idx = " << neighbor_count << std::endl;   

    // fill the neighbor nnz
    for(int i = 0; i < neighbor_count; i++){
        neighbor_nnz_h[i] = dist_nnz_h[neighbor_idx[i]];
        gpuErrchk( hipMemcpy(neighbor_nnz_per_row_d + i * rows_this_rank,
            dist_nnz_per_row_d + neighbor_idx[i] * rows_this_rank,
            rows_this_rank * sizeof(int), hipMemcpyHostToDevice) );
    }

    // alloc memory
    int **col_indices_d = new int*[neighbor_count];
    int **row_ptr_d = new int*[neighbor_count];
    for(int i = 0; i < neighbor_count; i++){
        gpuErrchk( hipMalloc((void **)&col_indices_d[i], neighbor_nnz_h[i] * sizeof(int)) );
        gpuErrchk( hipMalloc((void **)&row_ptr_d[i], (rows_this_rank + 1) * sizeof(int)) );
    }
    
    // create row ptr
    for(int i = 0; i < neighbor_count; i++){

        gpuErrchk(hipMemset(row_ptr_d[i], 0, (rows_this_rank + 1) * sizeof(int)));
        void     *temp_storage_d = NULL;
        size_t   temp_storage_bytes = 0;
        // determines temporary device storage requirements for inclusive prefix sum
        hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes,
            neighbor_nnz_per_row_d + i * rows_this_rank, (row_ptr_d[i])+1, rows_this_rank);

        // Allocate temporary storage for inclusive prefix sum
        gpuErrchk(hipMalloc(&temp_storage_d, temp_storage_bytes));
        // Run inclusive prefix sum
        // inclusive sum starting at second value to get the row ptr
        // which is the same as inclusive sum starting at first value and last value filled with nnz
        hipcub::DeviceScan::InclusiveSum(temp_storage_d, temp_storage_bytes,
            neighbor_nnz_per_row_d + i * rows_this_rank, (row_ptr_d[i])+1, rows_this_rank);

        // Free temporary storage
        gpuErrchk(hipFree(temp_storage_d)); 

    }

    // column indices
    for(int i = 0; i < neighbor_count; i++){
        int neighbour = neighbor_idx[i];
        int rows_neighbour = kmc_comm.counts_T[neighbour];
        int disp_neighbour = kmc_comm.displs_T[neighbour];

        int threads = 1024;
        int blocks = (rows_this_rank + threads - 1) / threads;
       
        double tol = eV_to_J * 0.01;                                                                // [eV] tolerance after which the barrier slope is considered
        int num_metals = 2;
        hipLaunchKernelGGL(assemble_T_col_indices, blocks, threads, 0, 0, gpubuf.atom_x, gpubuf.atom_y, gpubuf.atom_z,
                           gpubuf.metal_types, gpubuf.atom_element, gpubuf.atom_charge, gpubuf.atom_CB_edge,
                           gpubuf.lattice, pbc, nn_dist, tol,
                           num_source_inj, num_ground_ext, num_layers_contact,
                           num_metals, N_sub,
                           row_ptr_d[i],
                           col_indices_d[i], rows_this_rank, rows_neighbour, disp_this_rank, disp_neighbour);
        gpuErrchk( hipDeviceSynchronize() );
    }

    gpubuf.T_distributed = new Distributed_matrix(
        N_sub,
        kmc_comm.counts_T,
        kmc_comm.displs_T,
        neighbor_count,
        neighbor_idx,
        col_indices_d,
        row_ptr_d,
        neighbor_nnz_h,
        rocsparse_spmv_alg_csr_adaptive,
        kmc_comm.group_T
    );

    gpubuf.T_p_distributed = new Distributed_vector(
        N_sub,
        kmc_comm.counts_T,
        kmc_comm.displs_T,
        gpubuf.T_distributed->number_of_neighbours,
        gpubuf.T_distributed->neighbours,
        kmc_comm.group_T
    );

    for(int i = 0; i < neighbor_count; i++){
        gpuErrchk( hipFree(col_indices_d[i]) );
        gpuErrchk( hipFree(row_ptr_d[i]) );
    }   

    delete[] col_indices_d;
    delete[] row_ptr_d;
    delete[] neighbor_idx;
    delete[] dist_nnz_h;

    gpuErrchk( hipFree(dist_nnz_d) );
    gpuErrchk( hipFree(dist_nnz_per_row_d) );
    delete[] neighbor_nnz_h;
    gpuErrchk( hipFree(neighbor_nnz_per_row_d) );    //FREE MEMORY
    std::cout << "rank " << rank <<  "initialized sparsity of peice of T " << neighbor_count << std::endl;   

}