#include "hip/hip_runtime.h"
#include "gpu_solvers.h"
#include <omp.h>
// Constants needed:
constexpr double kB = 8.617333262e-5;           // [eV/K]

#define NUM_THREADS 512
#define MAX_NUM_LAYERS 5

// in GPU cache
__constant__ double E_gen_const[MAX_NUM_LAYERS];
__constant__ double E_rec_const[MAX_NUM_LAYERS];
__constant__ double E_Vdiff_const[MAX_NUM_LAYERS];
__constant__ double E_Odiff_const[MAX_NUM_LAYERS];

void get_gpu_info(char *gpu_string, int dev){
    struct hipDeviceProp_t dprop;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(dev);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Error: %s\n", hipGetErrorString(cudaStatus));
        // Handle the error or exit the program
        exit(EXIT_FAILURE);
    }

    hipGetDeviceProperties(&dprop, dev);
    strcpy(gpu_string,dprop.name);
}

void set_gpu(int dev){
 hipSetDevice(dev);
}

__global__ void build_event_list(const int N, const int nn, const int *neigh_idx, 
                                 const int *layer, const double *lattice, const int pbc, 
                                 const double *T_bg, const double *freq, const double *sigma, const double *k, 
                                 const double *posx, const double *posy, const double *posz,
                                 const double *potential_boundary, const double *potential_charge, const double *temperature,
                                 const ELEMENT *element, const int *charge, EVENTTYPE *event_type, double *event_prob)
{
    int total_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (int idx = total_tid; idx < N * nn; idx += total_threads) {

        EVENTTYPE event_type_ = NULL_EVENT;
        double P = 0.0;

        int i = static_cast<int>(floorf(idx / nn));
        int j = neigh_idx[idx];

        // condition for neighbor existing
        if (j >= 0 && j < N) {
            double dist = 1e-10 * site_dist_gpu(posx[i], posy[i], posz[i], 
                                                posx[j], posy[j], posz[j], 
                                                lattice[0], lattice[1], lattice[2], pbc);

            // Generation
            if (element[i] == DEFECT && element[j] == O_EL)
            {

                double E = 2 * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]));
                double zero_field_energy = E_gen_const[layer[j]]; 
                event_type_ = VACANCY_GENERATION;
                double Ekin = 0; // kB * (temperature[j] - (*T_bg)); //kB * (temperature[j] - temperature[i]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Recombination
            if (element[i] == OXYGEN_DEFECT && element[j] == VACANCY) 
            {
                int charge_abs = 2;
                double self_int_V = v_solve_gpu(dist, charge_abs, sigma, k);

                int charge_state = charge[i] - charge[j];
                double E = charge_state * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]) + (charge_state / 2) * self_int_V);
                double zero_field_energy = E_rec_const[layer[j]];

                event_type_ = VACANCY_RECOMBINATION;
                double Ekin = 0; //kB * (temperature[i] - (*T_bg)); //kB * (temperature[i] - temperature[j]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Vacancy diffusion
            if (element[i] == VACANCY && element[j] == O_EL)
            {

                double self_int_V = 0.0;
                if (charge[i] != 0)
                {
                    self_int_V = v_solve_gpu(dist, charge[i], sigma, k);
                }

                event_type_ = VACANCY_DIFFUSION;
                double E = (charge[i] - charge[j]) * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]) + self_int_V);
                double zero_field_energy = E_Vdiff_const[layer[j]];  
                double Ekin = 0;//kB * (temperature[i] - (*T_bg)); //kB * (temperature[j] - temperature[i]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Ion diffusion
            if (element[i] == OXYGEN_DEFECT && element[j] == DEFECT)
            {
                int charge_abs = 2;
                double self_int_V = 0.0;
                if (charge[i] != 0)
                {                    
                    self_int_V = v_solve_gpu(dist, charge_abs, sigma, k);
                }

                double E = (charge[i] - charge[j]) * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]) - self_int_V);
                double zero_field_energy = E_Odiff_const[layer[j]];

                event_type_ = ION_DIFFUSION;
                double Ekin = 0; //kB * (temperature[i] - (*T_bg)); //kB * (temperature[i] - temperature[j]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }
        }
        event_type[idx] = event_type_;
        event_prob[idx] = P;
    }
}

// builds only part of the event list
// from start_i to start_i + size_i
__global__ void build_event_list_split(const int N, const int size_i, const int start_i,
                                 const int nn, const int *neigh_idx, 
                                 const int *layer, const double *lattice, const int pbc, 
                                 const double *T_bg, const double *freq, const double *sigma, const double *k, 
                                 const double *posx, const double *posy, const double *posz,
                                 const double *potential_boundary, const double *potential_charge, const double *temperature,
                                 const ELEMENT *element, const int *charge, EVENTTYPE *event_type, double *event_prob)
{
    int total_tid = blockIdx.x * blockDim.x + threadIdx.x + start_i;
    int total_threads = blockDim.x * gridDim.x;

    for (int id = total_tid; id < size_i * nn; id += total_threads) {
        EVENTTYPE event_type_ = NULL_EVENT;
        double P = 0.0;

        int idx = id + start_i * nn;
        int i = idx / nn;
        int j = neigh_idx[idx];

        // condition for neighbor existing
        if (j >= 0 && j < N) {
            double dist = 1e-10 * site_dist_gpu(posx[i], posy[i], posz[i], 
                                                posx[j], posy[j], posz[j], 
                                                lattice[0], lattice[1], lattice[2], pbc);

            // Generation
            if (element[i] == DEFECT && element[j] == O_EL)
            {

                double E = 2 * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]));
                double zero_field_energy = E_gen_const[layer[j]]; 
                event_type_ = VACANCY_GENERATION;
                double Ekin = 0; // kB * (temperature[j] - (*T_bg)); //kB * (temperature[j] - temperature[i]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Recombination
            if (element[i] == OXYGEN_DEFECT && element[j] == VACANCY) 
            {
                int charge_abs = 2;
                double self_int_V = v_solve_gpu(dist, charge_abs, sigma, k);

                int charge_state = charge[i] - charge[j];
                double E = charge_state * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]) + (charge_state / 2) * self_int_V);
                double zero_field_energy = E_rec_const[layer[j]];

                event_type_ = VACANCY_RECOMBINATION;
                double Ekin = 0; //kB * (temperature[i] - (*T_bg)); //kB * (temperature[i] - temperature[j]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Vacancy diffusion
            if (element[i] == VACANCY && element[j] == O_EL)
            {

                double self_int_V = 0.0;
                if (charge[i] != 0)
                {
                    self_int_V = v_solve_gpu(dist, charge[i], sigma, k);
                }

                event_type_ = VACANCY_DIFFUSION;
                double E = (charge[i] - charge[j]) * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]) + self_int_V);
                double zero_field_energy = E_Vdiff_const[layer[j]];  
                double Ekin = 0;//kB * (temperature[i] - (*T_bg)); //kB * (temperature[j] - temperature[i]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }

            // Ion diffusion
            if (element[i] == OXYGEN_DEFECT && element[j] == DEFECT)
            {
                int charge_abs = 2;
                double self_int_V = 0.0;
                if (charge[i] != 0)
                {                    
                    self_int_V = v_solve_gpu(dist, charge_abs, sigma, k);
                }

                double E = (charge[i] - charge[j]) * ((potential_boundary[i] + potential_charge[i]) - (potential_boundary[j] + potential_charge[j]) - self_int_V);
                double zero_field_energy = E_Odiff_const[layer[j]];

                event_type_ = ION_DIFFUSION;
                double Ekin = 0; //kB * (temperature[i] - (*T_bg)); //kB * (temperature[i] - temperature[j]);
                double EA = zero_field_energy - E - Ekin;
                P = exp(-1 * EA / (kB * (*T_bg))) * (*freq);
            }
        }
        event_type[id] = event_type_;
        event_prob[id] = P;
    }
}


__global__ void zero_out_events(EVENTTYPE *event_type, double *event_prob, const int *neigh_idx, int N, int nn, int i_to_delete, int j_to_delete){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    int j;
    for (int id = idx; id < N * nn; id += blockDim.x * gridDim.x){
        i = id / nn;
        j = neigh_idx[id];

        if (i == i_to_delete || j == j_to_delete || i == j_to_delete || j == i_to_delete){
            event_type[id] = NULL_EVENT;
            event_prob[id] = 0.0;
        }
    }

}

__global__ void zero_out_events_split(EVENTTYPE *event_type, double *event_prob, const int *neigh_idx,
        const int size_i, const int start_i,
        int nn, int i_to_delete, int j_to_delete)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    int j;
    for (int id = idx; id < size_i * nn; id += blockDim.x * gridDim.x){
        i = id / nn + start_i;
        j = neigh_idx[id+start_i*nn];

        if ( j >=0 && (i == i_to_delete || j == j_to_delete || i == j_to_delete || j == i_to_delete)){
            event_type[id] = NULL_EVENT;
            event_prob[id] = 0.0;
        }
    }

}

double execute_kmc_step_gpu(const int N, const int nn, const int *neigh_idx, const int *site_layer,
                            const double *lattice, const int pbc, const double *T_bg, 
                            const double *freq, const double *sigma, const double *k,
                            const double *posx, const double *posy, const double *posz, 
                            const double *site_potential_boundary, const double *site_potential_charge, const double *site_temperature,
                            ELEMENT *site_element, int *site_charge, RandomNumberGenerator &rng, const int *neigh_idx_host){

    // **************************
    // **** Build Event List ****
    // **************************

    double time_event_list = 0.0;


    // the KMC event list arrays only exist in gpu memory
    EVENTTYPE *event_type; 
    double    *event_prob; 
    gpuErrchk( hipMalloc((void**)&event_type, N * nn * sizeof(EVENTTYPE)) );
    gpuErrchk( hipMalloc((void**)&event_prob, N * nn * sizeof(double)) );
                
    int num_threads = 512;
    int num_blocks = (N * nn - 1) / num_threads + 1;

    // populate the event_type and event_prob arrays:
    build_event_list<<<num_blocks, num_threads>>>(N, nn, neigh_idx, 
                                                  site_layer, lattice, pbc,
                                                  T_bg, freq, sigma, k,
                                                  posx, posy, posz, 
                                                  site_potential_boundary, site_potential_charge, site_temperature, 
                                                  site_element, site_charge, event_type, event_prob);

    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    // **************************
    // ** Event Execution Loop **
    // **************************

    // helper variables:
    // NOTE: INITIALIZE THESE ON GPU AND USE MEMCPY DEVICETODEVICE INSTEAD
    int two_host = 2;
    int two_neg_host = -2;
    int zero_host = 0;
    ELEMENT defect_element_host = DEFECT;
    ELEMENT O_defect_element_host = OXYGEN_DEFECT;
    ELEMENT vacancy_element_host = VACANCY;
    ELEMENT O_element_host = O_EL;

    int *two_d;
    gpuErrchk( hipMalloc((void**)&two_d, 1 * sizeof(int)) );
    gpuErrchk( hipMemcpy(two_d, &two_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
    int *two_neg_d;
    gpuErrchk( hipMalloc((void**)&two_neg_d, 1 * sizeof(int)) );
    gpuErrchk( hipMemcpy(two_neg_d, &two_neg_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
    int *zero_d;
    gpuErrchk( hipMalloc((void**)&zero_d, 1 * sizeof(int)) );
    gpuErrchk( hipMemcpy(zero_d, &zero_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
    ELEMENT *defect_element_d;
    gpuErrchk( hipMalloc((void**)&defect_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(defect_element_d, &defect_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
    ELEMENT *O_defect_element_d;
    gpuErrchk( hipMalloc((void**)&O_defect_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(O_defect_element_d, &O_defect_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
    ELEMENT *vacancy_element_d;
    gpuErrchk( hipMalloc((void**)&vacancy_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(vacancy_element_d, &vacancy_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
    ELEMENT *O_element_d;
    gpuErrchk( hipMalloc((void**)&O_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(O_element_d, &O_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );


    double *event_prob_cum;
    gpuErrchk( hipMalloc((void**)&event_prob_cum, N * nn * sizeof(double)) );
 
    double freq_host;
    gpuErrchk( hipMemcpy(&freq_host, freq, 1 * sizeof(double), hipMemcpyDeviceToHost) );


    double time_incl_sum = 0.0;
    double time_upper_bound = 0.0;
    double time_which_event = 0.0;
    double time_zero_prob = 0.0;

    double event_time = 0.0;
    int event_counter = 0;
    while (event_time < 1 / freq_host) {
        event_counter++;  
        // get the cumulative sum of the probabilities
        thrust::inclusive_scan(thrust::device, event_prob, event_prob + N * nn, event_prob_cum);

        // select an event
        double Psum_host;
        gpuErrchk( hipMemcpy(&Psum_host, event_prob_cum + N * nn - 1, sizeof(double), hipMemcpyDeviceToHost) );

        //TODO: cuda random number
        double number = rng.getRandomNumber() * Psum_host;
        int event_idx = thrust::upper_bound(thrust::device, event_prob_cum, event_prob_cum + N * nn, number) - event_prob_cum;
        // std::cout << "selected event: " << event_idx << "\n";

        EVENTTYPE sel_event_type = NULL_EVENT;
        gpuErrchk( hipMemcpy(&sel_event_type, event_type + event_idx, sizeof(EVENTTYPE), hipMemcpyDeviceToHost) );

        // test output:
        // double sel_event_prob;
        // gpuErrchk( hipMemcpy(&sel_event_prob, event_prob + event_idx, sizeof(double), hipMemcpyDeviceToHost) );
        // std::cout << "Selected event index: " << event_idx << " with type "
        //           << sel_event_type << " and probability " << sel_event_prob << std::endl;

        // get attributes of the sites involved:
        int i_host = static_cast<int>(floorf(event_idx / nn));
        int j_host;
        ELEMENT element_i_host, element_j_host;
        int charge_i_host, charge_j_host;

        gpuErrchk( hipMemcpy(&j_host, neigh_idx + event_idx, sizeof(int), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&element_i_host, site_element + i_host, sizeof(ELEMENT), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&element_j_host, site_element + j_host, sizeof(ELEMENT), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&charge_i_host, site_charge + i_host, sizeof(int), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(&charge_j_host, site_charge + j_host, sizeof(int), hipMemcpyDeviceToHost) );

        // ELEMENT *element_i_d = site_element + i_host;
        // ELEMENT *element_j_d = site_element + j_host;
        // int *charge_i_d = site_charge + i_host;
        // int *charge_j_d = site_charge + j_host;



        // Event execution loop
        switch (sel_event_type)
        {
        case VACANCY_GENERATION:
        {
            gpuErrchk( hipMemcpy(site_element + i_host, O_defect_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, vacancy_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, two_neg_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, two_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );



            break;
        }
        case VACANCY_RECOMBINATION:
        {
            gpuErrchk( hipMemcpy(site_element + i_host, defect_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, O_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, zero_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, zero_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );       

            break;
        }
        case VACANCY_DIFFUSION:
        {

            // problem: TODO swap needed (naive overwrite one first)
            // make swap kernel
            gpuErrchk( hipMemcpy(site_element + i_host, &element_j_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, &element_i_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, &charge_j_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, &charge_i_host, 1 * sizeof(int), hipMemcpyHostToDevice) );

            break;
        }
        case ION_DIFFUSION:
        {
            // gpuErrchk( hipMemcpy(site_element + i_host, element_j_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            // gpuErrchk( hipMemcpy(site_element + j_host, element_i_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            // gpuErrchk( hipMemcpy(site_charge + i_host, charge_j_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            // gpuErrchk( hipMemcpy(site_charge + j_host, charge_i_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );

            gpuErrchk( hipMemcpy(site_element + i_host, &element_j_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, &element_i_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, &charge_j_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, &charge_i_host, 1 * sizeof(int), hipMemcpyHostToDevice) );


            break;
        }
        default:
            print("error: unidentified event key found: ");
            print(sel_event_type);
        }

        int threads = 1024;
        int blocks = (N * nn + threads - 1) / threads;
        zero_out_events<<<blocks, threads>>>(event_type, event_prob,
            neigh_idx, N, nn, i_host, j_host);
        event_time = -log(rng.getRandomNumber()) / Psum_host;
    }


    std::cout << "Number of KMC events: " << event_counter << "\n";


    gpuErrchk( hipFree(event_prob_cum) );
    gpuErrchk( hipFree(event_type) );
    gpuErrchk( hipFree(event_prob) );

    return event_time;    
}

double execute_kmc_step_mpi(
        MPI_Comm comm,
        const int N,
        const int *count,
        const int *displs,
        const int nn, const int *neigh_idx, const int *site_layer,
        const double *lattice, const int pbc, const double *T_bg, 
        const double *freq, const double *sigma, const double *k,
        const double *posx, const double *posy, const double *posz, 
        const double *site_potential_boundary, const double *site_potential_charge, const double *site_temperature,
        ELEMENT *site_element, int *site_charge, RandomNumberGenerator &rng, const int *neigh_idx_host)
{


    int rank, size;
    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);

    // **************************
    // **** Build Event List ****
    // **************************

    double time_event_list = 0.0;

    // the KMC event list arrays only exist in gpu memory
    EVENTTYPE *event_type_local_d; 
    double    *event_prob_local_d; 
    gpuErrchk( hipMalloc((void**)&event_type_local_d, count[rank] * nn * sizeof(EVENTTYPE)) );
    gpuErrchk( hipMalloc((void**)&event_prob_local_d, count[rank] * nn * sizeof(double)) );
    double *event_prob_cum_local_d;
    gpuErrchk( hipMalloc((void**)&event_prob_cum_local_d, count[rank] * nn * sizeof(double)) );
    double *event_prob_cum_global_h;
    gpuErrchk(hipHostMalloc((void**)&event_prob_cum_global_h, size * sizeof(double)));
       

    int num_threads = 1024;
    int num_blocks = (count[rank] * nn - 1) / num_threads + 1;

    // populate the event_type and event_prob arrays:
    // only your part of the event list
    // TODO use COO
    build_event_list_split<<<num_blocks, num_threads>>>(N,
                                                count[rank], displs[rank],
                                                nn, neigh_idx, 
                                                site_layer, lattice, pbc,
                                                T_bg, freq, sigma, k,
                                                posx, posy, posz, 
                                                site_potential_boundary, site_potential_charge, site_temperature, 
                                                site_element, site_charge,
                                                event_type_local_d, event_prob_local_d);

    // **************************
    // ** Event Execution Loop **
    // **************************

    // helper variables:
    // NOTE: INITIALIZE THESE ON GPU AND USE MEMCPY DEVICETODEVICE INSTEAD
    int two_host = 2;
    int two_neg_host = -2;
    int zero_host = 0;
    ELEMENT defect_element_host = DEFECT;
    ELEMENT O_defect_element_host = OXYGEN_DEFECT;
    ELEMENT vacancy_element_host = VACANCY;
    ELEMENT O_element_host = O_EL;

    // constants on the device
    // TODO do only once
    int *two_d;
    gpuErrchk( hipMalloc((void**)&two_d, 1 * sizeof(int)) );
    gpuErrchk( hipMemcpy(two_d, &two_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
    int *two_neg_d;
    gpuErrchk( hipMalloc((void**)&two_neg_d, 1 * sizeof(int)) );
    gpuErrchk( hipMemcpy(two_neg_d, &two_neg_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
    int *zero_d;
    gpuErrchk( hipMalloc((void**)&zero_d, 1 * sizeof(int)) );
    gpuErrchk( hipMemcpy(zero_d, &zero_host, 1 * sizeof(int), hipMemcpyHostToDevice) );
    ELEMENT *defect_element_d;
    gpuErrchk( hipMalloc((void**)&defect_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(defect_element_d, &defect_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
    ELEMENT *O_defect_element_d;
    gpuErrchk( hipMalloc((void**)&O_defect_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(O_defect_element_d, &O_defect_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
    ELEMENT *vacancy_element_d;
    gpuErrchk( hipMalloc((void**)&vacancy_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(vacancy_element_d, &vacancy_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );
    ELEMENT *O_element_d;
    gpuErrchk( hipMalloc((void**)&O_element_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMemcpy(O_element_d, &O_element_host, 1 * sizeof(ELEMENT), hipMemcpyHostToDevice) );

    ELEMENT *element_i_d;
    ELEMENT *element_j_d;
    int *charge_i_d;
    int *charge_j_d;
    gpuErrchk( hipMalloc((void**)&element_i_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMalloc((void**)&element_j_d, 1 * sizeof(ELEMENT)) );
    gpuErrchk( hipMalloc((void**)&charge_i_d, 1 * sizeof(int)) );
    gpuErrchk( hipMalloc((void**)&charge_j_d, 1 * sizeof(int)) );


    int ijevent_to_delete[3];


    double event_time = 0.0;
    int event_counter = 0;

    double freq_h;
    gpuErrchk( hipMemcpy(&freq_h, freq, 1 * sizeof(double), hipMemcpyDeviceToHost) );
    while (event_time < 1 / freq_h) {
        event_counter++;  

        // get the cumulative sum of the probabilities
        thrust::inclusive_scan(thrust::device, event_prob_local_d, event_prob_local_d + count[rank] * nn, event_prob_cum_local_d);

        
        // select an event
        double Psum_host;
        gpuErrchk( hipMemcpy(&Psum_host, event_prob_cum_local_d + count[rank] * nn - 1, sizeof(double), hipMemcpyDeviceToHost) );
        
        MPI_Allgather(&Psum_host, 1, MPI_DOUBLE, event_prob_cum_global_h, 1, MPI_DOUBLE, comm);

        for (int i = 1; i < size; i++){
            event_prob_cum_global_h[i] += event_prob_cum_global_h[i-1];
        }

        //TODO: cuda random number
        double number = rng.getRandomNumber() * event_prob_cum_global_h[size-1];
        // figure out which rank has the number
        int source_rank;
        for (int i = 0; i < size; i++){
            if (number < event_prob_cum_global_h[i]){
                source_rank = i;
                break;
            }
        }

        if(rank == source_rank){
            // shift random number to the correct range
            if(rank > 0){
                number -= event_prob_cum_global_h[rank-1];
            }
        
            int event_idx = thrust::upper_bound(thrust::device, event_prob_cum_local_d, event_prob_cum_local_d + count[rank] * nn, number) - event_prob_cum_local_d;
            // std::cout << "selected event: " << event_idx << "\n";

            

            // get attributes of the sites involved:
            // i
            ijevent_to_delete[0] = event_idx / nn + displs[rank];
            // j
            gpuErrchk( hipMemcpy(&ijevent_to_delete[1], neigh_idx + displs[rank]*nn + event_idx , sizeof(int), hipMemcpyDeviceToHost) );
            // event type
            gpuErrchk( hipMemcpy(&ijevent_to_delete[2], event_type_local_d + event_idx, sizeof(EVENTTYPE), hipMemcpyDeviceToHost) );

            MPI_Bcast(ijevent_to_delete, 3, MPI_INT, source_rank, comm);
        }
        else{
            MPI_Bcast(ijevent_to_delete, 3, MPI_INT, source_rank, comm);
        }

        int i_host = ijevent_to_delete[0];
        int j_host = ijevent_to_delete[1];
        EVENTTYPE sel_event_type = static_cast<EVENTTYPE>(ijevent_to_delete[2]);

        gpuErrchk( hipMemcpy(element_i_d, site_element + i_host, sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
        gpuErrchk( hipMemcpy(element_j_d, site_element + j_host, sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
        gpuErrchk( hipMemcpy(charge_i_d, site_charge + i_host, sizeof(int), hipMemcpyDeviceToDevice) );
        gpuErrchk( hipMemcpy(charge_j_d, site_charge + j_host, sizeof(int), hipMemcpyDeviceToDevice) );

        // Event execution loop
        switch (sel_event_type)
        {
        case VACANCY_GENERATION:
        {
            gpuErrchk( hipMemcpy(site_element + i_host, O_defect_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, vacancy_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, two_neg_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, two_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );

            break;
        }
        case VACANCY_RECOMBINATION:
        {
            gpuErrchk( hipMemcpy(site_element + i_host, defect_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, O_element_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, zero_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, zero_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );       

            break;
        }
        case VACANCY_DIFFUSION:
        {
            gpuErrchk( hipMemcpy(site_element + i_host, element_j_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, element_i_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, charge_j_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, charge_i_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );

            break;
        }
        case ION_DIFFUSION:
        {
            gpuErrchk( hipMemcpy(site_element + i_host, element_j_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_element + j_host, element_i_d, 1 * sizeof(ELEMENT), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + i_host, charge_j_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );
            gpuErrchk( hipMemcpy(site_charge + j_host, charge_i_d, 1 * sizeof(int), hipMemcpyDeviceToDevice) );

            break;
        }
        default:
            print("error: unidentified event key found: ");
            print(sel_event_type);
        }

        int threads = 1024;
        int blocks = (count[rank] * nn + threads - 1) / threads;
        zero_out_events_split<<<blocks, threads>>>(event_type_local_d, event_prob_local_d, neigh_idx,
            count[rank], displs[rank],
            nn, i_host, j_host);
        event_time = -log(rng.getRandomNumber()) / event_prob_cum_global_h[size-1];
    }


    std::cout << "Number of KMC events: " << event_counter << "\n";


    gpuErrchk( hipFree(event_prob_cum_local_d) );
    gpuErrchk( hipFree(event_type_local_d) );
    gpuErrchk( hipFree(event_prob_local_d) );
    gpuErrchk(hipHostFree(event_prob_cum_global_h));
    gpuErrchk( hipFree(element_i_d) );
    gpuErrchk( hipFree(element_j_d) );
    gpuErrchk( hipFree(charge_i_d) );
    gpuErrchk( hipFree(charge_j_d) );


    // sleep(1);
    // MPI_Barrier(comm);
    // sleep(1);
    // exit(1);

    return event_time;    
}

#ifndef COMPILE_WITH_TESTS
void copytoConstMemory(std::vector<double> E_gen, std::vector<double> E_rec, std::vector<double> E_Vdiff, std::vector<double> E_Odiff)
{   
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_gen_const), E_gen.data(), E_gen.size() * sizeof(double)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_rec_const), E_rec.data(), E_rec.size() * sizeof(double)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_Vdiff_const), E_Vdiff.data(), E_Vdiff.size() * sizeof(double)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(E_Odiff_const), E_Odiff.data(), E_Odiff.size() * sizeof(double)) );
}
#endif

// *** The Graveyard of Code ***

    // dump A to file:
    // dump_csr_matrix_txt(N_interface, A_nnz, A_row_ptr_d, A_col_indices_d, A_data_d, kmc_step_count);

    // # if __CUDA_ARCH__>=200
    // printf("%i \n", tid);
    // #endif  

    // // debug - use floor() for i
    // std::vector<double> new_k(N * N);
    // gpuErrchk( hipMemcpy(new_k.data(), gpu_k , N * N * sizeof(double), hipMemcpyDeviceToHost) );
    // std::cout << "copied";
    // std::ofstream fout("new_k.txt");
    // for(int i = 0; i< N*N; i++){
    //     if (new_k[i] != 0){
    //         fout << new_k[i]; 
    //         fout << ' ';
    //     }
    // }
    // exit(1);

    // // debug
    // double *M = (double *)calloc(N, sizeof(double));
    // gpuErrchk( hipMemcpy(M, gpubuf.site_potential, N * sizeof(double), hipMemcpyDeviceToHost) );
    // std::cout << "copied\n";
    // std::ofstream fout2("gpu_M.txt");
    // for(int i = 0; i< N; i++){
    //     if (M[i] != 0){
    //         fout2 << M[i]; 
    //         fout2 << ' ';
    //     }
    // }


    // // ************************************************************
    // // 1. Convert D matrix to sparse COO format using the COOElement struct
    // int numNonZero = 0;

    // COOElement* d_cooData;
    // hipMalloc((void**)&d_cooData, N_interface * N_interface * sizeof(COOElement));

    // int* d_numNonZero;
    // hipMalloc((void**)&d_numNonZero, sizeof(int));
    // hipMemcpy(d_numNonZero, &numNonZero, sizeof(int), hipMemcpyHostToDevice);

    // int numThreadsPerBlock = 256;
    // int numBlocks = (N_interface + numThreadsPerBlock - 1) / numThreadsPerBlock;
    // extractCOOData<<<numBlocks, numThreadsPerBlock>>>(gpu_D, N_interface, d_cooData, d_numNonZero);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // hipMemcpy(&numNonZero, d_numNonZero, sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << "number of nonzeros in COO rep: " << numNonZero << "\n";

    // // ************************************************************
    // // 2. Sort and filter the coordinates
    // thrust::device_vector<COOElement> d_cooDataVec(d_cooData, d_cooData + numNonZero);
    
    // // Sort the COO data based on row-major order 
    // thrust::sort(thrust::device, d_cooDataVec.begin(), d_cooDataVec.end(), [] __device__ (const COOElement& a, const COOElement& b) {
    //     return a.row < b.row || (a.row == b.row && a.col < b.col);
    // });

    // // Remove duplicate entries 
    // auto newEnd = thrust::unique(thrust::device, d_cooDataVec.begin(), d_cooDataVec.end(), [] __device__ (const COOElement& a, const COOElement& b) {
    //     return a.row == b.row && a.col == b.col;
    // });
    // numNonZero = newEnd - d_cooDataVec.begin();

    // // ************************************************************
    // // 3. Extract CSR data from COO representation and nnz
    // int* d_csrRowPtr;
    // int* d_csrColIndices;
    // double* d_csrValues;

    // hipMalloc((void**)&d_csrRowPtr, (N_interface + 1) * sizeof(int));   // +1 for the row pointers
    // hipMalloc((void**)&d_csrColIndices, numNonZero * sizeof(int));      // Allocate space for the maximum number of non-zero elements
    // hipMalloc((void**)&d_csrValues, numNonZero * sizeof(double));       // Allocate space for the maximum number of non-zero elements
    // hipMemset(d_csrRowPtr, 0, (N_interface + 1) * sizeof(int));
    // gpuErrchk( hipDeviceSynchronize() );

    // extractCOOToCSR<<<numBlocks, numThreadsPerBlock>>>(thrust::raw_pointer_cast(d_cooDataVec.data()), numNonZero, N_interface, d_csrRowPtr, d_csrColIndices, d_csrValues);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // std::cout << "number of nonzeros in CSR rep: " << numNonZero << "\n";




// //debug
    // double *cpu_x = new double[(N_atom + 2) * (N_atom + 2)];
    // hipMemcpy(cpu_x, gpu_x, sizeof(double) * (N_atom + 2) * (N_atom + 2), hipMemcpyDeviceToHost);
    // std::cout << "printing X_gpu\n";
    // std::ofstream fout2("X_gpu_noT.txt");
    // int row, col;
    // for (row = 0; row < (N_atom + 2); row++) {
    // for (col = 0; col < (N_atom + 2); col++) {
    //     fout2 << cpu_x[row * (N_atom + 2) + col] << ' ';
    // }
    // fout2 << '\n';
    // }
    // fout2.close(); 
    // exit(1);
    // //debug end


    // // dense to sparse conversion:
    // hipsparseHandle_t cusparseHandle;
    // hipsparseCreate(&cusparseHandle);
    // hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_DEVICE);
    
    // gpuErrchk( hipMemcpy2D(gpu_A, (N_atom + 1) * sizeof(double), gpu_x, (N_atom + 2) * sizeof(double), (N_atom + 1) * sizeof(double), (N_atom + 1), hipMemcpyDeviceToDevice) );
    // int numRows = N_atom + 1;
    // int numCols = N_atom + 1;
    // double *csrValues;
    // int *csrRowPtr;
    // int *csrColInd;
    // int nnz = 0;
    // hipMalloc((void**) &csrRowPtr, (N_atom + 1 + 1) * sizeof(int));
    // denseToCSR(cusparseHandle, gpu_A, numRows, numCols, &csrValues, &csrRowPtr, &csrColInd, &nnz);
    // std::cout << "dense nnz: " << nnz << "\n";
